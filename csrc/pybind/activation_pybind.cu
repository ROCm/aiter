#include "hip/hip_runtime.h"
// SPDX-License-Identifier: MIT
// Copyright (c) 2024, Advanced Micro Devices, Inc. All rights reserved.
#include "activation.h"

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    m.def("silu_and_mul", &silu_and_mul, "Activation function used in SwiGLU.");
    m.def("gelu_and_mul", &gelu_and_mul, "Activation function used in GELU.");
    m.def("gelu_tanh_and_mul", &gelu_tanh_and_mul, "Activation function used in GELU tanh.");
}