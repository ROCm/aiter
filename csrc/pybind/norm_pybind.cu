#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024 Advanced Micro Devices, Inc.  All rights reserved.
 *
 * @Script: norm_pybind.cu
 * @Author: valarLip
 * @Email: lingpeng.jin@amd.com
 * @Create At: 2024-12-02 16:00:01
 * @Last Modified By: valarLip
 * @Last Modified At: 2025-01-03 16:34:45
 * @Description: This is description.
 */

#include "norm.h"

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    m.def("layernorm2d_fwd", &layernorm2d,
          py::arg("input"), py::arg("weight"), py::arg("bias"),
          py::arg("epsilon"), py::arg("x_bias") = std::nullopt);
    m.def("layernorm2d_fwd_with_add", &layernorm2d_with_add,
          py::arg("out"), py::arg("input"),
          py::arg("residual_in"), py::arg("residual_out"),
          py::arg("weight"), py::arg("bias"),
          py::arg("epsilon"), py::arg("x_bias") = std::nullopt);
    m.def("layernorm2d_fwd_with_smoothquant", &layernorm2d_with_smoothquant,
          py::arg("out"), py::arg("input"),
          py::arg("xscale"), py::arg("yscale"),
          py::arg("weight"), py::arg("bias"),
          py::arg("epsilon"), py::arg("x_bias") = std::nullopt);
    m.def("layernorm2d_fwd_with_add_smoothquant", &layernorm2d_with_add_smoothquant,
          py::arg("out"), py::arg("input"),
          py::arg("residual_in"), py::arg("residual_out"),
          py::arg("xscale"), py::arg("yscale"),
          py::arg("weight"), py::arg("bias"),
          py::arg("epsilon"), py::arg("x_bias") = std::nullopt);
    m.def("layernorm2d_fwd_with_dynamicquant", &layernorm2d_with_dynamicquant,
          py::arg("out"), py::arg("input"), 
          py::arg("yscale"), py::arg("weight"), py::arg("bias"),
          py::arg("epsilon"), py::arg("x_bias") = std::nullopt);
    m.def("layernorm2d_fwd_with_add_dynamicquant", &layernorm2d_with_add_dynamicquant,
          py::arg("out"), py::arg("input"),
          py::arg("residual_in"), py::arg("residual_out"),
          py::arg("yscale"), py::arg("weight"), py::arg("bias"),
          py::arg("epsilon"), py::arg("x_bias") = std::nullopt);
    // following are asm kernels
    m.def("layernorm2d_with_add_asm", &layernorm2d_with_add_asm);
    m.def("layernorm2d_with_add_smoothquant_asm", &layernorm2d_with_add_smoothquant_asm);
}