// SPDX-License-Identifier: MIT
// Copyright (C) 2024-2025, Advanced Micro Devices, Inc. All rights reserved.
#include "rocm_ops.hpp"
#include "topk_per_row.h"

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    TOPK_PER_ROW_PYBIND;
}
