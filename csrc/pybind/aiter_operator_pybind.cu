#include "hip/hip_runtime.h"
// SPDX-License-Identifier: MIT
// Copyright (c) 2024, Advanced Micro Devices, Inc. All rights reserved.
#include "aiter_operator.h"

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    m.def("add", &aiter_add, "apply for add with transpose and broadcast.");
    m.def("mul", &aiter_mul, "apply for mul with transpose and broadcast.");
    m.def("sub", &aiter_sub, "apply for sub with transpose and broadcast.");
    m.def("div", &aiter_div, "apply for div with transpose and broadcast.");
    m.def("add_", &aiter_add_, "apply for add_ with transpose and broadcast.");
    m.def("mul_", &aiter_mul_, "apply for mul_ with transpose and broadcast.");
    m.def("sub_", &aiter_sub_, "apply for sub_ with transpose and broadcast.");
    m.def("div_", &aiter_div_, "apply for div_ with transpose and broadcast.");
    m.def("sigmoid", &aiter_sigmoid, "apply for sigmoid.");
    m.def("tanh", &aiter_tanh, "apply for tanh.");
}
