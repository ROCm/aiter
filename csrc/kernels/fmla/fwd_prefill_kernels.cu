#include "hip/hip_runtime.h"
// SPDX-License-Identifier: MIT
// Copyright (c) 2025, Advanced Micro Devices, Inc. All rights reserved.

#include <ck_tile/ops/fmha.hpp>
#include <ck_tile/core/tensor/tile_scatter_gather.hpp>
#include "fwd_kernels_params.hpp"
#include "fwd_prefill_kernels_pipelines.hpp"

// =====================================================================================================================
// Kernel Functions
//

template <typename Traits>
CK_TILE_DEVICE static auto GetTileIndex(const int32_t num_splits)
{
    const auto f = [](int32_t dividend, int32_t divisor) {
        int32_t quotient = dividend / divisor;
        int32_t modulus  = dividend - quotient * divisor;
        return ck_tile::make_tuple(quotient, modulus);
    };

    const auto [mid, split_id] = f(blockIdx.x, num_splits);
    const int32_t hid          = blockIdx.y;
    const int32_t bid          = blockIdx.z;

    return ck_tile::make_tuple(mid, split_id, hid, bid);
}

template <typename Policy, typename scalar_t = typename Policy::InOutType>
CK_TILE_DEVICE static auto MakeQDram(const scalar_t* p_data,
                                     const int32_t size_s_ori,
                                     const int32_t stride_s,
                                     const int32_t hq_hk_ratio,
                                     const int32_t stride_h)
{
    using Traits = typename Policy::Traits;

    const auto q_dram_naive = [&] {
        if constexpr(Traits::kXqaStrategy == XqaStrategy::Internal)
        {
            const auto view = ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s_ori, hq_hk_ratio, Traits::kSizeD),
                ck_tile::make_tuple(stride_s, stride_h, 1),
                ck_tile::number<Policy::GetAlignmentQ()>{},
                ck_tile::number<1>{});
            return ck_tile::transform_tensor_view(
                view,
                ck_tile::make_tuple(
                    ck_tile::make_merge_transform(ck_tile::make_tuple(size_s_ori, hq_hk_ratio)),
                    ck_tile::make_pass_through_transform(Traits::kSizeD)),
                ck_tile::make_tuple(ck_tile::sequence<0, 1>{}, ck_tile::sequence<2>{}),
                ck_tile::make_tuple(ck_tile::sequence<0>{}, ck_tile::sequence<1>{}));
        }
        else
        {
            return ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s_ori, Traits::kSizeD),
                ck_tile::make_tuple(stride_s, 1),
                ck_tile::number<Policy::GetAlignmentQ()>{},
                ck_tile::number<1>{});
        }
    }();

    return ck_tile::pad_tensor_view(
        q_dram_naive,
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{}, ck_tile::number<Traits::kBlockK0>{}),
        ck_tile::sequence<false, Traits::kPadHeadDimQ>{});
}

template <typename Policy, typename scalar_t = typename Policy::InOutType>
CK_TILE_DEVICE static auto MakeKDram(
    const scalar_t* p_data,
    const int32_t   height,
    const int32_t   stride_s)
{
    using Traits = typename Policy::Traits;

    const auto k_dram_naive = ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
        p_data, // will update this pointer if using paged-kvcache
        ck_tile::make_tuple(height, Traits::kSizeD),
        ck_tile::make_tuple(stride_s, 1),
        ck_tile::number<Policy::GetAlignmentK()>{},
        ck_tile::number<1>{});

    return ck_tile::pad_tensor_view(
        k_dram_naive,
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockN0>{}, ck_tile::number<Traits::kBlockK0>{}),
        ck_tile::sequence<true, Traits::kPadHeadDimQ>{});
}

template <typename Policy, typename scalar_t = typename Policy::InOutType>
CK_TILE_DEVICE static auto MakeVDram(
    const scalar_t* p_data,
    const int32_t   length,
    const int32_t   stride_s)
{
    using Traits = typename Policy::Traits;

    // Assuming Value is row-major just like Key.
    const auto v_dram_naive = ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
        p_data, // will update this pointer if using paged-kvcache
        ck_tile::make_tuple(length, Traits::kSizeDV),
        ck_tile::make_tuple(stride_s, 1),
        ck_tile::number<Policy::GetAlignmentV()>{},
        ck_tile::number<1>{});

    const auto v_dram_transposed = ck_tile::transform_tensor_view(
        v_dram_naive,
        ck_tile::make_tuple(ck_tile::make_pass_through_transform(Traits::kSizeDV),
                            ck_tile::make_pass_through_transform(length)),
        ck_tile::make_tuple(ck_tile::sequence<1>{}, ck_tile::sequence<0>{}),
        ck_tile::make_tuple(ck_tile::sequence<0>{}, ck_tile::sequence<1>{}));

    return ck_tile::pad_tensor_view(
        v_dram_transposed,
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockN1>{},
                            ck_tile::number<Traits::kBlockK1>{}),
        ck_tile::sequence<Traits::kPadHeadDimV, Traits::kPadSeqLenK>{});
}

template <typename Policy, typename Lengths, typename scalar_t>
CK_TILE_DEVICE static auto MakeLseAccDram(scalar_t* p_data,
                                          const Lengths& window_lengths,
                                          const int32_t size_s_ori,
                                          const int32_t hq_hk_ratio,
                                          const int32_t stride_h)
{
    using Traits = typename Policy::Traits;

    const auto lse_acc_dram_naive = [&] {
        if constexpr(Traits::kXqaStrategy == XqaStrategy::Internal)
        {
            // transpose + merge: (hq_hk_ratio, seqlen_q) -> (seqlenq * hq_hk_ratio)
            const auto view = ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(hq_hk_ratio, size_s_ori),
                ck_tile::make_tuple(stride_h, 1),
                ck_tile::number<1>{},
                ck_tile::number<1>{});
            return ck_tile::transform_tensor_view(
                view,
                ck_tile::make_tuple(
                    ck_tile::make_merge_transform(ck_tile::make_tuple(size_s_ori, hq_hk_ratio))),
                ck_tile::make_tuple(ck_tile::sequence<1, 0>{}),
                ck_tile::make_tuple(ck_tile::sequence<0>{}));
        }
        else
        {
            return ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s_ori),
                ck_tile::make_tuple(1),
                ck_tile::number<1>{},
                ck_tile::number<1>{});
        }
    }();

    return ck_tile::pad_tensor_view(
        lse_acc_dram_naive,
        window_lengths,
        ck_tile::sequence<Traits::kPadSeqLenQ>{});
}

template <typename Policy, typename scalar_t>
CK_TILE_DEVICE static auto MakeOutAccDram(scalar_t* p_data,
                                          const int32_t size_s_ori,
                                          const int32_t stride_s,
                                          const int32_t hq_hk_ratio,
                                          const int32_t stride_h)
{
    using Traits = typename Policy::Traits;

    const auto o_acc_dram_naive = [&] {
        if constexpr(Traits::kXqaStrategy == XqaStrategy::Internal)
        {
            // merge: (seqlen_q, hq_hk_ratio, headdim) -> (seqlen_q*hq_hk_ratio, headdim)
            const auto view = ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s_ori, hq_hk_ratio, Traits::kSizeDV),
                ck_tile::make_tuple(stride_s, stride_h, 1),
                ck_tile::number<Policy::GetAlignmentOacc()>{},
                ck_tile::number<1>{});
            return ck_tile::transform_tensor_view(
                view,
                ck_tile::make_tuple(
                    ck_tile::make_merge_transform(ck_tile::make_tuple(size_s_ori, hq_hk_ratio)),
                    ck_tile::make_pass_through_transform(Traits::kSizeDV)),
                ck_tile::make_tuple(ck_tile::sequence<0, 1>{}, ck_tile::sequence<2>{}),
                ck_tile::make_tuple(ck_tile::sequence<0>{}, ck_tile::sequence<1>{}));
        }
        else
        {
            return ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s_ori, Traits::kSizeDV),
                ck_tile::make_tuple(stride_s, 1),
                ck_tile::number<Policy::GetAlignmentOacc()>{},
                ck_tile::number<1>{});
        }
    }();

    return ck_tile::pad_tensor_view(
        o_acc_dram_naive,
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{}, ck_tile::number<Traits::kBlockN1>{}),
        ck_tile::sequence<Traits::kPadSeqLenQ, Traits::kPadHeadDimV>{});
}

template <typename Policy, typename Lengths, typename scalar_t>
CK_TILE_DEVICE static auto MakeLseDram(scalar_t* p_data,
                                       const Lengths& window_lenghts,
                                       const int32_t size_s_ori,
                                       const int32_t hq_hk_ratio,
                                       const int32_t stride_h)
{
    using Traits = typename Policy::Traits;

    const auto lse_dram_naive = [&] {
        if constexpr(Traits::kXqaStrategy == XqaStrategy::Internal)
        {
            // transpose + merge: (hq_hk_ratio, seqlen_q) -> (seqlenq * hq_hk_ratio)
            const auto view = ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(hq_hk_ratio, size_s_ori),
                ck_tile::make_tuple(stride_h, 1),
                ck_tile::number<Policy::GetAlignmentLse()>{},
                ck_tile::number<1>{});
            return ck_tile::transform_tensor_view(
                view,
                ck_tile::make_tuple(
                    ck_tile::make_merge_transform(ck_tile::make_tuple(size_s_ori, hq_hk_ratio))),
                ck_tile::make_tuple(ck_tile::sequence<1, 0>{}),
                ck_tile::make_tuple(ck_tile::sequence<0>{}));
        }
        else
        {

            return ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s_ori),
                ck_tile::make_tuple(1),
                ck_tile::number<Policy::GetAlignmentLse()>{},
                ck_tile::number<1>{});
        }
    }();

    return ck_tile::pad_tensor_view(
        lse_dram_naive, window_lenghts, ck_tile::sequence<Traits::kPadSeqLenQ>{});
}

template <typename Policy, typename scalar_t>
CK_TILE_DEVICE static auto MakeOutDram(scalar_t* p_data,
                                       const int32_t size_s_ori,
                                       const int32_t stride_s,
                                       const int32_t hq_hk_ratio,
                                       const int32_t stride_h)
{
    using Traits = typename Policy::Traits;

    const auto o_dram_naive = [&] {
        if constexpr(Traits::kXqaStrategy == XqaStrategy::Internal)
        {
            // merge: (seqlen_q, hq_hk_ratio, headdim) -> (seqlen_q * hq_hk_ratio, headdim)
            const auto view = ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s_ori, hq_hk_ratio, Traits::kSizeDV),
                ck_tile::make_tuple(stride_s, stride_h, 1),
                ck_tile::number<Policy::GetAlignmentO()>{},
                ck_tile::number<1>{});
            return ck_tile::transform_tensor_view(
                view,
                ck_tile::make_tuple(
                    ck_tile::make_merge_transform(ck_tile::make_tuple(size_s_ori, hq_hk_ratio)),
                    ck_tile::make_pass_through_transform(Traits::kSizeDV)),
                ck_tile::make_tuple(ck_tile::sequence<0, 1>{}, ck_tile::sequence<2>{}),
                ck_tile::make_tuple(ck_tile::sequence<0>{}, ck_tile::sequence<1>{}));
        }
        else
        {
            return ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s_ori, Traits::kSizeDV),
                ck_tile::make_tuple(stride_s, 1),
                ck_tile::number<Policy::GetAlignmentO()>{},
                ck_tile::number<1>{});
        }
    }();

    return ck_tile::pad_tensor_view(
        o_dram_naive,
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{}, ck_tile::number<Traits::kBlockN1>{}),
        ck_tile::sequence<Traits::kPadSeqLenQ, Traits::kPadHeadDimV>{});
}
// =====================================================================================================================
// Kernel Entry
//

template <typename Traits, typename scalar_t, typename acc_t, typename out_t, bool kIsCausal, bool kDoSplit>
__launch_bounds__(Traits::kNumThreads, Traits::kWaveOccupancy)
__global__ void kn_fmla_fwd_splictkv_prefill(
    const FlashMlaPrefillFwdParams params)
{
    using Policy = FlashMlaPrefillPolicy<Traits, scalar_t, acc_t>;

    // allocate LDS
    __shared__ uint8_t p_smem[Policy::GetSmemSize()];

    const auto [tile_m_id, split_id, hqid, bid] =
        kDoSplit ? GetTileIndex<Traits>(params.num_splits) : GetTileIndex<Traits>(1);
    const auto hqid_xqa =
        (Traits::kXqaStrategy == XqaStrategy::Internal) ? hqid * params.hq_hk_ratio : hqid;
    const auto hkid   = hqid_xqa / params.hq_hk_ratio;
    const int32_t mid = __builtin_amdgcn_readfirstlane(tile_m_id * Traits::kBlockM);

    constexpr bool enableXqa = (Traits::kXqaStrategy != XqaStrategy::Disable);
    // Define causal mask
    using Mask             = ck_tile::SimplifiedGenericAttentionMask<kIsCausal, enableXqa>;
    const int32_t seqlen_k = __builtin_amdgcn_readfirstlane(params.p_seqlens_k[bid]);
    Mask mask              = kIsCausal ? Mask{params.size_s_ori,
                                              seqlen_k - params.size_s_ori + 1,
                                              params.size_s_pk,
                                              seqlen_k,
                                              params.mask_y_ratio_mdiv}
                                       : Mask{params.size_s_pk, seqlen_k};

    constexpr auto q_dram_window_lengths =
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{}, ck_tile::number<Traits::kBlockK0>{});
    constexpr auto q_nope_dram_window_lengths =
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{}, ck_tile::number<Traits::kSizeNope>{});
    constexpr auto q_rope_dram_window_lengths =
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{}, ck_tile::number<Traits::kSizeRope>{});
    constexpr auto k_dram_window_lengths =
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockN0>{}, ck_tile::number<Traits::kBlockK0>{});
    constexpr auto v_dram_window_lengths =
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockN1>{}, ck_tile::number<Traits::kBlockK1>{});

    const scalar_t* p_query = reinterpret_cast<const scalar_t*>(params.p_query) +
                              int64_t(hqid_xqa) * params.stride_h_q +   // head offset
                              int64_t(bid) * params.stride_b_q;     // batch offset
    const scalar_t* p_key   = reinterpret_cast<const scalar_t*>(params.p_key) +
                              int64_t(hkid) * params.stride_h_k;    // head offset
    const scalar_t* p_value = reinterpret_cast<const scalar_t*>(params.p_value) +
                              int64_t(hkid) * params.stride_h_v;    // head offset
    const int32_t*  p_block_table = params.p_block_table +
                                    int64_t(bid) * params.block_table_batch_stride; // batch offset

    const int32_t kv_cache_width = params.num_page_blocks * params.page_block_size;

    const auto q_dram = MakeQDram<Policy>(
        p_query, params.size_s_tr, params.stride_s_q, params.hq_hk_ratio, params.stride_h_q);
    const auto k_dram = MakeKDram<Policy>(p_key,   kv_cache_width, params.stride_s_k);
    const auto v_dram = MakeVDram<Policy>(p_value, kv_cache_width, params.stride_s_v);    

    auto q_dram_window = ck_tile::make_tile_window(q_dram, q_dram_window_lengths, {mid, 0});
    auto k_dram_window = ck_tile::make_tile_window(k_dram, k_dram_window_lengths, {0, 0});
    auto v_dram_window = ck_tile::make_tile_window(v_dram, v_dram_window_lengths, {0, 0});

    if constexpr (kDoSplit)
    {
        acc_t* p_lse_acc = reinterpret_cast<acc_t*>(params.p_softmax_lseaccum) +
                           int64_t(hqid_xqa) * params.stride_h_lseacc +     // head offset
                           int64_t(bid) * params.stride_b_lseacc +      // batch offset
                           int64_t(split_id) * params.stride_sp_lseacc; // split offset
        out_t* p_out_acc = reinterpret_cast<out_t*>(params.p_output_accum) +
                           int64_t(hqid_xqa) * params.stride_h_oacc +      // head offset
                           int64_t(bid) * params.stride_b_oacc +       // batch offset
                           int64_t(split_id) * params.stride_sp_oacc;  // split offset

        auto lse_acc_dram_window_lengths =
            ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{});
        auto out_acc_dram_window_lengths =
            ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{}, ck_tile::number<Traits::kBlockN1>{});

        const auto lse_acc_dram = MakeLseAccDram<Policy>(p_lse_acc,
                                                         lse_acc_dram_window_lengths,
                                                         params.size_s_tr,
                                                         params.hq_hk_ratio,
                                                         params.stride_h_lseacc);
        const auto out_acc_dram = MakeOutAccDram<Policy>(p_out_acc,
                                                         params.size_s_tr,
                                                         params.stride_s_oacc,
                                                         params.hq_hk_ratio,
                                                         params.stride_h_oacc);

        auto lse_acc_dram_window =
            ck_tile::make_tile_window(lse_acc_dram, lse_acc_dram_window_lengths, {mid});
        auto out_acc_dram_window =
            ck_tile::make_tile_window(out_acc_dram, out_acc_dram_window_lengths, {mid, 0});


        if constexpr (!Traits::kKVLoadOnce) {
            kn_fmla_fwd_splitkv_prefill_tile<Traits, scalar_t, acc_t, out_t>(
                q_dram_window,
                k_dram_window,
                v_dram_window,
                lse_acc_dram_window,
                out_acc_dram_window,
                p_block_table,
                __builtin_amdgcn_readfirstlane(params.page_block_size),
                __builtin_amdgcn_readfirstlane(params.stride_s_k),
                __builtin_amdgcn_readfirstlane(params.stride_s_v),
                seqlen_k,
                params.num_splits,
                split_id,
                mask,
#if FMLA_FWD_FAST_EXP2
                static_cast<float>(params.scale_softmax * ck_tile::log2e_v<>),
#else
                params.scale_softmax,
#endif
                p_smem);
        }
        else
        {
            const auto q_nope_dram_window = ck_tile::make_tile_window(
                q_dram,
                q_nope_dram_window_lengths,
                {mid, 0});
            const auto q_rope_dram_window = ck_tile::make_tile_window(
                q_dram,
                q_rope_dram_window_lengths,
                {mid, Traits::kSizeNope});
            kn_fmla_fwd_splitkv_prefill_load_once_tile<Traits, scalar_t, acc_t, out_t>(
                q_nope_dram_window,
                q_rope_dram_window,
                k_dram,
                lse_acc_dram_window,
                out_acc_dram_window,
                p_block_table,
                __builtin_amdgcn_readfirstlane(params.page_block_size),
                __builtin_amdgcn_readfirstlane(params.stride_s_k),
                __builtin_amdgcn_readfirstlane(params.stride_s_v),
                seqlen_k,
                params.num_splits,
                split_id,
                mask,
#if FMLA_FWD_FAST_EXP2
                static_cast<float>(params.scale_softmax * ck_tile::log2e_v<>),
#else
                params.scale_softmax,
#endif
                p_smem);
        }
    }
    else
    {
        // Assuming lse is in shape [b, h, s] and is contiguous
        acc_t* p_lse =
            reinterpret_cast<acc_t*>(params.p_softmax_lse) +
            (int64_t(bid) * params.size_h_tr + hqid_xqa) * params.size_s_tr; // batch+head offset
        out_t* p_out = reinterpret_cast<out_t*>(params.p_output) +
                       int64_t(hqid_xqa) * params.stride_h_o + // head offset
                       int64_t(bid) * params.stride_b_o;       // batch offset

        auto lse_dram_window_lengths =
            ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{});
        auto out_dram_window_lengths =
            ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{}, ck_tile::number<Traits::kBlockN1>{});

        const auto lse_dram = MakeLseDram<Policy>(p_lse,
                                                  lse_dram_window_lengths,
                                                  params.size_s_tr,
                                                  params.hq_hk_ratio,
                                                  params.stride_h_lse);
        const auto out_dram = MakeOutDram<Policy>(
            p_out, params.size_s_tr, params.stride_s_o, params.hq_hk_ratio, params.stride_h_o);

        auto lse_dram_window =
            ck_tile::make_tile_window(lse_dram, lse_dram_window_lengths, {mid});
        auto out_dram_window =
            ck_tile::make_tile_window(out_dram, out_dram_window_lengths, {mid, 0});

        if constexpr (!Traits::kKVLoadOnce)
        {
            kn_fmla_fwd_splitkv_prefill_tile<Traits, scalar_t, acc_t, out_t>(
                q_dram_window,
                k_dram_window,
                v_dram_window,
                lse_dram_window,
                out_dram_window,
                p_block_table,
                __builtin_amdgcn_readfirstlane(params.page_block_size),
                __builtin_amdgcn_readfirstlane(params.stride_s_k),
                __builtin_amdgcn_readfirstlane(params.stride_s_v),
                seqlen_k,
                1, // num_splits
                0, // split_id
                mask,
#if FMLA_FWD_FAST_EXP2
                static_cast<float>(params.scale_softmax * ck_tile::log2e_v<>),
#else
                params.scale_softmax,
#endif
                p_smem);
        }
        else
        {
            const auto q_nope_dram_window = ck_tile::make_tile_window(
                q_dram,
                q_nope_dram_window_lengths,
                {mid, 0});
            const auto q_rope_dram_window = ck_tile::make_tile_window(
                q_dram,
                q_rope_dram_window_lengths,
                {mid, Traits::kSizeNope});
            kn_fmla_fwd_splitkv_prefill_load_once_tile<Traits, scalar_t, acc_t, out_t>(
                q_nope_dram_window,
                q_rope_dram_window,
                k_dram,
                lse_dram_window,
                out_dram_window,
                p_block_table,
                __builtin_amdgcn_readfirstlane(params.page_block_size),
                __builtin_amdgcn_readfirstlane(params.stride_s_k),
                __builtin_amdgcn_readfirstlane(params.stride_s_v),
                seqlen_k,
                1, // num_splits
                0, // split_id
                mask,
#if FMLA_FWD_FAST_EXP2
                static_cast<float>(params.scale_softmax * ck_tile::log2e_v<>),
#else
                params.scale_softmax,
#endif
                p_smem);
        }
    }
}

template <typename Traits, int32_t kMaxSplits, typename out_t, typename in_t>
__global__ void kn_fmla_fwd_splictkv_prefill_combine(
    const FlashMlaPrefillFwdParams params)
{
    using Policy  = FlashMlaCombineKernelPolicy<Traits, out_t, in_t>;
    using index_t = int64_t;

    __shared__ in_t lds_lse_scale[kMaxSplits];

    const int32_t bidx = blockIdx.z;

    const int32_t num_splits   = params.num_splits;
    const int32_t split_offset = bidx * params.num_splits;
    assert((num_splits > 1) && (num_splits <= kMaxSplits));

    const int32_t lane_id          = ck_tile::get_lane_id();
    const int32_t hidx             = blockIdx.y;
    const int32_t sidx             = blockIdx.x;
    const int32_t hsidx            = hidx * params.size_s_tr + sidx;
    const int32_t shidx            = hidx + sidx * params.size_h_tr;
    const int32_t size_hs          = params.size_h_tr * params.size_s_tr;
    const index_t offset_lse_accum = split_offset * size_hs + hsidx; // offset to split 0
    const index_t offset_lse       = bidx * size_hs + hsidx;

    if (ck_tile::get_warp_id() == 0)
    {
        const in_t* p_lse_accum = reinterpret_cast<in_t*>(params.p_softmax_lseaccum) + offset_lse_accum;
        in_t* p_lse             = reinterpret_cast<in_t*>(params.p_softmax_lse) + offset_lse;

        constexpr int32_t kNumLsePerThr = ck_tile::integer_divide_ceil(kMaxSplits, ck_tile::get_warp_size());
        in_t local_lse[kNumLsePerThr];

        // Load thread local LSE and get local max LSE
        in_t max_lse = -ck_tile::numeric<in_t>::infinity();
        #pragma unroll
        for (int32_t i = 0; i < kNumLsePerThr; ++i)
        {
            const int32_t split_idx = i * ck_tile::get_warp_size() + lane_id;
            const in_t lse =
                (split_idx < num_splits) ? p_lse_accum[split_idx * size_hs] : -ck_tile::numeric<in_t>::infinity();
            local_lse[i] = lse;
            max_lse = ck_tile::max(max_lse, lse);
        }

        // Get global max LSE
        #pragma unroll
        for (int32_t offset = ck_tile::get_warp_size() / 2; offset > 0; offset /= 2)
        {
            max_lse = ck_tile::max(max_lse, __shfl_xor(max_lse, offset));
        }

        // Get sum of LSE
        in_t sum_lse = 0.f;
        #pragma unroll
        for (int32_t i = 0; i < kNumLsePerThr; ++i)
        {
#ifdef FMLA_FWD_FAST_EXP2
            static_assert(0, "have not figured out if need exp2 here");
#endif
            sum_lse += ck_tile::exp(local_lse[i] - max_lse);
        }
        #pragma unroll
        for (int32_t offset = ck_tile::get_warp_size() / 2; offset > 0; offset /= 2)
        {
            sum_lse += __shfl_xor(sum_lse, offset);
        }

        // Get global LSE
        const auto [global_lse, output_lse] = [&]() {
            if ((sum_lse == 0.f) || (sum_lse != sum_lse))
            {
                return ck_tile::make_tuple(ck_tile::numeric<in_t>::infinity(), -ck_tile::numeric<in_t>::infinity());
            }
            else
            {
                const in_t lse = ck_tile::log(sum_lse) + max_lse;
                return ck_tile::make_tuple(lse, lse);
            }
        } ();

        if (lane_id == 0)
        {
            *p_lse = output_lse;
        }

        // Write LSE to LDS
        #pragma unroll
        for (int32_t i = 0; i < kNumLsePerThr; ++i)
        {
            const int32_t split_idx = i * ck_tile::get_warp_size() + lane_id;
            if (split_idx < num_splits)
            {
                lds_lse_scale[split_idx] = ck_tile::exp(local_lse[i] - global_lse);
            }
        }
    }

    __builtin_amdgcn_sched_barrier(0);
    ck_tile::block_sync_lds();

    static_assert(Traits::kSizeDV % Traits::kNumThreadsCombine == 0);

    auto oaccu_window =
        Policy::MakeOaccuTileWindow(params.p_output_accum, shidx, size_hs, split_offset, num_splits);

    auto reg_out = ck_tile::make_static_distributed_tensor<in_t>(
        decltype(ck_tile::load_tile(oaccu_window))::get_tile_distribution());
    ck_tile::set_tile(reg_out, 0.f);

    for (int32_t split_idx = 0; split_idx < num_splits; ++split_idx)
    {
        const in_t lse_scale = lds_lse_scale[split_idx];
        auto oaccu = ck_tile::load_tile(oaccu_window);
        ck_tile::sweep_tile(oaccu, [&](auto idx) {
            reg_out(idx) += lse_scale * oaccu(idx);
        });
        ck_tile::move_tile_window(oaccu_window, {size_hs, 0});
    }

    auto dram_out = Policy::MakeOutputTileWindow(
        static_cast<out_t*>(params.p_output) +
        bidx * params.stride_b_o + hidx * params.stride_h_o + sidx * params.stride_s_o);
    ck_tile::store_tile(dram_out, ck_tile::cast_tile<out_t>(reg_out));
}

// =====================================================================================================================
// Dispatch
//

template <typename Traits, typename scalar_t, typename acc_t, typename out_t, bool kIsCausal>
void dispatch_fmla_fwd_splictkv_prefill(
    const FlashMlaPrefillFwdParams& params)
{
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const int32_t num_blk =
        ck_tile::integer_divide_ceil(params.size_s_pk, Traits::kBlockM) * params.num_splits;
    const dim3 grid_attn = dim3(num_blk, params.size_h_pk, params.size_b);
    const dim3 grid_comb = dim3(params.size_s_tr, params.size_h_tr, params.size_b);


    if (params.num_splits > 1)
    {
        // out_t is not take into consideration when doing splits because combine shader is always expected to do
        // the final output type conversion.
        auto kn_attn = &kn_fmla_fwd_splictkv_prefill<Traits, scalar_t, acc_t, acc_t, kIsCausal, true>;
        auto kn_comb =
            (params.num_splits <= 32)  ? &kn_fmla_fwd_splictkv_prefill_combine<Traits, 32,  scalar_t, acc_t> :
            // (params.num_splits <= 64)  ? &kn_fmla_fwd_splictkv_prefill_combine<Traits, 64,  scalar_t, acc_t> :
            // (params.num_splits <= 96)  ? &kn_fmla_fwd_splictkv_prefill_combine<Traits, 96,  scalar_t, acc_t> :
            // (params.num_splits <= 128) ? &kn_fmla_fwd_splictkv_prefill_combine<Traits, 128, scalar_t, acc_t> :
            static_cast<decltype(kn_fmla_fwd_splictkv_prefill_combine<Traits, 32, scalar_t, acc_t>)*>(nullptr);
        TORCH_CHECK(kn_comb != nullptr, "num_splits is larger than expected (<=128) !");
        kn_attn<<<grid_attn, Traits::kNumThreads, 0, stream>>>(params);
        kn_comb<<<grid_comb, Traits::kNumThreadsCombine, 0, stream>>>(params);
    }
    else
    {
        auto kn_attn = &kn_fmla_fwd_splictkv_prefill<Traits, scalar_t, acc_t, out_t, kIsCausal, false>;
        kn_attn<<<grid_attn, Traits::kNumThreads, 0, stream>>>(params);
    }
}

// =====================================================================================================================
// Interfaces
//
#define DISPATCH_FMLA_TYPES(TYPE, IS_CAUSAL, NAME, ...)                      \
    switch ((TYPE))                                                          \
    {                                                                        \
        case at::ScalarType::BFloat16:                                       \
        {                                                                    \
            using scalar_t = ck_tile::bf16_t;                                \
            using out_t = std::conditional_t<kForceOutAcc, acc_t, scalar_t>; \
            if ((IS_CAUSAL))                                                 \
            {                                                                \
                constexpr bool Is_causal = true;                             \
                __VA_ARGS__;                                                 \
            }                                                                \
            else                                                             \
            {                                                                \
                constexpr bool Is_causal = false;                            \
                __VA_ARGS__;                                                 \
            }                                                                \
            break;                                                           \
        }                                                                    \
        case at::ScalarType::Half:                                           \
        {                                                                    \
            using scalar_t = ck_tile::fp16_t;                                \
            using out_t = std::conditional_t<kForceOutAcc, acc_t, scalar_t>; \
            if ((IS_CAUSAL))                                                 \
            {                                                                \
                constexpr bool Is_causal = true;                             \
                __VA_ARGS__;                                                 \
            }                                                                \
            else                                                             \
            {                                                                \
                constexpr bool Is_causal = false;                            \
                __VA_ARGS__;                                                 \
            }                                                                \
            break;                                                           \
        }                                                                    \
        default:                                                             \
            TORCH_CHECK(false, NAME " does't support ",                      \
                        toString((TYPE)), ".");                              \
    }

int num_splits_heuristic(int batch_nhead_mblocks, int num_SMs, int num_n_blocks, int max_splits)
{
    int32_t result = 1;

    if (batch_nhead_mblocks < 0.8f * num_SMs)
    {
        max_splits = std::min(max_splits, std::min(num_SMs, num_n_blocks));
        float max_efficiency = 0.f;
        std::vector<float> efficiency;
        efficiency.reserve(max_splits);

        // Some splits are not eligible. For example, if we have 64 blocks and choose 11 splits,
        // we'll have 6 * 10 + 4 blocks. If we choose 12 splits, we'll have 6 * 11 + (-2) blocks
        // (i.e. it's 11 splits anyway).
        // So we check if the number of blocks per split is the same as the previous num_splits.
        auto is_split_eligible = [&num_n_blocks](int num_splits) {
            return (num_splits == 1) ||
                (ck_tile::integer_divide_ceil(num_n_blocks, num_splits) !=
                 ck_tile::integer_divide_ceil(num_n_blocks, num_splits - 1));
        };

        for(int num_splits = 1; num_splits <= max_splits; num_splits++)
        {
            if(!is_split_eligible(num_splits))
            {
                efficiency.push_back(0.f);
            }
            else
            {
                float n_waves = float(batch_nhead_mblocks * num_splits) / num_SMs;
                float eff     = n_waves / ceil(n_waves);
                if(eff > max_efficiency)
                {
                    max_efficiency = eff;
                }
                efficiency.push_back(eff);
            }
        }

        for(int num_splits = 1; num_splits <= max_splits; num_splits++)
        {
            if(!is_split_eligible(num_splits))
            {
                continue;
            }

            if(efficiency[num_splits - 1] >= 0.85 * max_efficiency)
            {
                result = num_splits;
                break;
            }
        }
    }

    return result;
}

template <typename Traits>
int32_t calculate_num_splits(
    const int32_t size_b,
    const int32_t size_h,
    const int32_t size_s)
{
    hipDevice_t dev;
    hipDeviceProp_t dev_prop;
    ck_tile::hip_check_error(hipGetDevice(&dev));
    ck_tile::hip_check_error(hipGetDeviceProperties(&dev_prop, dev));
    const int32_t cu_count = dev_prop.multiProcessorCount;

    const int32_t num_m_blocks = ck_tile::integer_divide_ceil(size_s, Traits::kBlockM);
    const int32_t num_n_blocks = ck_tile::integer_divide_ceil(Traits::kSizeDV, Traits::kBlockN1);

    return num_splits_heuristic(size_b * size_h * num_m_blocks, cu_count * Traits::kCuReuse, num_n_blocks, 128);
}

std::vector<torch::Tensor> flash_mla_fwd_prefill_with_kvcache_impl(
    torch::Tensor&       query,
    const torch::Tensor& key_cache,
    const torch::Tensor& value_cache,
    const int32_t        head_size_v,
    const torch::Tensor& cache_seqlens,
    const torch::Tensor& block_table,
    const float          softmax_scale,
    const bool           is_causal)
{
    constexpr bool kKVLoadOnce         = true;
    constexpr XqaStrategy kXqaStrategy = XqaStrategy::Internal;
    //TODO:
    // cases need maintenance:
    //     warp4 + load_once=false + occ=2
    //     warp8 + load_once=true  + occ=1
    // targe case:
    // warp4 + load_once=true + occ=1
    //                             dqk  dv   m0  n0  n1   #warp  wave_occu
    using Traits = std::conditional_t<kKVLoadOnce,
        FlashMlaPrefillKernelTrait<576, 512, 64, 16, 512, 8,     1,   kKVLoadOnce, kXqaStrategy>,
        FlashMlaPrefillKernelTrait<576, 512, 64, 64, 256, 4,     2,   kKVLoadOnce, kXqaStrategy>>;
    constexpr bool kForceOutAcc = false;
    using acc_t                 = float;

    torch::Tensor vcache = value_cache.data_ptr() ? value_cache : key_cache;

    auto opts = query.options();
    static_assert(std::is_same_v<acc_t, float>);
    auto opts_acc = opts.dtype(torch::kFloat32);

    const int32_t batch_size      = query.size(0);
    const int32_t seqlen_q_ori    = query.size(1);
    const int32_t num_heads_q_ori = query.size(2);
    int32_t seqlen_q              = seqlen_q_ori;
    int32_t num_heads_q           = num_heads_q_ori;

    const int32_t head_size = query.size(3);
    TORCH_CHECK((head_size == 576) && (head_size_v == 512), "Only support QK head dim 576 and V head dim 512!");

    const int32_t num_blocks      = key_cache.size(0);
    const int32_t page_block_size = key_cache.size(1);
    const int32_t num_heads_k     = key_cache.size(2);

    TORCH_CHECK(num_heads_q % num_heads_k == 0,
                "Number of heads in key/value must divide number of heads in query");

    const int32_t hq_hk_ratio_ori = num_heads_q_ori / num_heads_k;
    int32_t hq_hk_ratio = hq_hk_ratio_ori;
    int32_t mask_y_ratio      = 1;

    if constexpr(Traits::kXqaStrategy != XqaStrategy::Disable)
    {
        seqlen_q     = seqlen_q_ori * hq_hk_ratio_ori;
        num_heads_q  = num_heads_k;
        mask_y_ratio = hq_hk_ratio_ori;
        if constexpr(Traits::kXqaStrategy == XqaStrategy::External) {
            hq_hk_ratio = 1;
            if(num_heads_k == 1)
            {
                query = query.reshape({batch_size, seqlen_q, num_heads_q, head_size});
            }
            else
            {
                query = query.view({batch_size, seqlen_q_ori, num_heads_q, hq_hk_ratio_ori, head_size})
                            .transpose(2, 3)
                            .reshape({batch_size, seqlen_q, num_heads_q, head_size});
            }
        }
    }

    const int32_t num_splits = calculate_num_splits<Traits>(batch_size, num_heads_q, seqlen_q);
    const bool    do_splits = num_splits > 1;

    int32_t seqlen_q_tr = Traits::kXqaStrategy == XqaStrategy::Internal ? seqlen_q_ori : seqlen_q;
    int32_t num_heads_q_tr = Traits::kXqaStrategy == XqaStrategy::Internal ? num_heads_q_ori : num_heads_q;
    // Combine shader, which only exists when num_splits > 1, will conduct type convert by default and force.
    // Thus, kForceOutAcc doesn't work in this case.
    auto output = torch::empty({batch_size, seqlen_q_tr, num_heads_q_tr, head_size_v},
                               (kForceOutAcc && !do_splits) ? opts_acc : opts);
    auto softmax_lse = torch::empty({batch_size, num_heads_q_tr, seqlen_q_tr}, opts_acc);

    FlashMlaPrefillFwdParams params = {};

    params.num_splits    = num_splits;
    params.p_seqlens_k   = cache_seqlens.data_ptr<int32_t>();
    params.p_block_table = block_table.data_ptr<int32_t>();

    params.p_query            = query.data_ptr();
    params.p_key              = key_cache.data_ptr();
    params.p_value            = vcache.data_ptr();
    params.p_output           = output.data_ptr();
    params.p_softmax_lse      = softmax_lse.data_ptr();

    params.size_b                   = batch_size;
    params.size_s_pk                = seqlen_q;
    params.size_s_ori               = seqlen_q_ori;
    params.size_s_tr                = seqlen_q_tr;
    params.size_h_pk                = num_heads_q;
    params.size_h_ori               = num_heads_q_ori;
    params.size_h_tr                = num_heads_q_tr;
    params.hq_hk_ratio              = hq_hk_ratio;
    params.block_table_batch_stride = block_table.stride(0);
    params.num_page_blocks          = num_blocks;
    params.page_block_size          = page_block_size;
    params.scale_softmax            = softmax_scale;

    params.mask_y_ratio_mdiv = ck_tile::mdiv{static_cast<uint32_t>(mask_y_ratio)};

    params.stride_b_q   = query.stride(0);
    params.stride_s_q   = query.stride(1);
    params.stride_h_q   = query.stride(2);
    params.stride_b_k   = key_cache.stride(0);
    params.stride_s_k   = key_cache.stride(1); // size_hk * size_d
    params.stride_h_k   = key_cache.stride(2);
    params.stride_b_v   = vcache.stride(0);
    params.stride_s_v   = vcache.stride(1); // size_hk * size_d
    params.stride_h_v   = vcache.stride(2);
    params.stride_b_o   = output.stride(0);
    params.stride_s_o   = output.stride(1);
    params.stride_h_o   = output.stride(2);
    params.stride_h_lse = softmax_lse.stride(1);

    if(num_splits > 1)
    {
        auto output_accum =
            torch::empty({batch_size, num_splits, seqlen_q_tr, num_heads_q_tr, head_size_v}, opts_acc);
        auto softmax_lseaccum =
            torch::empty({batch_size, num_splits, num_heads_q_tr, seqlen_q_tr}, opts_acc);

        params.p_softmax_lseaccum = softmax_lseaccum.data_ptr();
        params.p_output_accum     = output_accum.data_ptr();
        params.stride_b_oacc      = output_accum.stride(0);
        params.stride_h_oacc      = output_accum.stride(3);
        params.stride_sp_oacc     = output_accum.stride(1);
        params.stride_s_oacc      = output_accum.stride(2);
        params.stride_b_lseacc    = softmax_lseaccum.stride(0);
        params.stride_h_lseacc    = softmax_lseaccum.stride(2);
        params.stride_sp_lseacc   = softmax_lseaccum.stride(1);
    }

    DISPATCH_FMLA_TYPES(
        query.scalar_type(),
        is_causal,
        "fmla_fwd",
        [&](){
            dispatch_fmla_fwd_splictkv_prefill<Traits, scalar_t, acc_t, out_t, Is_causal>(params);
        }();
    );
    // assert(is_causal == false);
    // assert(query.scalar_type() == at::ScalarType::BFloat16);
    // using scalar_t = ck_tile::bf16_t;
    // using out_t = std::conditional_t<kForceOutAcc, acc_t, scalar_t>;
    // dispatch_fmla_fwd_splictkv_prefill<Traits, scalar_t, acc_t, out_t, false>(params);

    if constexpr(Traits::kXqaStrategy == XqaStrategy::External)
    {
        // post process for out and softmax_lse
        if(num_heads_k == 1)
        {
            output = output.reshape({batch_size, seqlen_q_ori, num_heads_q_ori, head_size_v});
        }
        else
        {
            output = output.view({batch_size, seqlen_q_ori, hq_hk_ratio_ori, num_heads_q, head_size_v})
                         .transpose(2, 3)
                         .reshape({batch_size, seqlen_q_ori, num_heads_q_ori, head_size_v});
        }
        softmax_lse = softmax_lse.view({batch_size, num_heads_q, seqlen_q_ori, hq_hk_ratio_ori})
                          .transpose(2, 3)
                          .reshape({batch_size, num_heads_q_ori, seqlen_q_ori});
    }

    return {output.to(opts), softmax_lse};
}
