#include "hip/hip_runtime.h"
// SPDX-License-Identifier: MIT
// Copyright (C) 2024-2025, Advanced Micro Devices, Inc. All rights reserved.
#include <ATen/hip/HIPContext.h>
#include <ATen/hip/impl/HIPGuardImplMasqueradingAsCUDA.h>
#include <torch/all.h>

#include "dispatch_utils.h"
#include <hipcub/hipcub.hpp>
#include <hipcub/util_type.hpp>

namespace aiter {

static inline __device__ uint16_t extractBinIdx(float x)
{
    union
    {
        __half h;
        uint16_t u16;
    } tmp;
    tmp.h   = __float2half_rn(x);
    tmp.u16 = (x < 0.f) ? (~tmp.u16 & 0xffff) : (tmp.u16 | 0x8000);
    return 511 - (tmp.u16 >> 7);
}

template <int kNumThreadsPerBlock = 512, int kNumBins = 512, int kTopK = 2048>
__device__ void topk_per_row_kernel(const float* logits,
                                    const int rowStart,
                                    const int rowEnd,
                                    const int rowIdx,
                                    int* outIndices,
                                    int stride0,
                                    int stride1)
{
    // The number of elements per thread for the final top-k sort.
    static constexpr int kNumTopKItemsPerThread = kTopK / kNumThreadsPerBlock;
    // The class to sort the elements during the final top-k sort.
    using TopKSort =
        hipcub::BlockRadixSort<float, kNumThreadsPerBlock, kNumTopKItemsPerThread, int>;

    // The number of slots for the final pass.
    static constexpr int kNumFinalItems = 3072;
    // The number of elements per thread for the final sort.
    static constexpr int kNumFinalItemsPerThread = kNumFinalItems / kNumThreadsPerBlock;
    // The class to sort the elements during the final pass.
    using FinalSort =
        hipcub::BlockRadixSort<float, kNumThreadsPerBlock, kNumFinalItemsPerThread, int>;

    // The class to compute the inclusive prefix-sum over the histogram.
    using Scan = hipcub::BlockScan<int, kNumThreadsPerBlock>;

    // Shared memory to compute the block scan.
    __shared__ typename Scan::TempStorage smemScan;

    // The structure to store the final items (for the final pass).
    struct FinalItems
    {
        // Shared memory to store the indices for the final pass.
        int indices[kNumFinalItems];
        // Shared memory to store the logits for the final pass.
        float logits[kNumFinalItems];
    };

    // Shared memory to compute the block sort.
    __shared__ union
    {
        FinalItems items;
        typename FinalSort::TempStorage finalSort;
        typename TopKSort::TempStorage topKSort;
    } smemFinal;

    // Shared memory to store the histogram.
    __shared__ int smemHistogram[kNumBins];
    // Shared memory to store the selected indices.
    __shared__ int smemIndices[kTopK];
    // Shared memory to store the threshold bin.
    __shared__ int smemThresholdBinIdx[1];
    // Shared memory counter to register the candidates for the final phase.
    __shared__ int smemFinalDstIdx[1];

    // The length of the row.
    int rowLen = rowEnd - rowStart;

    // Shortcut if the length of the row is smaller than Top-K. Indices are not
    // sorted by their corresponding logit.
    if(rowLen <= kTopK)
    {
        for(int rowIt = threadIdx.x; rowIt < rowLen; rowIt += kNumThreadsPerBlock)
        {
            int idx                            = rowStart + rowIt;
            outIndices[rowIdx * kTopK + rowIt] = idx - rowStart;
        }
        for(int rowIt = rowLen + threadIdx.x; rowIt < kTopK; rowIt += kNumThreadsPerBlock)
        {
            outIndices[rowIdx * kTopK + rowIt] = -1;
        }
        return;
    }

    // Clear the histogram.
    if(threadIdx.x < kNumBins)
    {
        smemHistogram[threadIdx.x] = 0;
    }

    // Make sure the histogram is ready.
    __syncthreads();

    // Fetch elements one-by-one.
    for(int rowIt = rowStart + threadIdx.x; rowIt < rowEnd; rowIt += kNumThreadsPerBlock)
    {
        int64_t offset = ((int64_t)rowIdx) * stride0 + ((int64_t)rowIt) * stride1;
        uint16_t idx   = extractBinIdx(logits[offset]);
        atomicAdd(&smemHistogram[idx], 1);
    }

    // Make sure the histogram is ready.
    __syncthreads();

    // Read the values from SMEM.
    int binCount{0};
    if(threadIdx.x < kNumBins)
    {
        binCount = smemHistogram[threadIdx.x];
    }

    // Make sure each thread has read its value.
    __syncthreads();

    // Compute the prefix sum.
    int prefixSum{0}, totalSum{0};
    Scan(smemScan).ExclusiveSum(binCount, prefixSum, totalSum);

    // Update the histogram with the prefix sums.
    if(threadIdx.x < kNumBins)
    {
        smemHistogram[threadIdx.x] = prefixSum;
    }

    // Make sure the data is in shared memory.
    __syncthreads();

    // Find the last valid bin.
    if(threadIdx.x < kNumBins)
    {
        int nextPrefixSum = threadIdx.x == kNumBins - 1 ? totalSum : smemHistogram[threadIdx.x + 1];
        if(prefixSum < kTopK && nextPrefixSum >= kTopK)
        {
            smemThresholdBinIdx[0] = threadIdx.x;
        }
    }

    // Clear the counter to store the items for the final phase.
    if(threadIdx.x == 0)
    {
        smemFinalDstIdx[0] = 0;
    }

    // Make sure the data is in shared memory.
    __syncthreads();

    // The threshold bin.
    int thresholdBinIdx = smemThresholdBinIdx[0];

    // Fetch elements one-by-one and populate the shared memory buffers.
    for(int rowIt = rowStart + threadIdx.x; rowIt < rowEnd; rowIt += kNumThreadsPerBlock)
    {
        int64_t offset = ((int64_t)rowIdx) * stride0 + ((int64_t)rowIt) * stride1;
        float logit    = logits[offset];
        uint16_t idx   = extractBinIdx(logit);
        if(idx < thresholdBinIdx)
        {
            int dstIdx          = atomicAdd(&smemHistogram[idx], 1);
            smemIndices[dstIdx] = rowIt;
        }
        else if(idx == thresholdBinIdx)
        {
            int dstIdx = atomicAdd(&smemFinalDstIdx[0], 1);
            if(dstIdx < kNumFinalItems)
            {
                smemFinal.items.logits[dstIdx]  = logit;
                smemFinal.items.indices[dstIdx] = rowIt;
            }
        }
    }

    // Make sure the elements are in shared memory.
    __syncthreads();

    // The logits of the elements to be sorted in the final pass.
    float finalLogits[kNumFinalItemsPerThread];
    // The indices of the elements to be sorted in the final pass.
    int finalIndices[kNumFinalItemsPerThread];

// Init.
#pragma unroll
    for(int ii = 0; ii < kNumFinalItemsPerThread; ++ii)
    {
        finalLogits[ii] = -FLT_MAX;
    }

// Read the elements from SMEM.
#pragma unroll
    for(int ii = 0; ii < kNumFinalItemsPerThread; ++ii)
    {
        int srcIdx = ii * kNumThreadsPerBlock + threadIdx.x;
        if(srcIdx < smemFinalDstIdx[0])
        {
            finalLogits[ii]  = smemFinal.items.logits[srcIdx];
            finalIndices[ii] = smemFinal.items.indices[srcIdx];
        }
    }

    // Make sure the shared memory has been read.
    __syncthreads();

    // Sort the elements.
    FinalSort(smemFinal.finalSort).SortDescendingBlockedToStriped(finalLogits, finalIndices);

    // Copy the data back to the shared memory storage.
    int baseIdx = thresholdBinIdx > 0 ? smemHistogram[thresholdBinIdx - 1] : 0;
#pragma unroll
    for(int ii = 0; ii < kNumFinalItemsPerThread; ++ii)
    {
        int srcIdx = ii * kNumThreadsPerBlock + threadIdx.x;
        int dstIdx = baseIdx + srcIdx;
        if(dstIdx < kTopK)
        {
            smemIndices[dstIdx] = finalIndices[ii];
        }
    }

    // Make sure the data is in shared memory.
    __syncthreads();

// Store to global memory.
#pragma unroll
    for(int ii = 0; ii < kNumTopKItemsPerThread; ++ii)
    {
        int offset         = rowIdx * kTopK + ii * kNumThreadsPerBlock + threadIdx.x;
        outIndices[offset] = smemIndices[ii * kNumThreadsPerBlock + threadIdx.x] - rowStart;
    }
}

template <int kNumThreadsPerBlock = 512>
static __global__ void topk_per_row(const float* logits,
                                    const int* rowStarts,
                                    const int* rowEnds,
                                    int* outIndices,
                                    int stride0,
                                    int stride1)
{
    // The number of bins in the histogram.
    static constexpr int kNumBins = 512;

    // The top-k width.
    static constexpr int kTopK = 2048;

    // The row computed by this block.
    int rowIdx = blockIdx.x;

    // The range of logits within the row.
    int rowStart = rowStarts[rowIdx];
    int rowEnd   = rowEnds[rowIdx];

    topk_per_row_kernel<kNumThreadsPerBlock, kNumBins, kTopK>(
        logits, rowStart, rowEnd, rowIdx, outIndices, stride0, stride1);
}

template <int kNumThreadsPerBlock = 512>
static __global__ void topk_per_row_decode(
    const float* logits, const int* seqLens, int* outIndices, int stride0, int stride1, int next_n)
{
    // The number of bins in the histogram.
    static constexpr int kNumBins = 512;

    // The top-k width.
    static constexpr int kTopK = 2048;

    // The row computed by this block.
    int rowIdx = blockIdx.x;

    // The range of logits within the row.
    int rowStart = 0;
    int seq_len  = seqLens[rowIdx / next_n];
    int rowEnd   = seq_len - next_n + (rowIdx % next_n) + 1;

    topk_per_row_kernel<kNumThreadsPerBlock, kNumBins, kTopK>(
        logits, rowStart, rowEnd, rowIdx, outIndices, stride0, stride1);
}

} // namespace aiter

void topk_per_row(const torch::Tensor& logits,
                  const torch::Tensor& rowStarts,
                  const torch::Tensor& rowEnds,
                  torch::Tensor& indices,
                  int64_t numRows,
                  int64_t stride0,
                  int64_t stride1)
{
    // Compute the results on the device.
    constexpr int kNumThreadsPerBlock = 512;

    // The top-k width.
    static constexpr int kTopK = 2048;

    const hipStream_t stream = at::hip::getCurrentHIPStream();

    aiter::topk_per_row<kNumThreadsPerBlock>
        <<<numRows, kNumThreadsPerBlock, 0, stream>>>(logits.data_ptr<float>(),
                                                      rowStarts.data_ptr<int>(),
                                                      rowEnds.data_ptr<int>(),
                                                      indices.data_ptr<int>(),
                                                      static_cast<int>(stride0),
                                                      static_cast<int>(stride1));
}

void topk_per_row_decode(const torch::Tensor& logits,
                         int64_t next_n,
                         const torch::Tensor& seqLens,
                         torch::Tensor& indices,
                         int64_t numRows,
                         int64_t stride0,
                         int64_t stride1)
{
    // Compute the results on the device.
    constexpr int kNumThreadsPerBlock = 512;
    const hipStream_t stream          = at::hip::getCurrentHIPStream();

    aiter::topk_per_row_decode<kNumThreadsPerBlock>
        <<<numRows, kNumThreadsPerBlock, 0, stream>>>(logits.data_ptr<float>(),
                                                      seqLens.data_ptr<int>(),
                                                      indices.data_ptr<int>(),
                                                      static_cast<int>(stride0),
                                                      static_cast<int>(stride1),
                                                      static_cast<int>(next_n));
}
