#include "hip/hip_runtime.h"
// SPDX-License-Identifier: MIT
// Copyright (C) 2024-2025, Advanced Micro Devices, Inc. All rights reserved.
#include <ATen/hip/HIPContext.h>
#include <ATen/hip/impl/HIPGuardImplMasqueradingAsCUDA.h>
#include <torch/all.h>

#include "dispatch_utils.h"
#include <hipcub/hipcub.hpp>
#include <hipcub/util_type.hpp>

namespace aiter {

static inline __device__ uint16_t extractBinIdx(float x)
{
    union
    {
        __half h;
        uint16_t u16;
    } tmp;
    tmp.h   = __float2half_rn(x);
    tmp.u16 = (x < 0.f) ? (~tmp.u16 & 0xffff) : (tmp.u16 | 0x8000);
    return 511 - (tmp.u16 >> 7);
}

using fp32x1 = __attribute__((__ext_vector_type__(1))) float;
using fp32x2 = __attribute__((__ext_vector_type__(2))) float;
using fp32x4 = __attribute__((__ext_vector_type__(4))) float;

template <int vec>
struct to_vector;

template <>
struct to_vector<1>
{
    using type = fp32x1;
};

template <>
struct to_vector<2>
{
    using type = fp32x2;
};

template <>
struct to_vector<4>
{
    using type = fp32x4;
};

template <int kNumThreadsPerBlock = 512, int kNumBins = 512, int kTopK = 2048, int Vector = 4>
__device__ void topk_per_row_kernel(const float* logits,
                                    const int rowStart,
                                    const int rowEnd,
                                    const int rowIdx,
                                    int* outIndices,
                                    int stride0,
                                    int stride1)
{
    // The number of elements per thread for the final top-k sort.
    static constexpr int kNumTopKItemsPerThread = kTopK / kNumThreadsPerBlock;
    // The class to sort the elements during the final top-k sort.
    using TopKSort =
        hipcub::BlockRadixSort<float, kNumThreadsPerBlock, kNumTopKItemsPerThread, int>;

    // The number of slots for the final pass.
    static constexpr int kNumFinalItems = 3072;
    // The number of elements per thread for the final sort.
    static constexpr int kNumFinalItemsPerThread = kNumFinalItems / kNumThreadsPerBlock;
    // The class to sort the elements during the final pass.
    using FinalSort =
        hipcub::BlockRadixSort<float, kNumThreadsPerBlock, kNumFinalItemsPerThread, int>;

    // The class to compute the inclusive prefix-sum over the histogram.
    using Scan = hipcub::BlockScan<int, kNumThreadsPerBlock>;

    using VectorType = typename to_vector<Vector>::type;

    // Shared memory to compute the block scan.
    __shared__ typename Scan::TempStorage smemScan;

    // The structure to store the final items (for the final pass).
    struct FinalItems
    {
        // Shared memory to store the indices for the final pass.
        int indices[kNumFinalItems];
        // Shared memory to store the logits for the final pass.
        float logits[kNumFinalItems];
    };

    // Shared memory to compute the block sort.
    __shared__ union
    {
        FinalItems items;
        typename FinalSort::TempStorage finalSort;
        typename TopKSort::TempStorage topKSort;
    } smemFinal;

    // Shared memory to store the histogram.
    __shared__ int smemHistogram[kNumBins];
    // Shared memory to store the selected indices.
    __shared__ int smemIndices[kTopK];
    // Shared memory to store the threshold bin.
    __shared__ int smemThresholdBinIdx[1];
    // Shared memory counter to register the candidates for the final phase.
    __shared__ int smemFinalDstIdx[1];

    // The length of the row.
    int rowLen = rowEnd - rowStart;

    // Shortcut if the length of the row is smaller than Top-K. Indices are not
    // sorted by their corresponding logit.
    if(rowLen <= kTopK)
    {
        for(int rowIt = threadIdx.x; rowIt < rowLen; rowIt += kNumThreadsPerBlock)
        {
            int idx                            = rowStart + rowIt;
            outIndices[rowIdx * kTopK + rowIt] = idx - rowStart;
        }
        for(int rowIt = rowLen + threadIdx.x; rowIt < kTopK; rowIt += kNumThreadsPerBlock)
        {
            outIndices[rowIdx * kTopK + rowIt] = -1;
        }
        return;
    }

    // Clear the histogram.
    if(threadIdx.x < kNumBins)
    {
        smemHistogram[threadIdx.x] = 0;
    }

    // Make sure the histogram is ready.
    __syncthreads();

    // Fetch elements one-by-one.
    for(int rowIt = rowStart + threadIdx.x; rowIt < (rowEnd + Vector - 1) / Vector;
        rowIt += kNumThreadsPerBlock)
    {
        int64_t offset = ((int64_t)rowIdx) * (stride0 / Vector) + ((int64_t)rowIt) * stride1;
        auto v         = reinterpret_cast<const VectorType*>(logits)[offset];

#pragma unroll
        for(int j = 0; j < Vector; j++)
        {
            float logit  = (rowIt * Vector + j) < rowEnd ? v[j] : -INFINITY;
            uint16_t idx = extractBinIdx(logit);
            atomicAdd(&smemHistogram[idx], 1);
        }
    }

    // Make sure the histogram is ready.
    __syncthreads();

    // Read the values from SMEM.
    int binCount{0};
    if(threadIdx.x < kNumBins)
    {
        binCount = smemHistogram[threadIdx.x];
    }

    // Make sure each thread has read its value.
    __syncthreads();

    // Compute the prefix sum.
    int prefixSum{0}, totalSum{0};
    Scan(smemScan).ExclusiveSum(binCount, prefixSum, totalSum);

    // Update the histogram with the prefix sums.
    if(threadIdx.x < kNumBins)
    {
        smemHistogram[threadIdx.x] = prefixSum;
    }

    // Make sure the data is in shared memory.
    __syncthreads();

    // Find the last valid bin.
    if(threadIdx.x < kNumBins)
    {
        int nextPrefixSum = threadIdx.x == kNumBins - 1 ? totalSum : smemHistogram[threadIdx.x + 1];
        if(prefixSum < kTopK && nextPrefixSum >= kTopK)
        {
            smemThresholdBinIdx[0] = threadIdx.x;
        }
    }

    // Clear the counter to store the items for the final phase.
    if(threadIdx.x == 0)
    {
        smemFinalDstIdx[0] = 0;
    }

    // Make sure the data is in shared memory.
    __syncthreads();

    // The threshold bin.
    int thresholdBinIdx = smemThresholdBinIdx[0];

    // Fetch elements one-by-one and populate the shared memory buffers.
    for(int rowIt = rowStart + threadIdx.x; rowIt < (rowEnd + Vector - 1) / Vector;
        rowIt += kNumThreadsPerBlock)
    {
        int64_t offset = ((int64_t)rowIdx) * stride0 / Vector + ((int64_t)rowIt) * stride1;
        auto v         = reinterpret_cast<const VectorType*>(logits)[offset];

#pragma unroll
        for(auto j = 0; j < Vector; j++)
        {
            float logit = (rowIt * Vector + j) < rowEnd ? v[j] : -INFINITY;
            // float logit    = v[j];
            uint16_t idx = extractBinIdx(logit);
            if(idx < thresholdBinIdx)
            {
                int dstIdx          = atomicAdd(&smemHistogram[idx], 1);
                smemIndices[dstIdx] = Vector * rowIt + j;
            }
            else if(idx == thresholdBinIdx)
            {
                int dstIdx = atomicAdd(&smemFinalDstIdx[0], 1);
                if(dstIdx < kNumFinalItems)
                {
                    smemFinal.items.logits[dstIdx]  = logit;
                    smemFinal.items.indices[dstIdx] = Vector * rowIt + j;
                }
            }
        }
    }

    // Make sure the elements are in shared memory.
    // __syncthreads();

    // The logits of the elements to be sorted in the final pass.
    float finalLogits[kNumFinalItemsPerThread];
    // The indices of the elements to be sorted in the final pass.
    int finalIndices[kNumFinalItemsPerThread];

// Init.
#pragma unroll
    for(int ii = 0; ii < kNumFinalItemsPerThread; ++ii)
    {
        finalLogits[ii] = -FLT_MAX;
    }

    __syncthreads();

// Read the elements from SMEM.
#pragma unroll
    for(int ii = 0; ii < kNumFinalItemsPerThread; ++ii)
    {
        int srcIdx = ii * kNumThreadsPerBlock + threadIdx.x;
        if(srcIdx < smemFinalDstIdx[0])
        {
            finalLogits[ii]  = smemFinal.items.logits[srcIdx];
            finalIndices[ii] = smemFinal.items.indices[srcIdx];
        }
    }

    // Make sure the shared memory has been read.
    __syncthreads();

    // Sort the elements.
    FinalSort(smemFinal.finalSort).SortDescendingBlockedToStriped(finalLogits, finalIndices);

    // Copy the data back to the shared memory storage.
    int baseIdx = thresholdBinIdx > 0 ? smemHistogram[thresholdBinIdx - 1] : 0;
#pragma unroll
    for(int ii = 0; ii < kNumFinalItemsPerThread; ++ii)
    {
        int srcIdx = ii * kNumThreadsPerBlock + threadIdx.x;
        int dstIdx = baseIdx + srcIdx;
        if(dstIdx < kTopK)
        {
            smemIndices[dstIdx] = finalIndices[ii];
        }
    }

    // Make sure the data is in shared memory.
    __syncthreads();

// Store to global memory.
#pragma unroll
    for(int ii = 0; ii < kNumTopKItemsPerThread; ++ii)
    {
        int offset         = rowIdx * kTopK + ii * kNumThreadsPerBlock + threadIdx.x;
        outIndices[offset] = smemIndices[ii * kNumThreadsPerBlock + threadIdx.x] - rowStart;
    }
}

template <int kNumThreadsPerBlock = 512, int Vector = 4>
static __global__ void topk_per_row(const float* logits,
                                    const int* rowStarts,
                                    const int* rowEnds,
                                    int* outIndices,
                                    int stride0,
                                    int stride1)
{
    // The number of bins in the histogram.
    static constexpr int kNumBins = 512;

    // The top-k width.
    static constexpr int kTopK = 2048;

    // The row computed by this block.
    int rowIdx = blockIdx.x;

    // The range of logits within the row.
    int rowStart = rowStarts[rowIdx];
    int rowEnd   = rowEnds[rowIdx];

    topk_per_row_kernel<kNumThreadsPerBlock, kNumBins, kTopK, Vector>(
        logits, rowStart, rowEnd, rowIdx, outIndices, stride0, stride1);
}

template <int kNumThreadsPerBlock = 512, int Vector = 4>
static __global__ void topk_per_row_decode(
    const float* logits, const int* seqLens, int* outIndices, int stride0, int stride1, int next_n)
{
    // The number of bins in the histogram.
    static constexpr int kNumBins = kNumThreadsPerBlock;

    // The top-k width.
    static constexpr int kTopK = 2048;

    // The row computed by this block.
    int rowIdx = blockIdx.x;

    // The range of logits within the row.
    int rowStart = 0;
    int seq_len  = seqLens[rowIdx / next_n];
    int rowEnd   = seq_len - next_n + (rowIdx % next_n) + 1;

    topk_per_row_kernel<kNumThreadsPerBlock, kNumBins, kTopK, Vector>(
        logits, rowStart, rowEnd, rowIdx, outIndices, stride0, stride1);
}

} // namespace aiter

void topk_per_row(const torch::Tensor& logits,
                  const torch::Tensor& rowStarts,
                  const torch::Tensor& rowEnds,
                  torch::Tensor& indices,
                  int64_t numRows,
                  int64_t stride0,
                  int64_t stride1)
{
    // Compute the results on the device.
    constexpr int kNumThreadsPerBlock = 512;

    // The top-k width.
    static constexpr int kTopK = 2048;

    const hipStream_t stream = at::hip::getCurrentHIPStream();

    if(stride0 % 4 == 0)
        aiter::topk_per_row<kNumThreadsPerBlock, 4>
            <<<numRows, kNumThreadsPerBlock, 0, stream>>>(logits.data_ptr<float>(),
                                                          rowStarts.data_ptr<int>(),
                                                          rowEnds.data_ptr<int>(),
                                                          indices.data_ptr<int>(),
                                                          static_cast<int>(stride0),
                                                          static_cast<int>(stride1));
    else
        aiter::topk_per_row<kNumThreadsPerBlock, 1>
            <<<numRows, kNumThreadsPerBlock, 0, stream>>>(logits.data_ptr<float>(),
                                                          rowStarts.data_ptr<int>(),
                                                          rowEnds.data_ptr<int>(),
                                                          indices.data_ptr<int>(),
                                                          static_cast<int>(stride0),
                                                          static_cast<int>(stride1));
}

void topk_per_row_decode(const torch::Tensor& logits,
                         int64_t next_n,
                         const torch::Tensor& seqLens,
                         torch::Tensor& indices,
                         int64_t numRows,
                         int64_t stride0,
                         int64_t stride1)
{
    // Compute the results on the device.
    constexpr int kNumThreadsPerBlock = 1024;
    const hipStream_t stream          = at::hip::getCurrentHIPStream();

    if(stride0 % 4 == 0)
        aiter::topk_per_row_decode<kNumThreadsPerBlock, 4>
            <<<numRows, kNumThreadsPerBlock, 0, stream>>>(logits.data_ptr<float>(),
                                                          seqLens.data_ptr<int>(),
                                                          indices.data_ptr<int>(),
                                                          static_cast<int>(stride0),
                                                          static_cast<int>(stride1),
                                                          static_cast<int>(next_n));
    else
        aiter::topk_per_row_decode<kNumThreadsPerBlock, 1>
            <<<numRows, kNumThreadsPerBlock, 0, stream>>>(logits.data_ptr<float>(),
                                                          seqLens.data_ptr<int>(),
                                                          indices.data_ptr<int>(),
                                                          static_cast<int>(stride0),
                                                          static_cast<int>(stride1),
                                                          static_cast<int>(next_n));
}
