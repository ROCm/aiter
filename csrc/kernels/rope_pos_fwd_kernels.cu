#include "hip/hip_runtime.h"
// SPDX-License-Identifier: MIT
// Copyright (c) 2025, Advanced Micro Devices, Inc. All rights reserved.

#include "rope_common.h"

// =====================================================================================================================
// Interfaces
//

/**
 * @brief Compute Rotational Positional Encoding on @param input. Results are written in @param output.
 *        Cosine and sine of frequency should have been calculated and specified in @param cos and @param sin.
 *        @param positions is an indirect buffer storing the index of value in @param cos and @param sin used to
 *        calculate with current input element.
 *
 * @param output       [s, b, h, d]
 * @param input        [s, b, h, d]
 * @param cos          [max_pos, 1, 1, d // 2] if @param reuse_freqs_front_part else [max_pos, 1, 1, d]
 * @param sin          [max_pos, 1, 1, d // 2] if @param reuse_freqs_front_part else [max_pos, 1, 1, d]
 * @param positions    [s, b]
 * @param rotate_style 0: NEOX style, 1: GPT-J style
 * @param nope_first   If true, back part in last dimension of input is rotated. Otherwise, the front part is rotated.
 */
void rope_cached_positions_fwd_impl(
    torch::Tensor&       output,
    const torch::Tensor& input,
    const torch::Tensor& cos,
    const torch::Tensor& sin,
    const torch::Tensor& positions,
    const int32_t        rotate_style,
    const bool           reuse_freqs_front_part,
    const bool           nope_first)
{
    // Get sizes of input and output
    const int32_t size_s = min(input.size(0), positions.size(0));
    const int32_t size_b = min(input.size(1), positions.size(1));
    const int32_t size_h = input.size(2);
    const int32_t size_d = input.size(3);
    const int32_t size_f = cos.size(3);

    // Get strides of input
    const int32_t stride_i_s = input.stride(0);
    const int32_t stride_i_b = input.stride(1);
    const int32_t stride_i_h = input.stride(2);
    const int32_t stride_i_d = input.stride(3);

    // Get strides of output
    const int32_t stride_o_s = output.stride(0);
    const int32_t stride_o_b = output.stride(1);
    const int32_t stride_o_h = output.stride(2);
    const int32_t stride_o_d = output.stride(3);

    // Get strides of positions and offsets
    assert(1 == positions.stride(1) && 2 == positions.dim());
    const int32_t max_position = cos.size(0);

    DISPATCH_ROPE_TYPES_PARAMS(
        input.scalar_type(),
        cos.scalar_type(),
        rotate_style,
        reuse_freqs_front_part,
        nope_first,
        "dispatch_1c_sbhd_cached_indirect<OpCachedFwd, ...>",
        dispatch_1c_sbhd_cached_indirect<OpCachedFwd, RotateStyle, ReuseFreqsFrontPart, NopeFirst>(
            output.data_ptr<scalar_t_0>(),
            input.data_ptr<scalar_t_0>(),
            cos.data_ptr<scalar_t_1>(),
            sin.data_ptr<scalar_t_1>(),
            positions.data_ptr<int64_t>(),
            max_position,
            size_s, size_b, size_h, size_d,
            size_f, // size of last dimension of freqs.
            stride_i_s, stride_i_b, stride_i_h, stride_i_d,
            stride_o_s, stride_o_b, stride_o_h, stride_o_d););
}

/**
 * @brief Compute Rotational Positional Encoding on 2 channels: @param input_x and @param input_y. Results are written
 *        in @param output_x and @param output_y respectively.
 *        Cosine and sine of frequency should have been calculated and specified in @param cos and @param sin.
 *        @param positions is an indirect buffer storing the index of value in @param cos and @param sin used to
 *        calculate with current input element.
 *
 * @param output_x     [s, b, h, d]
 * @param output_y     [s, b, h, d]
 * @param input_x      [s, b, h, d]
 * @param input_y      [s, b, h, d]
 * @param cos          [max_pos, 1, 1, d // 2] if @param reuse_freqs_front_part else [max_pos, 1, 1, d]
 * @param sin          [max_pos, 1, 1, d // 2] if @param reuse_freqs_front_part else [max_pos, 1, 1, d]
 * @param positions    [s, b]
 * @param rotate_style 0: NEOX style, 1: GPT-J style
 * @param nope_first   If true, back part in last dimension of input is rotated. Otherwise, the front part is rotated.
 */
void rope_cached_positions_2c_fwd_impl(
    torch::Tensor&       output_x,
    torch::Tensor&       output_y,
    const torch::Tensor& input_x,
    const torch::Tensor& input_y,
    const torch::Tensor& cos,
    const torch::Tensor& sin,
    const torch::Tensor& positions,
    const int32_t        rotate_style,
    const bool           reuse_freqs_front_part,
    const bool           nope_first)
{
    // Get sizes of input and output
    const int32_t size_s   = min(input_x.size(0), positions.size(0));
    const int32_t size_b   = min(input_x.size(1), positions.size(1));
    const int32_t size_h_x = input_x.size(2);
    const int32_t size_h_y = input_y.size(2);
    const int32_t size_d   = input_x.size(3);
    const int32_t size_f   = cos.size(3);

    // Get strides of input
    const int32_t stride_ix_s = input_x.stride(0);
    const int32_t stride_ix_b = input_x.stride(1);
    const int32_t stride_ix_h = input_x.stride(2);
    const int32_t stride_ix_d = input_x.stride(3);
    const int32_t stride_iy_s = input_y.stride(0);
    const int32_t stride_iy_b = input_y.stride(1);
    const int32_t stride_iy_h = input_y.stride(2);
    const int32_t stride_iy_d = input_y.stride(3);

    // Get strides of output
    const int32_t stride_ox_s = output_x.stride(0);
    const int32_t stride_ox_b = output_x.stride(1);
    const int32_t stride_ox_h = output_x.stride(2);
    const int32_t stride_ox_d = output_x.stride(3);
    const int32_t stride_oy_s = output_y.stride(0);
    const int32_t stride_oy_b = output_y.stride(1);
    const int32_t stride_oy_h = output_y.stride(2);
    const int32_t stride_oy_d = output_y.stride(3);

    // Get strides of positions and offsets
    assert(1 == positions.stride(1) && 2 == positions.dim());
    const int32_t max_position = cos.size(0);

    DISPATCH_ROPE_TYPES_PARAMS(
        input_x.scalar_type(),
        cos.scalar_type(),
        rotate_style,
        reuse_freqs_front_part,
        nope_first,
        "dispatch_2c_sbhd_cached_indirect<OpCachedFwd, ...>",
        dispatch_2c_sbhd_cached_indirect<OpCachedFwd, RotateStyle, ReuseFreqsFrontPart, NopeFirst>(
            output_x.data_ptr<scalar_t_0>(),
            output_y.data_ptr<scalar_t_0>(),
            input_x.data_ptr<scalar_t_0>(),
            input_y.data_ptr<scalar_t_0>(),
            cos.data_ptr<scalar_t_1>(),
            sin.data_ptr<scalar_t_1>(),
            positions.data_ptr<int64_t>(),
            max_position,
            size_s, size_b, size_h_x, size_h_y, size_d,
            size_f, // size of last dimension of freqs.
            stride_ix_s, stride_ix_b, stride_ix_h, stride_ix_d,
            stride_iy_s, stride_iy_b, stride_iy_h, stride_iy_d,
            stride_ox_s, stride_ox_b, stride_ox_h, stride_ox_d,
            stride_oy_s, stride_oy_b, stride_oy_h, stride_oy_d););
}

/**
 * @brief Compute Rotational Positional Encoding on @param input. Results are written in @param output.
 *        Cosine and sine of frequency should have been calculated and specified in @param cos and @param sin.
 *        @param positions and @param offsets are indirect buffers storing the index of value in @param cos and
 *        @param sin used to calculate with current input element. The corresponding values in @param positions and
 *        @param offsets are added together to get the final index.
 *
 * @param output       [s, b, h, d]
 * @param input        [s, b, h, d]
 * @param cos          [max_pos, 1, 1, d // 2] if @param reuse_freqs_front_part else [max_pos, 1, 1, d]
 * @param sin          [max_pos, 1, 1, d // 2] if @param reuse_freqs_front_part else [max_pos, 1, 1, d]
 * @param positions    [s, b]
 * @param offsets      [s, b]
 * @param rotate_style 0: NEOX style, 1: GPT-J style
 * @param nope_first   If true, back part in last dimension of input is rotated. Otherwise, the front part is rotated.
 */
void rope_cached_positions_offsets_fwd_impl(
    torch::Tensor&       output,
    const torch::Tensor& input,
    const torch::Tensor& cos,
    const torch::Tensor& sin,
    const torch::Tensor& positions,
    const torch::Tensor& offsets,
    const int32_t        rotate_style,
    const bool           reuse_freqs_front_part,
    const bool           nope_first)
{
    // Get sizes of input and output
    const int32_t size_s = min(min(input.size(0), positions.size(0)), offsets.size(0));
    const int32_t size_b = min(min(input.size(1), positions.size(1)), offsets.size(1));
    const int32_t size_h = input.size(2);
    const int32_t size_d = input.size(3);
    const int32_t size_f = cos.size(3);

    // Get strides of input
    const int32_t stride_i_s = input.stride(0);
    const int32_t stride_i_b = input.stride(1);
    const int32_t stride_i_h = input.stride(2);
    const int32_t stride_i_d = input.stride(3);

    // Get strides of output
    const int32_t stride_o_s = output.stride(0);
    const int32_t stride_o_b = output.stride(1);
    const int32_t stride_o_h = output.stride(2);
    const int32_t stride_o_d = output.stride(3);

    // Get strides of positions and offsets
    assert(1 == positions.stride(1) && 2 == positions.dim());
    assert(1 == offsets.stride(1)   && 2 == offsets.dim());
    const int32_t max_position = cos.size(0);

    DISPATCH_ROPE_TYPES_PARAMS(
        input.scalar_type(),
        cos.scalar_type(),
        rotate_style,
        reuse_freqs_front_part,
        nope_first,
        "dispatch_1c_sbhd_cached_indirect2<OpCachedFwd, ...>",
        dispatch_1c_sbhd_cached_indirect2<OpCachedFwd, RotateStyle, ReuseFreqsFrontPart, NopeFirst>(
            output.data_ptr<scalar_t_0>(),
            input.data_ptr<scalar_t_0>(),
            cos.data_ptr<scalar_t_1>(),
            sin.data_ptr<scalar_t_1>(),
            positions.data_ptr<int64_t>(),
            offsets.data_ptr<int64_t>(),
            max_position,
            size_s, size_b, size_h, size_d,
            size_f, // size of last dimension of freqs.
            stride_i_s, stride_i_b, stride_i_h, stride_i_d,
            stride_o_s, stride_o_b, stride_o_h, stride_o_d););
}

/**
 * @brief Compute Rotational Positional Encoding on 2 channels: @param input_x and @param input_y. Results are written
 *        in @param output_x and @param output_y respectively.
 *        Cosine and sine of frequency should have been calculated and specified in @param cos and @param sin.
 *        @param positions and @param offsets are indirect buffers storing the index of value in @param cos and
 *        @param sin used to calculate with current input element. The corresponding values in @param positions and
 *        @param offsets are added together to get the final index.
 *
 * @param output_x     [s, b, h, d]
 * @param output_y     [s, b, h, d]
 * @param input_x      [s, b, h, d]
 * @param input_y      [s, b, h, d]
 * @param cos          [max_pos, 1, 1, d // 2] if @param reuse_freqs_front_part else [max_pos, 1, 1, d]
 * @param sin          [max_pos, 1, 1, d // 2] if @param reuse_freqs_front_part else [max_pos, 1, 1, d]
 * @param positions    [s, b]
 * @param offsets      [s, b]
 * @param rotate_style 0: NEOX style, 1: GPT-J style
 * @param nope_first   If true, back part in last dimension of input is rotated. Otherwise, the front part is rotated.
 */
void rope_cached_positions_offsets_2c_fwd_impl(
    torch::Tensor&       output_x,
    torch::Tensor&       output_y,
    const torch::Tensor& input_x,
    const torch::Tensor& input_y,
    const torch::Tensor& cos,
    const torch::Tensor& sin,
    const torch::Tensor& positions,
    const torch::Tensor& offsets,
    const int32_t        rotate_style,
    const bool           reuse_freqs_front_part,
    const bool           nope_first)
{
    // Get sizes of input and output
    const int32_t size_s   = min(min(input_x.size(0), positions.size(0)), offsets.size(0));
    const int32_t size_b   = min(min(input_x.size(1), positions.size(1)), offsets.size(1));
    const int32_t size_h_x = input_x.size(2);
    const int32_t size_h_y = input_y.size(2);
    const int32_t size_d   = input_x.size(3);
    const int32_t size_f   = cos.size(3);

    // Get strides of input
    const int32_t stride_ix_s = input_x.stride(0);
    const int32_t stride_ix_b = input_x.stride(1);
    const int32_t stride_ix_h = input_x.stride(2);
    const int32_t stride_ix_d = input_x.stride(3);
    const int32_t stride_iy_s = input_y.stride(0);
    const int32_t stride_iy_b = input_y.stride(1);
    const int32_t stride_iy_h = input_y.stride(2);
    const int32_t stride_iy_d = input_y.stride(3);

    // Get strides of output
    const int32_t stride_ox_s = output_x.stride(0);
    const int32_t stride_ox_b = output_x.stride(1);
    const int32_t stride_ox_h = output_x.stride(2);
    const int32_t stride_ox_d = output_x.stride(3);
    const int32_t stride_oy_s = output_y.stride(0);
    const int32_t stride_oy_b = output_y.stride(1);
    const int32_t stride_oy_h = output_y.stride(2);
    const int32_t stride_oy_d = output_y.stride(3);

    // Get strides of positions and offsets
    assert(1 == positions.stride(1) && 2 == positions.dim());
    assert(1 == offsets.stride(1)   && 2 == offsets.dim());
    const int32_t max_position = cos.size(0);

    DISPATCH_ROPE_TYPES_PARAMS(
        input_x.scalar_type(),
        cos.scalar_type(),
        rotate_style,
        reuse_freqs_front_part,
        nope_first,
        "dispatch_2c_sbhd_cached_indirect2<OpCachedFwd, ...>",
        dispatch_2c_sbhd_cached_indirect2<OpCachedFwd, RotateStyle, ReuseFreqsFrontPart, NopeFirst>(
            output_x.data_ptr<scalar_t_0>(),
            output_y.data_ptr<scalar_t_0>(),
            input_x.data_ptr<scalar_t_0>(),
            input_y.data_ptr<scalar_t_0>(),
            cos.data_ptr<scalar_t_1>(),
            sin.data_ptr<scalar_t_1>(),
            positions.data_ptr<int64_t>(),
            offsets.data_ptr<int64_t>(),
            max_position,
            size_s, size_b, size_h_x, size_h_y, size_d,
            size_f, // size of last dimension of freqs.
            stride_ix_s, stride_ix_b, stride_ix_h, stride_ix_d,
            stride_iy_s, stride_iy_b, stride_iy_h, stride_iy_d,
            stride_ox_s, stride_ox_b, stride_ox_h, stride_ox_d,
            stride_oy_s, stride_oy_b, stride_oy_h, stride_oy_d););
}