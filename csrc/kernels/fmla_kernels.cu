#include "hip/hip_runtime.h"
// SPDX-License-Identifier: MIT
// Copyright (c) 2025, Advanced Micro Devices, Inc. All rights reserved.

#include <torch/python.h>
#include <c10/cuda/CUDAGuard.h>
#include <ck_tile/core.hpp>
#include <ck_tile/host.hpp>
#include <ck_tile/host.hpp>
#include <ck_tile/ops/gemm.hpp>
#include <ck_tile/ops/gemm.hpp>

#include <ck_tile/ops/reduce/block/block_reduce.hpp>
#include <ck_tile/ops/fmha/block/page_block_navigator.hpp>

// =====================================================================================================================
// Definitions and helper structures
//

template <int32_t kSizeD_,
          int32_t kSizeDV_,
          int32_t kBlockM_,
          int32_t kBlockN_,
          int32_t kNumWarps_>
struct FlashMlaKernelTrait
{
    static constexpr int32_t kSizeD                  = kSizeD_;    // hidden dimension size of query and key
    static constexpr int32_t kSizeDV                 = kSizeDV_;   // hidden dimension size of value
    static constexpr int32_t kNumWarps               = kNumWarps_;
    static constexpr int32_t kNumThreads             = kNumWarps * warpSize;
    static constexpr int32_t kNumWarpsSoftmax        = 2;
    static constexpr int32_t kNumThreadsSoftmax      = kNumWarpsSoftmax * warpSize;
    static constexpr int32_t kNumWarpsCombine        = 2;
    static constexpr int32_t kNumThreadsCombine      = kNumWarpsCombine * ck_tile::get_warp_size();
    static constexpr int32_t kBlockM                 = kBlockM_;
    static constexpr int32_t kBlockN                 = kBlockN_;
    static constexpr int32_t kFixedOverheadNumBlocks = 5;
    static constexpr int32_t kMaxBatchSize           = 4096;

    static constexpr int32_t kLdsOffsetP        = 2 * kBlockN * kSizeD * 2;
    static constexpr int32_t kLdsOffsetScale    = kLdsOffsetP + kBlockN * kBlockM * 2;
    static constexpr int32_t kLdsOffsetMax      = kLdsOffsetScale + kNumThreadsSoftmax * 4;
    static constexpr int32_t kLdsOffsetSum      = kLdsOffsetMax + kNumThreadsSoftmax * 4;

    static_assert(kSizeD % 64 == 0);
    static_assert(kSizeDV % 64 == 0);
    static_assert(kSizeD >= kSizeDV);

    using Gemm0BlockWarps = ck_tile::sequence<4, 1, 1>;
    using Gemm1BlockWarps = ck_tile::sequence<4, 1, 1>;
    using Gemm0WarpTile = ck_tile::sequence<16, 16, 16>;
    using Gemm1WarpTile = ck_tile::sequence<16, 16, 16>;

    static constexpr int32_t kNumGemm0Warps = kNumWarps_;
    static constexpr int32_t kNumGemm1Warps = kNumWarps_;
    static constexpr int32_t kBlockSize = kNumWarps * warpSize;
    static constexpr bool TransposeC = true;
    static constexpr bool GemmPVLds = true;

    static constexpr int32_t kStages = 2;
};

// using FlashMlaKernelTraitsInstance = FlashMlaKernelTrait<576, 512, 64, 64, 4>;
using FlashMlaKernelTraitsInstance = FlashMlaKernelTrait<576, 512, 64, 16, 4>;

template <typename Traits, typename scalar_t, typename acc_t>
struct FlashMlaKernelPolicy
{
private:
    constexpr static auto q_dram_window_lengths = ck_tile::make_tuple(Traits::kBlockM, Traits::kSizeD);
    constexpr static auto lse_dram_window_lengths = ck_tile::make_tuple(Traits::kBlockM);
    constexpr static auto o_dram_window_lengths = ck_tile::make_tuple(Traits::kBlockM, Traits::kSizeDV);
    constexpr static auto I0 = ck_tile::number<0>{};
    constexpr static auto I1 = ck_tile::number<1>{};
    constexpr static auto kPackSize = 16 / sizeof(scalar_t);

    CK_TILE_DEVICE static constexpr auto MakeKLdsBlockDescriptor()
    {
        constexpr auto k_lds_block_desc = ck_tile::make_naive_tensor_descriptor(
            ck_tile::make_tuple(Traits::kStages * Traits::kBlockN,
                                Traits::kSizeD / kPackSize,
                                kPackSize),
            ck_tile::make_tuple((Traits::kSizeD / kPackSize + 1) * kPackSize,
                                kPackSize,
                                1),
            ck_tile::number<kPackSize>{},
            I1);
        constexpr auto k_lds_block_desc_merge = ck_tile::transform_tensor_descriptor(
            k_lds_block_desc,
            ck_tile::make_tuple(
                    ck_tile::make_pass_through_transform(ck_tile::number<Traits::kStages * Traits::kBlockN>{}),
                    ck_tile::make_merge_transform(
                        ck_tile::make_tuple(ck_tile::number<Traits::kSizeDV / kPackSize>{},
                        kPackSize))),
            ck_tile::make_tuple(ck_tile::sequence<0>{}, ck_tile::sequence<1, 2>{}),
            ck_tile::make_tuple(ck_tile::sequence<0>{}, ck_tile::sequence<1>{}));

        return k_lds_block_desc_merge;
    }

    CK_TILE_DEVICE static constexpr auto MakeVLdsBlockDescriptor()
    {
        constexpr auto v_lds_block_desc = ck_tile::make_naive_tensor_descriptor(
            ck_tile::make_tuple(Traits::kStages * Traits::kBlockN,
                                Traits::kSizeDV / kPackSize,
                                kPackSize),
            ck_tile::make_tuple((Traits::kSizeD / kPackSize + 1) * kPackSize,
                                kPackSize,
                                1),
            ck_tile::number<kPackSize>{},
            I1);

        constexpr auto v_lds_block_desc_transpose = ck_tile::transform_tensor_descriptor(
            v_lds_block_desc,
            ck_tile::make_tuple(
                    ck_tile::make_pass_through_transform(ck_tile::number<Traits::kStages * Traits::kBlockN>{}),
                    ck_tile::make_merge_transform(
                        ck_tile::make_tuple(ck_tile::number<Traits::kSizeDV / kPackSize>{},
                        kPackSize))),
            ck_tile::make_tuple(ck_tile::sequence<0>{}, ck_tile::sequence<1, 2>{}),
            ck_tile::make_tuple(ck_tile::sequence<1>{}, ck_tile::sequence<0>{}));
        return v_lds_block_desc_transpose;
    }

    CK_TILE_DEVICE static constexpr auto MakeVLds2RegBlockDistribution()
    {
        constexpr auto config = decltype(GetPVBlockGemm())::Policy::template GetWarpGemmMWarpNWarp<GemmPVProblem>();
        using WarpGemm        = ck_tile::remove_cvref_t<decltype(config.template at<0>())>;

        constexpr int32_t MWarp = Traits::Gemm1BlockWarps::at(ck_tile::number<0>{});
        constexpr int32_t NWarp = Traits::Gemm1BlockWarps::at(ck_tile::number<1>{});

        constexpr int32_t kNPerBlock = Traits::kSizeDV;
        constexpr int32_t kKPerBlock = Traits::kBlockN;

        constexpr int32_t NIterPerWarp = kNPerBlock / (NWarp * WarpGemm::kN);
        constexpr int32_t KIterPerWarp = kKPerBlock / WarpGemm::kK;

        constexpr auto vt_lds_outer_encode =
            ck_tile::tile_distribution_encoding<
                ck_tile::sequence<MWarp>,
                ck_tile::tuple<ck_tile::sequence<NIterPerWarp, NWarp>, ck_tile::sequence<KIterPerWarp>>,
                ck_tile::tuple<ck_tile::sequence<0, 1>>,
                ck_tile::tuple<ck_tile::sequence<0, 1>>,
                ck_tile::sequence<1, 2>,
                ck_tile::sequence<0, 0>>{};

        constexpr auto vt_lds_shuffle_block_dstr_encode = ck_tile::detail::make_embed_tile_distribution_encoding(
            vt_lds_outer_encode, typename WarpGemm::BWarpDstrEncoding{});

        return ck_tile::make_static_tile_distribution(vt_lds_shuffle_block_dstr_encode);
    }

public:
    using GemmQKProblem = ck_tile::BlockGemmProblem<
        scalar_t,
        scalar_t,
        acc_t,
        Traits::kNumGemm0Warps * ck_tile::get_warp_size(), 
        ck_tile::TileGemmShape<ck_tile::sequence<Traits::kBlockM,
                                                 Traits::kBlockN,
                                                 Traits::kSizeD>,
            typename Traits::Gemm0BlockWarps,
            typename Traits::Gemm0WarpTile>>;

    using GemmPVProblem = ck_tile::BlockGemmProblem<
        scalar_t,
        scalar_t,
        acc_t,
        Traits::kNumGemm1Warps * ck_tile::get_warp_size(),
        ck_tile::TileGemmShape<ck_tile::sequence<Traits::kBlockM,
                                                 Traits::kSizeDV,
                                                 Traits::kBlockN>,
            typename Traits::Gemm1BlockWarps,
            typename Traits::Gemm1WarpTile>>;

    CK_TILE_DEVICE static constexpr auto GetQKBlockGemm()
    {
        constexpr auto warp_gemm = []() {
            constexpr int32_t WarpGemmM = Traits::Gemm0WarpTile::at(ck_tile::number<0>{});
            if constexpr(std::is_same_v<scalar_t, ck_tile::half_t> && 
                         std::is_same_v<acc_t, float>)
            {
                if constexpr(WarpGemmM == 32)
                    return ck_tile::WarpGemmMfmaF16F16F32M32N32K16SwizzleBTransposedCDistribution{};
                else if constexpr(WarpGemmM == 16)
                    return ck_tile::WarpGemmMfmaF16F16F32M16N16K16TransposedCDistribution{};
                else
                    return ck_tile::WarpGemmMfmaF16F16F32M4N64K16{};
            }
            else if constexpr(std::is_same_v<scalar_t, ck_tile::bf16_t> &&
                              std::is_same_v<acc_t, float>)
            {
                if constexpr(WarpGemmM == 32)
                    return ck_tile::WarpGemmMfmaBf16Bf16F32M32N32K16SwizzleBTransposedCDistribution{};
                else if constexpr(WarpGemmM == 16)
                    return ck_tile::WarpGemmMfmaBf16Bf16F32M16N16K16TransposedCDistribution{};
                else
                    return ck_tile::WarpGemmMfmaBf16Bf16F32M4N64K16{};
            }
        }();

        using BlockGemmPolicy = ck_tile::BlockGemmARegBSmemCRegV2CustomPolicy<
            scalar_t,
            scalar_t,
            acc_t,
            typename Traits::Gemm0BlockWarps,
            decltype(warp_gemm)>;

        if constexpr(1 < Traits::kNumGemm0Warps)
            return ck_tile::BlockGemmARegBSmemCRegV2<GemmQKProblem, BlockGemmPolicy>{};
        else
            return ck_tile::BlockGemmARegBSmemCRegOneWarpV1<GemmQKProblem, BlockGemmPolicy>{};
    }

    CK_TILE_DEVICE static constexpr auto GetPVBlockGemm()
    {
        constexpr auto warp_gemm = ck_tile::WarpGemmMfmaDispatcher<
            scalar_t,
            scalar_t,
            acc_t,
            Traits::Gemm1WarpTile::at(ck_tile::number<0>{}),
            Traits::Gemm1WarpTile::at(ck_tile::number<1>{}),
            Traits::Gemm1WarpTile::at(ck_tile::number<2>{}),
            Traits::TransposeC>{};

        if constexpr (Traits::GemmPVLds)
        {
            using BlockGemmPolicy =
                ck_tile::BlockGemmARegBSmemCRegV2CustomPolicy<scalar_t,
                                                     scalar_t,
                                                     acc_t,
                                                     typename Traits::Gemm1BlockWarps,
                                                     decltype(warp_gemm)>;
            return ck_tile::BlockGemmARegBSmemCRegV2<GemmPVProblem, BlockGemmPolicy>{};
        }
        else
        {
            using BlockGemmPolicy =
                ck_tile::BlockGemmARegBRegCRegV1CustomPolicy<scalar_t,
                                                     scalar_t,
                                                     acc_t,
                                                     typename Traits::Gemm1BlockWarps,
                                                     decltype(warp_gemm)>;
            return ck_tile::BlockGemmARegBRegCRegV1<GemmPVProblem, BlockGemmPolicy>{};
        }
    }

    CK_TILE_DEVICE static auto MakeQDramTileWindow(
        const scalar_t* p_query_in,
        const int32_t size_s,
        const int32_t stride_s_q)
    {
        // q: [batch, size_s, size_h, sizeD]
        auto q_dram_naive =
            ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_query_in,
                ck_tile::make_tuple(size_s, Traits::kSizeD), // lengths
                ck_tile::make_tuple(stride_s_q, 1),  // strides
                ck_tile::number<Traits::kSizeD>{},  // last dim alignment
                I1);                               // last dim stride

        // q_tile per block: [kBlockM, kSizeD], q load once
        auto q_dram_padding = ck_tile::pad_tensor_view(
            q_dram_naive,
            q_dram_window_lengths,
            ck_tile::sequence<true, false>{});

        return ck_tile::make_tile_window(
			q_dram_padding,
			q_dram_window_lengths,
			{0, 0},
			ck_tile::remove_cvref_t<decltype(GetQKBlockGemm())>::template MakeABlockTileDistribution<
				Traits::kBlockM,
				Traits::kSizeD>());
    }

    CK_TILE_DEVICE static auto MakeLSEDramTileWindow(
        acc_t* p_lse_out,
        const int32_t size_s,
        const ck_tile::index_t begin_idx = 0)
    {
        const auto lse_dram =
            ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_lse_out,
                ck_tile::make_tuple(size_s),
                ck_tile::make_tuple(1),
                I1,
                I1);

        // lseacc window lengths: [BlockM]
        return ck_tile::make_tile_window(lse_dram, lse_dram_window_lengths, {begin_idx});
    }

    template<typename ODataType>
    CK_TILE_DEVICE static auto MakeODramTileWindow(
        ODataType* p_output_out,
        const int32_t size_s,
        const ck_tile::index_t begin_idx = 0)
    {
        const auto o_dram = 
            ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_output_out,
                ck_tile::make_tuple(size_s, Traits::kSizeDV),
                ck_tile::make_tuple(Traits::kSizeDV, 1),
                ck_tile::number<Traits::kSizeDV>{},
                I1);

        return ck_tile::make_tile_window(o_dram, o_dram_window_lengths, {begin_idx, 0});
    }

    CK_TILE_DEVICE static auto MakeKLdsTileWindow(scalar_t* k_lds_ptr)
    {
        auto k_lds = ck_tile::make_tensor_view<ck_tile::address_space_enum::lds>(
            k_lds_ptr, MakeKLdsBlockDescriptor());

        auto k_st_lds_window = ck_tile::make_tile_window(k_lds,
            ck_tile::make_tuple(ck_tile::number<Traits::kBlockN>{},
                                ck_tile::number<Traits::kSizeD>{}), {0, 0});
        auto k_ld_lds_window = ck_tile::make_tile_window(k_lds,
            ck_tile::make_tuple(ck_tile::number<Traits::kBlockN>{},
                                ck_tile::number<Traits::kSizeD>{}), {0, 0});

        return ck_tile::make_tuple(k_st_lds_window, k_ld_lds_window);
    }

    CK_TILE_DEVICE static auto MakeVLdsTileWindow(scalar_t* v_lds_ptr)
    {
        auto v_lds = ck_tile::make_tensor_view<ck_tile::address_space_enum::lds>(
            v_lds_ptr, MakeVLdsBlockDescriptor());

        if constexpr (Traits::GemmPVLds)
            return ck_tile::make_tile_window(v_lds,
                ck_tile::make_tuple(ck_tile::number<Traits::kSizeDV>{},
                                    ck_tile::number<Traits::kBlockN>{}),
                {0, 0},
                MakeVLds2RegBlockDistribution());
        else
            return ck_tile::make_tile_window(v_lds,
                ck_tile::make_tuple(ck_tile::number<Traits::kSizeDV>{},
                                    ck_tile::number<Traits::kBlockN>{}),
                {0, 0});

    }

    CK_TILE_DEVICE static auto MakeKPageBlockNavigator(
        void* p_key,
        const int32_t* p_block_table,
        const int32_t page_block_size,
        const int32_t stride_b_k,
        const int32_t stride_s_k,
        const int32_t batch_offset,
        const int32_t fixed_offset,
        const int32_t seqlen_kv_end)
    {
        const auto make_k_dram = [&](const scalar_t* data, int32_t height) {
            const auto k_dram_naive = ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                data,
                ck_tile::make_tuple(height, Traits::kSizeD),
                ck_tile::make_tuple(stride_s_k, 1),
                ck_tile::number<Traits::kSizeD>{},  // last dim alignment
                I1);

            return ck_tile::pad_tensor_view(
                k_dram_naive,
                ck_tile::make_tuple(page_block_size, Traits::kSizeD),
                ck_tile::sequence<true, false>{});
        };

        const auto* block_indices = p_block_table + batch_offset;

        const int32_t num_blocks =
            ck_tile::integer_divide_ceil(seqlen_kv_end, page_block_size);

        return ck_tile::make_page_block_navigator<const scalar_t, 0>(
            reinterpret_cast<scalar_t*>(p_key),
            stride_b_k,
            fixed_offset,
            block_indices,
            num_blocks,
            page_block_size, // page_size
            make_k_dram(nullptr, page_block_size),
            make_k_dram(nullptr,
                        (seqlen_kv_end - (num_blocks - 1) * page_block_size)));
    }

    // TODO: control the speed of k copy
    CK_TILE_DEVICE static constexpr auto MakeKDramTileDistribution()
    {
        constexpr int32_t RepeatsK = 9;
        constexpr int32_t RepeatsN = 1;

        constexpr int32_t kVectorN = 1; // for continous K copy
        constexpr int32_t VectorKMax = 16 / sizeof(scalar_t);

        constexpr int32_t ThreadsPerKMin = Traits::kSizeD / VectorKMax / RepeatsK;
        constexpr int32_t kThrPerBlockN =
            ck_tile::min(Traits::kBlockN / kVectorN, Traits::kBlockSize / ThreadsPerKMin);
        constexpr int32_t kThrPerBlockK = Traits::kBlockSize / kThrPerBlockN;

        constexpr int32_t kNumWarpN = Traits::kNumWarps;
        constexpr int32_t kNumWarpK = 1;

        constexpr int32_t kThrPerWarpN = kThrPerBlockN / kNumWarpN;
        constexpr int32_t kThrPerWarpK = ck_tile::get_warp_size() / kThrPerWarpN;

        constexpr int32_t kVectorK = Traits::kSizeD / RepeatsK / kThrPerBlockK;

        return ck_tile::make_static_tile_distribution(
            ck_tile::tile_distribution_encoding<
                ck_tile::sequence<>,
                ck_tile::tuple<ck_tile::sequence<RepeatsN, kNumWarpN, kThrPerWarpN, kVectorN>,
                               ck_tile::sequence<RepeatsK, kNumWarpK, kThrPerWarpK, kVectorK>>,
                ck_tile::tuple<ck_tile::sequence<1, 2>, ck_tile::sequence<1, 2>>,
                ck_tile::tuple<ck_tile::sequence<1, 1>, ck_tile::sequence<2, 2>>,
                ck_tile::sequence<1, 1, 2, 2>,
                ck_tile::sequence<0, 3, 0, 3>>{});

        // return ck_tile::make_static_tile_distribution(
        //     ck_tile::tile_distribution_encoding<
        //         ck_tile::sequence<>,
        //         ck_tile::tuple<ck_tile::sequence<1, 2, 8, 1>,
        //                        ck_tile::sequence<9, 2, 8, 4>>,
        //         ck_tile::tuple<ck_tile::sequence<1, 2>, ck_tile::sequence<1, 2>>,
        //         ck_tile::tuple<ck_tile::sequence<1, 1>, ck_tile::sequence<2, 2>>,
        //         ck_tile::sequence<1, 1, 2, 2>,
        //         ck_tile::sequence<0, 3, 0, 3>>{});
    }

    
    CK_TILE_DEVICE static constexpr auto MakePShuffleTileDistribution()
    {
        constexpr auto p_encoding = decltype(GetQKBlockGemm().MakeCBlockTile())::get_tile_distribution().get_static_tile_distribution_encoding();
		constexpr auto N = p_encoding.hs_lengthss_.at(I0);
		constexpr auto K = p_encoding.hs_lengthss_.at(I1);

        return ck_tile::make_static_tile_distribution(
            ck_tile::tile_distribution_encoding<ck_tile::sequence<K[1]>,
                   ck_tile::tuple<ck_tile::sequence<N[0], N[1], N[2]>, ck_tile::sequence<K[2], K[3], K[4]>>,
                   ck_tile::tuple<ck_tile::sequence<1, 0>, ck_tile::sequence<2, 1>>,
                   ck_tile::tuple<ck_tile::sequence<1, 0>, ck_tile::sequence<1, 2>>,
                   ck_tile::sequence<1, 2, 2>,
                   ck_tile::sequence<0, 0, 2>>{});
    }
};


//====================================

template <typename Traits, typename scalar_t, typename acc_t>
struct FlashMlaCombineKernelPolicy
{
private:
    // Returns count of warps which don't contain any idle thread.
    template <int32_t NumWarps, int32_t M, int32_t N>
    CK_TILE_HOST_DEVICE static constexpr auto GetMaxNumWarpsForTile()
    {
        static_assert(NumWarps == 1 || NumWarps == 2 || NumWarps == 4);
        constexpr int32_t ElemPerThread = (M * N) / (NumWarps * ck_tile::get_warp_size());
        if constexpr(0 < ElemPerThread)
        {
            return NumWarps;
        }
        else
        {
            return GetMaxNumWarpsForTile<NumWarps / 2, M, N>();
        }
    }

    // Returns vector size for given warp count for handing the specified matrix.
    template <int32_t NumWarps, int32_t M, int32_t N, typename DataType>
    CK_TILE_HOST_DEVICE static constexpr auto GetVectorSizeForTile()
    {
        constexpr int32_t MaxNumWarps = GetMaxNumWarpsForTile<NumWarps, M, N>();
        constexpr int32_t ElemPerThread = (M * N) / (MaxNumWarps * ck_tile::get_warp_size());
        constexpr int32_t MaxNPerThread = 16 / sizeof(DataType);
        return ck_tile::min(MaxNPerThread, ElemPerThread);
    }

    template <typename DataType>
    CK_TILE_DEVICE static constexpr auto MakeOutputTileDistribution()
    {
        constexpr int32_t kVectorN     = GetVectorSizeForTile<Traits::kNumWarpsCombine, 1, Traits::kSizeDV, DataType>();
        constexpr int32_t kThrPerWarpN = ck_tile::get_warp_size();
        constexpr int32_t kNumWarpN    = Traits::kNumWarpsCombine;

        return ck_tile::make_static_tile_distribution(
            ck_tile::tile_distribution_encoding<
                ck_tile::sequence<>,    // no replicate
                ck_tile::tuple<ck_tile::sequence<1>,
                               ck_tile::sequence<kNumWarpN, kThrPerWarpN, kVectorN>>,
                ck_tile::tuple<ck_tile::sequence<2>, ck_tile::sequence<2>>,
                ck_tile::tuple<ck_tile::sequence<0>, ck_tile::sequence<1>>,
                ck_tile::sequence<1, 2>,
                ck_tile::sequence<0, 2>>{});
    }

public:
    CK_TILE_DEVICE static auto MakeOaccuTileWindow(
        void* p_output_accum,
        const int32_t hsidx,
        const int32_t size_hs,
        const int32_t split_offset,
        const int32_t num_splits)
    {
        const int32_t offset_oaccum = split_offset * size_hs * Traits::kSizeDV;

        // Shape of tensor for a block: [num_splits, Traits::kSizeDV]
        const auto naive_view =
            ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                reinterpret_cast<acc_t*>(p_output_accum) + offset_oaccum,
                ck_tile::make_tuple(num_splits * size_hs, Traits::kSizeDV), // lengths
                ck_tile::make_tuple(Traits::kSizeDV, 1),                    // strides
                ck_tile::number<Traits::kSizeDV>{},                         // last dim alignment
                ck_tile::number<1>{});                                      // last dim stride

        // Each thread group handles tile whose shape is [1, Traits::kSizeDV]
        const auto tile_window = ck_tile::make_tile_window(
            naive_view,
            ck_tile::make_tuple(ck_tile::number<1>{},               // window size
                                ck_tile::number<Traits::kSizeDV>{}),
            {hsidx, 0});                          // origin

        return ck_tile::make_tile_window(tile_window, MakeOutputTileDistribution<acc_t>());
    }

    CK_TILE_DEVICE static auto MakeOutputTileWindow(
        void* p_output,
        const int32_t offset_b,
        const int32_t offset_s,
        const int32_t offset_h)
    {
        scalar_t* p_out = reinterpret_cast<scalar_t*>(p_output) + offset_b + offset_s + offset_h;

        const auto naive_view =
            ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_out,
                ck_tile::make_tuple(1, Traits::kSizeDV),    // lengths
                ck_tile::make_tuple(Traits::kSizeDV, 1),    // strides
                ck_tile::number<Traits::kSizeDV>{},         // last dim alignment
                ck_tile::number<1>{});                      // last dim stride

        const auto tile_window = ck_tile::make_tile_window(
            naive_view,
            ck_tile::make_tuple(ck_tile::number<1>{},               // window size
                                ck_tile::number<Traits::kSizeDV>{}),
            {0, 0});                                                // origin

        return ck_tile::make_tile_window(tile_window, MakeOutputTileDistribution<scalar_t>());
    }
};

//====================================

union TileSchedulerMetaData
{
    struct Core
    {
        int32_t begin_batch_idx;
        int32_t begin_seqlen_idx;
        int32_t end_batch_idx;
        int32_t end_seqlen_idx;
        int32_t begin_n_split_idx;
    };
    uint32_t data[8];
    Core core;
};
constexpr size_t TileSchedulerMetaDataSizeInDw = sizeof(TileSchedulerMetaData) / sizeof(int32_t);
constexpr size_t TileSchedulerMetaDataSizeInInt4 = sizeof(TileSchedulerMetaData) / sizeof(int4);

struct FlashMlaFwdParams
{
    int32_t* __restrict__ p_cu_seqlens_k;
    int32_t* __restrict__ p_block_table;
    int32_t* __restrict__ p_tile_scheduler_metadata;
    int32_t* __restrict__ p_num_splits;
    
    void* __restrict__ p_query;
    void* __restrict__ p_key;
    void* __restrict__ p_value;
    void* __restrict__ p_output;
    void* __restrict__ p_softmax_lse;
    void* __restrict__ p_softmax_lseaccum;
    void* __restrict__ p_output_accum;

    int32_t size_b;
    int32_t size_s;
    int32_t size_h;
    int32_t hq_hk_ratio;
    int32_t num_groups;
    int32_t num_cu_parts;
    int64_t block_table_batch_stride;
    int32_t page_block_size;
    float   scale_softmax;
    float   scale_softmax_log2;
    bool    is_causal;

    // Use int64_t if there is int32 overflow case. For now, just use int32 to save sgpr and prevent using
    // spill table.
    using index_t = int32_t;

    index_t stride_b_q;     // stride in batch of query
    index_t stride_s_q;     //    ... in sequence ...
    index_t stride_h_q;     //    ... in head ...
    index_t stride_b_k;     // stride in batch of key
    index_t stride_s_k;     //    ... in sequence ...
    index_t stride_h_k;     //    ... in head ...
    index_t stride_b_v;     // stride in batch of value
    index_t stride_s_v;     //    ... in sequence ...
    index_t stride_h_v;     //    ... in head ...
    index_t stride_b_o;     // stride in batch of output
    index_t stride_s_o;     //    ... in sequence ...
    index_t stride_h_o;     //    ... in head ...
};

// =====================================================================================================================
// Kernel Entries
//

template <typename Traits, typename scalar_t, typename acc_t, bool Is_causal>
__global__ void flash_fwd_splitkv_mla_kernel(
    const FlashMlaFwdParams params)
{
    using Policy  = FlashMlaKernelPolicy<Traits, scalar_t, float>;

    constexpr int32_t kSizeD             = Traits::kSizeD; 
    constexpr int32_t kSizeDV            = Traits::kSizeDV; 
    constexpr int32_t kNumThreads        = Traits::kNumThreads;
    constexpr int32_t kNumThreadsSoftmax = Traits::kNumThreadsSoftmax;
    constexpr int32_t kBlockM            = Traits::kBlockM;
    constexpr int32_t kBlockN            = Traits::kBlockN;
    constexpr int32_t kLdsOffsetP        = 2 * kBlockN * kSizeD;
    constexpr int32_t kLdsOffsetScale    = kLdsOffsetP + kNumThreadsSoftmax;
    constexpr int32_t kLdsOffsetMax      = kLdsOffsetScale + kNumThreadsSoftmax;
    constexpr int32_t kLdsOffsetSum      = kLdsOffsetMax + kNumThreadsSoftmax;

    constexpr int32_t kPackScalar = 16 / sizeof(scalar_t);
    constexpr int32_t kPackAcc = 16 / sizeof(scalar_t);
    constexpr int32_t kKPack = kPackScalar;

    constexpr auto I0 = ck_tile::number<0>{};
    constexpr auto I1 = ck_tile::number<1>{};
    constexpr auto IBlockM = ck_tile::number<kBlockM>{};
    constexpr auto IBlockN = ck_tile::number<kBlockN>{};
    constexpr auto IPack = ck_tile::number<kKPack>{};

    const int32_t i_block_m   = blockIdx.x;
    const int32_t i_nhead     = blockIdx.y;
    const int32_t i_nhead_k   = i_nhead / params.hq_hk_ratio;
    const int32_t i_partition = blockIdx.z;

    const ck_tile::index_t i_m0 = __builtin_amdgcn_readfirstlane(i_block_m * kBlockM);

    const auto f_max = [](auto e0, auto e1) { return max(e0, e1); };
    const auto f_sum = [](auto e0, auto e1) { return e0 + e1; };

    extern __shared__ char shared_memory[];
	char *shared_ptr = (char *)(((size_t)shared_memory + 255) & ~255);

    const int32_t tidx = threadIdx.x; 

    auto gemm_0 = Policy::GetQKBlockGemm();

    auto s_acc = gemm_0.MakeCBlockTile();
    using SBlockTileType = decltype(ck_tile::cast_tile<acc_t>(s_acc));
    using MLBlockTileType = decltype(ck_tile::block_tile_reduce<acc_t>(
        SBlockTileType{}, ck_tile::sequence<1>{}, f_max, acc_t{0}));
    auto m = MLBlockTileType{};
    auto l = MLBlockTileType{};

    auto gemm_1 = Policy::GetPVBlockGemm();
    auto o_acc = gemm_1.MakeCBlockTile();

    scalar_t* kv_lds_ptr = reinterpret_cast<scalar_t*>(shared_ptr);

    auto p_shuffle_distribution = [&]() {
        auto p_encoding = s_acc.get_tile_distribution().get_static_tile_distribution_encoding();
		constexpr auto N = p_encoding.hs_lengthss_.at(I0);
		constexpr auto K = p_encoding.hs_lengthss_.at(I1);
		constexpr auto N0 = N[0];
		constexpr auto N1 = N[1];
		constexpr auto N2 = N[2];
		constexpr auto K0 = K[0];
		constexpr auto K1 = K[1];
		constexpr auto K2 = K[2];
		constexpr auto K3 = K[3];
		constexpr auto K4 = K[4];

        return make_static_tile_distribution(
            ck_tile::tile_distribution_encoding<ck_tile::sequence<K1>,
                                       ck_tile::tuple<ck_tile::sequence<N0, N1, N2>, ck_tile::sequence<K2, K3, K4>>,
                                       ck_tile::tuple<ck_tile::sequence<1, 0>, ck_tile::sequence<2, 1>>,
                                       ck_tile::tuple<ck_tile::sequence<1, 0>, ck_tile::sequence<1, 2>>,
                                       ck_tile::sequence<1, 2, 2>,
                                       ck_tile::sequence<0, 0, 2>>{});
    }();

    TileSchedulerMetaData metadata;
    reinterpret_cast<int4*>(&(metadata.data))[0] = reinterpret_cast<int4*>(
        params.p_tile_scheduler_metadata)[i_partition * TileSchedulerMetaDataSizeInInt4];
    reinterpret_cast<int4*>(&(metadata.data))[1] = reinterpret_cast<int4*>(
        params.p_tile_scheduler_metadata)[i_partition * TileSchedulerMetaDataSizeInInt4 + 1];

    const int32_t begin_batch_idx   = metadata.core.begin_batch_idx;
    const int32_t begin_seqlen_idx  = metadata.core.begin_seqlen_idx;
    const int32_t end_batch_idx     = metadata.core.end_batch_idx;
    const int32_t end_seqlen_idx    = metadata.core.end_seqlen_idx;
    const int32_t begin_n_split_idx = metadata.core.begin_n_split_idx;

    for (int32_t i_batch = begin_batch_idx; i_batch <= end_batch_idx; ++i_batch)
    {
        const int32_t i_split = i_batch == begin_batch_idx ? begin_n_split_idx : 0;
        const int32_t seqlen_k    = params.p_cu_seqlens_k[i_batch];
        const int32_t n_block_min = i_batch == begin_batch_idx ? begin_seqlen_idx / kBlockN : 0;
        const int32_t n_block_max = i_batch == end_batch_idx ? ck_tile::integer_divide_ceil(end_seqlen_idx, kBlockN) : ck_tile::integer_divide_ceil(seqlen_k, kBlockN);
        const bool NoSplit = n_block_min == 0 && n_block_max == ck_tile::integer_divide_ceil(seqlen_k, kBlockN);
        const int32_t split_seqlen_k_begin = i_batch == begin_batch_idx ? begin_seqlen_idx : 0;
        const int32_t split_seqlen_k_end = i_batch == end_batch_idx ? end_seqlen_idx : seqlen_k;

        int32_t i_block_n = n_block_max - 1;

        const int32_t total_seqlen_kv = (n_block_max - n_block_min) * kBlockN;

        // if (!NoSplit) continue;

        if (i_batch > begin_batch_idx)
        {
            __syncthreads();
        }

        ck_tile::clear_tile(o_acc);
        ck_tile::clear_tile(m);
        ck_tile::clear_tile(l);

        const int32_t q_offset = i_batch * params.stride_b_q +
                                 i_block_m * kBlockM * params.stride_s_q +
                                 i_nhead * params.stride_h_q;
        auto q_dram_window = Policy::MakeQDramTileWindow(
            reinterpret_cast<scalar_t*>(params.p_query) + q_offset,
            params.size_s,
            params.stride_s_q);
        auto q = load_tile(q_dram_window);

        auto k_page_block_navigator = Policy::MakeKPageBlockNavigator(
            params.p_key,
            params.p_block_table,
            params.page_block_size,
            params.stride_b_k,
            params.stride_s_k,
            params.block_table_batch_stride * i_batch,
            params.stride_h_k * i_nhead_k,
            split_seqlen_k_end);

        constexpr static auto k_dram_window_lengths = ck_tile::make_tuple(Traits::kBlockN, Traits::kSizeD);

        auto [i_page_block_k_tail, k_dram_window_tail] = k_page_block_navigator.make_tile_window(
            k_dram_window_lengths, {(n_block_max - 1) * kBlockN, 0}, Policy::MakeKDramTileDistribution());


        auto [k_st_lds_window, k_ld_lds_window] = Policy::MakeKLdsTileWindow(kv_lds_ptr);

        auto v_ld_lds_window = Policy::MakeVLdsTileWindow(kv_lds_ptr);

        int32_t k_st_lds_offset = kBlockN;
        int32_t k_ld_lds_offset = kBlockN;
        int32_t v_ld_lds_offset = kBlockN;

        int32_t st_stage = 0;
        int32_t ld_stage = 0;

        auto k_block_tile = ck_tile::load_tile(k_dram_window_tail);
        ck_tile::store_tile(k_st_lds_window, k_block_tile);

        auto [i_page_block_k, k_dram_window] = k_page_block_navigator.make_tile_window(
            k_dram_window_lengths, {(n_block_max - 2) * kBlockN, 0}, Policy::MakeKDramTileDistribution());

        ck_tile::move_tile_window(k_st_lds_window, {kBlockN, 0});
        ++st_stage;

        constexpr int n_masking_steps = !Is_causal ? 1 : ck_tile::integer_divide_ceil(kBlockM, kBlockN) + 1;
        int masking_step = n_masking_steps;
        for (; i_block_n > n_block_min; --masking_step, --i_block_n)
        {
			ck_tile::clear_tile(s_acc);
            ck_tile::block_sync_lds();
            gemm_0(s_acc,
                   q,
                   k_ld_lds_window);

#ifdef ZZDebug
            ck_tile::block_sync_lds();
            if ((tidx == DEBUG_TID || tidx == 0) && blockIdx.x == 0 && blockIdx.y == 0 && (blockIdx.z == 0) && (i_block_n >= n_block_max - 2 || i_block_n <= n_block_min + 2))
            {
                auto debug_k_lds = ck_tile::load_tile(debug_k_ld_lds_window);
                const auto span_k2d = decltype(debug_k_lds)::get_distributed_spans();
                sweep_tile_span(span_k2d[I0], [&](auto idx0) {
                    sweep_tile_span(span_k2d[I1], [&](auto idx1) {
                        const auto i_j_idx = ck_tile::make_tuple(idx0, idx1);
                        const auto tile_idx = get_x_indices_from_distributed_indices(
                            debug_k_lds.get_tile_distribution(), i_j_idx);
                        auto row_id = tile_idx.at(I0);
                        auto col_id = tile_idx.at(I1);
                        printf("k_block_tile blockidx %d index [%d, %d] %f \n", i_block_n, row_id, col_id, ck_tile::type_convert<float>(k_block_tile[i_j_idx]));
                        printf("debug_k_lds blockidx %d index [%d, %d] %f \n", i_block_n, row_id, col_id, ck_tile::type_convert<float>(debug_k_lds[i_j_idx]));
                    });
                    printf("\n");
                });
            }
#endif

            auto k_block_tile = ck_tile::load_tile(k_dram_window);
            store_tile(k_st_lds_window, k_block_tile);
            i_page_block_k = k_page_block_navigator.move_tile_window(i_page_block_k, k_dram_window, {-kBlockN, 0});

            if (++st_stage % Traits::kStages == 0) 
                ck_tile::move_tile_window(k_st_lds_window, {-(Traits::kStages - 1) * kBlockN, 0});
                // k_st_lds_offset = -2 * kBlockN;
            else
                ck_tile::move_tile_window(k_st_lds_window, {kBlockN, 0});
                // k_st_lds_offset


#ifdef ZZDebug
            //TODO: s_acc is ready
            if (tidx == DEBUG_TID && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && (i_block_n >= n_block_max - 2 || i_block_n <= n_block_min + 2))
            {
                __syncthreads();
                const auto span_2d = decltype(s_acc)::get_distributed_spans();
                sweep_tile_span(span_2d[I0], [&](auto idx0) {
                    sweep_tile_span(span_2d[I1], [&](auto idx1) {
                        const auto i_j_idx = ck_tile::make_tuple(idx0, idx1);
                        const auto tile_idx = get_x_indices_from_distributed_indices(
                            s_acc.get_tile_distribution(), i_j_idx);

                        auto row_id = tile_idx.at(I0);
                        auto col_id = tile_idx.at(I1);

                        printf("blockid %d s_acc [%d, %d]: %f", i_block_n, row_id, col_id, ck_tile::type_convert<float>(s_acc[i_j_idx]));
                    });
                    printf("\n");
                });
            }
#endif

            const bool is_masking_step = masking_step > 0;
            const bool is_first_masking_step = masking_step == n_masking_steps;


            // if seq_len == 1, never need to add mask to s
            if (is_masking_step) {
                constexpr auto sacc_spans = decltype(s_acc)::get_distributed_spans();
                ck_tile::sweep_tile_span(sacc_spans[I0], [&](auto idx0) {
                    // constexpr auto i_idx = ck_tile::make_tuple(idx0);
                    ck_tile::sweep_tile_span(sacc_spans[I1], [&](auto idx1) {
                        constexpr auto i_j_idx = ck_tile::make_tuple(idx0, idx1);
                        const auto tile_idx = get_x_indices_from_distributed_indices(
                            s_acc.get_tile_distribution(), make_tuple(idx0, idx1));
                        auto row_id = tile_idx.at(ck_tile::number<0>{});
                        auto col_id = tile_idx.at(ck_tile::number<1>{});
                        if constexpr (!Is_causal)
                        {
                            if (col_id >= int(seqlen_k - i_block_n * kBlockN))
                                s_acc(i_j_idx) = -ck_tile::numeric<acc_t>::infinity();
// #ifdef ZZDebug
//                             if (tidx == DEBUG_TID && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && i_block_n == n_block_max - 1)
//                                 printf("s_acc [%d, %d]: %f", row_id, col_id, ck_tile::type_convert<float>(s_acc[i_j_idx]));
// #endif
                        }
                        else
                        {
                            int32_t col_limit_right = seqlen_k - 1 - i_block_n * kBlockN -
                                (params.size_s - 1 - (i_block_m * kBlockM + row_id)) / params.num_groups;
                            if (col_id > col_limit_right)
                                s_acc(i_j_idx) = -ck_tile::numeric<acc_t>::infinity();
#ifdef ZZDebug
                            if ((tidx == DEBUG_TID || tidx == 0) && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && (i_block_n >= n_block_max - 2 || i_block_n <= n_block_min + 2))
                                printf("blockIdx %d s_acc [%d, %d]: %f", i_block_n, row_id, col_id, ck_tile::type_convert<float>(s_acc[i_j_idx]));
#endif
                        }
                    });
                });
            }

            auto m_local = ck_tile::block_tile_reduce<acc_t>(
                s_acc,
                ck_tile::sequence<1>{},
                f_max,
                -ck_tile::numeric<acc_t>::infinity());
            block_tile_reduce_sync(m_local, f_max, ck_tile::bool_constant<false>{});

            const auto m_old = m;

            ck_tile::tile_elementwise_inout(
                [](auto& e0, auto e1, auto e2) { e0 = max(e1, e2); }, m, m_old, m_local);

            auto p_compute = ck_tile::make_static_distributed_tensor<acc_t>(
                s_acc.get_tile_distribution());

            constexpr auto p_spans = decltype(p_compute)::get_distributed_spans();
            ck_tile::sweep_tile_span(p_spans[I0], [&](auto idx0) {
                constexpr auto i_idx = ck_tile::make_tuple(idx0);
                auto row_max = params.scale_softmax_log2 * m[i_idx];
                ck_tile::sweep_tile_span(p_spans[I1], [&](auto idx1) {
                    constexpr auto i_j_idx = ck_tile::make_tuple(idx0, idx1);
                    p_compute(i_j_idx) = exp2(params.scale_softmax_log2 * s_acc[i_j_idx] - row_max);
#ifdef ZZDebug
                    const auto tile_idx = get_x_indices_from_distributed_indices(
                        p_compute.get_tile_distribution(), make_tuple(idx0, idx1));
                    auto row_id = tile_idx.at(ck_tile::number<0>{});
                    auto col_id = tile_idx.at(ck_tile::number<1>{});
                    if ((tidx == DEBUG_TID || tidx == 0) && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && (i_block_n >= n_block_max - 2 || i_block_n <= n_block_min + 2))
                        printf("blockIdx %d p_compute [%d, %d]: %f \n", i_block_n, row_id, col_id, ck_tile::type_convert<float>(p_compute[i_j_idx]));
#endif
                });
            });

            auto rowsum_p = ck_tile::block_tile_reduce<acc_t>(
                p_compute, ck_tile::sequence<1>{}, f_sum, acc_t{0});
            ck_tile::block_tile_reduce_sync(rowsum_p, f_sum, ck_tile::bool_constant<false>{});

            // l{j}, Oacc{j}
            if constexpr (Traits::TransposeC)
            {
                constexpr auto o_spans = decltype(o_acc)::get_distributed_spans();
                ck_tile::sweep_tile_span(o_spans[I0], [&](auto idx0) {
                    constexpr auto i_idx = ck_tile::make_tuple(idx0);
                    const auto tmp = exp2(params.scale_softmax_log2 * m_old[i_idx] - params.scale_softmax_log2 * m[i_idx]);
                    l(i_idx) = tmp * l[i_idx] + rowsum_p[i_idx];

#ifdef ZZDebug
                    const auto tile_idx = get_x_indices_from_distributed_indices(
                        l.get_tile_distribution(), make_tuple(idx0));
                    auto row_id = tile_idx.at(ck_tile::number<0>{});
                    if ((tidx == DEBUG_TID || tidx == 0) && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && (i_block_n >= n_block_max - 2 || i_block_n <= n_block_min + 2))
                    {
                        printf("l [%d]: %f \n", row_id, ck_tile::type_convert<float>(l[i_idx]));
                        printf("m [%d]: %f, \n", row_id, ck_tile::type_convert<float>(m[i_idx]));
                        printf("m_local [%d]: %f \n", row_id, ck_tile::type_convert<float>(m_local[i_idx]));
                        printf("tmp [%d]: %f \n", row_id, ck_tile::type_convert<float>(tmp));

                    }
#endif

                    ck_tile::sweep_tile_span(o_spans[I1], [&](auto idx1) {
                        constexpr auto i_j_idx = ck_tile::make_tuple(idx0, idx1);
                        o_acc(i_j_idx) = o_acc[i_j_idx] * tmp;
#ifdef ZZDebug
                        const auto tile_idx = get_x_indices_from_distributed_indices(o_acc.get_tile_distribution(), i_j_idx);
                        auto row_id = tile_idx.at(ck_tile::number<0>{});
                        auto col_id = tile_idx.at(ck_tile::number<1>{});
                        if ((tidx == DEBUG_TID || tidx == 0) && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && (i_block_n >= n_block_max - 2 || i_block_n <= n_block_min + 2))
                            printf("blockidx: %d, tid:%d o_acc_scaled [%d, %d]: %f \n", i_block_n, tidx, row_id, col_id, ck_tile::type_convert<float>(o_acc[i_j_idx]));
#endif
                    });
                });
            }
            else
            {
                constexpr auto o_spans = decltype(o_acc)::get_distributed_spans();
                ck_tile::sweep_tile_span(o_spans[I1], [&](auto idx1) {
                    constexpr auto j_idx = ck_tile::make_tuple(idx1);
                    const auto tmp = exp2(params.scale_softmax_log2 * m_old[j_idx] - params.scale_softmax_log2 * m[j_idx]);
                    l(j_idx) = tmp * l[j_idx] + rowsum_p[j_idx];

#ifdef ZZDebug
                    const auto tile_idx = get_x_indices_from_distributed_indices(
                        l.get_tile_distribution(), make_tuple(idx1));
                    auto row_id = tile_idx.at(ck_tile::number<0>{});
                    if ((tidx == DEBUG_TID || tidx == 0) && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && (i_block_n >= n_block_max - 2 || i_block_n <= n_block_min + 2))
                    {
                        printf("l [%d]: %f", row_id, ck_tile::type_convert<float>(l[j_idx]));
                        printf("\n");
                        printf("m [%d]: %f", row_id, ck_tile::type_convert<float>(m[j_idx]));
                        printf("\n");
                        printf("m_local [%d]: %f", row_id, ck_tile::type_convert<float>(m_local[j_idx]));
                        printf("\n");
                    }
#endif

                    ck_tile::sweep_tile_span(o_spans[I1], [&](auto idx0) {
                        constexpr auto i_j_idx = ck_tile::make_tuple(idx0, idx1);
// #ifdef ZZDebug
//                         const auto tile_idx = get_x_indices_from_distributed_indices(o_acc.get_tile_distribution(), i_j_idx);
//                         auto row_id = tile_idx.at(ck_tile::number<0>{});
//                         auto col_id = tile_idx.at(ck_tile::number<1>{});
//
//                         if ((tidx == DEBUG_TID || tidx == 0) && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && (i_block_n >= n_block_max - 2 || i_block_n <= n_block_min + 2))
//                             printf("blockIdx %d, tid:%d o_acc [%d, %d]: %f \t", i_block_n, tidx, row_id, col_id, ck_tile::type_convert<float>(o_acc[i_j_idx]));
// #endif
                        o_acc(i_j_idx) = o_acc[i_j_idx] * tmp;

// #ifdef ZZDebug
//                         if ((tidx == DEBUG_TID || tidx == 0) && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && (i_block_n >= n_block_max - 2 || i_block_n <= n_block_min + 2))
//                             printf("blockIdx %d, tid:%d o_acc_scaled [%d, %d]: %f \n", i_block_n, tidx, row_id, col_id, ck_tile::type_convert<float>(o_acc[i_j_idx]));
// #endif
                    });
                });
            }


// #ifdef ZZDebug
//             if ((tidx == DEBUG_TID || tidx == 0) && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && i_block_n == n_block_max - 1)
//             {
//                 auto v_tile = load_tile(debug_v_ld_lds_window);
//                 const auto span_2d = decltype(v_tile)::get_distributed_spans();
//                 sweep_tile_span(span_2d[I0], [&](auto idx0) {
//                     sweep_tile_span(span_2d[I1], [&](auto idx1) {
//                         const auto i_j_idx = ck_tile::make_tuple(idx0, idx1);
//                         const auto tile_idx = get_x_indices_from_distributed_indices(
//                             v_tile.get_tile_distribution(), i_j_idx);
//
//                         auto row_id = tile_idx.at(I0);
//                         auto col_id = tile_idx.at(I1);
//
//                         printf("v_tile [%d, %d]: %f", row_id, col_id, ck_tile::type_convert<float>(v_tile[i_j_idx]));
//                     });
//                     printf("\n");
//                 });
//             }
// #endif

        // auto shuffled_k_block_tile = make_static_distributed_tensor<KDataType>(
        //     Policy::template MakeShuffledKRegWriteBlockDescriptor<Problem>());

            auto p = ck_tile::cast_tile<scalar_t>(p_compute);
            if constexpr (Traits::GemmPVLds)
            {
                gemm_1(o_acc,
                       p,
                       v_ld_lds_window);
            }
            else
            {
                auto p_gemm_in = ck_tile::make_static_distributed_tensor<scalar_t>(
                    Policy::MakePShuffleTileDistribution(),
                    p.get_thread_buffer());
                auto v_tile = ck_tile::load_tile(v_ld_lds_window);

                __syncthreads();
                gemm_1(o_acc,
                       p_gemm_in,
                       v_tile);
            }

// #ifdef ZZDebug
// 			// __syncthreads();
// 			// const auto span_2d = decltype(p_gemm_in)::get_distributed_spans();
// 			// sweep_tile_span(span_2d[I0], [&](auto idx0) {
// 			// 	sweep_tile_span(span_2d[I1], [&](auto idx1) {
// 			// 		const auto i_j_idx = ck_tile::make_tuple(idx0, idx1);
// 			// 		const auto tile_idx_p = get_x_indices_from_distributed_indices(
// 			// 			p_gemm_in.get_tile_distribution(), i_j_idx);
// 			// 		const auto tile_idx_v = get_x_indices_from_distributed_indices(
// 			// 			v_tile.get_tile_distribution(), i_j_idx);
// 			//
// 			// 		if ((tidx == DEBUG_TID || tidx == 0) && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && (i_block_n >= n_block_max - 2 || i_block_n <= n_block_min + 2))
// 			// 		{
// 			// 			printf("blockIdx %d, p_gemm_in [%d, %d]: %f \n", i_block_n, tile_idx_p.at(I1), tile_idx_p.at(I0), ck_tile::type_convert<float>(p_gemm_in[i_j_idx]));
// 			// 			printf("blockIdx %d, v_tile [%d, %d]: %f \n", i_block_n, tile_idx_v.at(I1), tile_idx_v.at(I0), ck_tile::type_convert<float>(v_tile[i_j_idx]));
// 			// 		}
// 			//
// 			// 		if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && (i_block_n >= n_block_max - 2 || i_block_n <= n_block_min + 2))
// 			// 		{
//    //                      const auto tile_idx_o = get_x_indices_from_distributed_indices(
//    //                          o_acc.get_tile_distribution(), i_j_idx);
//    //                      printf("blockIdx %d, tid: %d o_acc [%d, %d]: %f\n", i_block_n, tidx, tile_idx_o.at(I1), tile_idx_o.at(I0), ck_tile::type_convert<float>(o_acc[i_j_idx]));
// 			// 		}
// 			//
// 			// 	});
// 			// 	// if ((tidx == DEBUG_TID || tidx == 0) && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && i_block_n == n_block_max - 1)
// 			// 	// 	printf("\n");
// 			// });
//             move_tile_window(debug_k_ld_lds_window, {k_ld_lds_offset, 0});
//             move_tile_window(debug_v_ld_lds_window, {0, v_ld_lds_offset});
// #endif

            if (++ld_stage % Traits::kStages == 0) 
            {
                ck_tile::move_tile_window(k_ld_lds_window, {-(Traits::kStages - 1) * kBlockN, 0});
                ck_tile::move_tile_window(v_ld_lds_window, {0, -(Traits::kStages - 1) * kBlockN});
            }
            else
            {
                ck_tile::move_tile_window(k_ld_lds_window, {kBlockN, 0});
                ck_tile::move_tile_window(v_ld_lds_window, {0, kBlockN});
            }

            // k_ld_lds_offset = -k_ld_lds_offset;
            // v_ld_lds_offset = -v_ld_lds_offset;
        }

        // tail block
        {
			ck_tile::clear_tile(s_acc);
            ck_tile::block_sync_lds();
            gemm_0(s_acc,
                   q,
                   k_ld_lds_window);

            auto m_local = ck_tile::block_tile_reduce<acc_t>(
                s_acc,
                ck_tile::sequence<1>{},
                f_max,
                -ck_tile::numeric<acc_t>::infinity());
            ck_tile::block_tile_reduce_sync(m_local, f_max, ck_tile::bool_constant<false>{});

            const auto m_old = m;

            ck_tile::tile_elementwise_inout(
                [](auto& e0, auto e1, auto e2) { e0 = max(e1, e2); }, m, m_old, m_local);

            auto p_compute = ck_tile::make_static_distributed_tensor<acc_t>(
                s_acc.get_tile_distribution());

            constexpr auto p_spans = decltype(p_compute)::get_distributed_spans();
            sweep_tile_span(p_spans[I0], [&](auto idx0) {
                constexpr auto i_idx = ck_tile::make_tuple(idx0);
                auto row_max = params.scale_softmax_log2 * m[i_idx];
                sweep_tile_span(p_spans[I1], [&](auto idx1) {
                    constexpr auto i_j_idx = ck_tile::make_tuple(idx0, idx1);
                    p_compute(i_j_idx) = exp2(params.scale_softmax_log2 * s_acc[i_j_idx] - row_max);
                });
            });

            auto rowsum_p = ck_tile::block_tile_reduce<acc_t>(
                p_compute, ck_tile::sequence<1>{}, f_sum, acc_t{0});
            ck_tile::block_tile_reduce_sync(rowsum_p, f_sum, ck_tile::bool_constant<false>{});

            // l{j}, Oacc{j}
            constexpr auto o_spans = decltype(o_acc)::get_distributed_spans();
            sweep_tile_span(o_spans[I0], [&](auto idx0) {
                constexpr auto i_idx = ck_tile::make_tuple(idx0);
                const auto tmp = exp2(params.scale_softmax_log2 * m_old[i_idx] - params.scale_softmax_log2 * m[i_idx]);
                l(i_idx) = tmp * l[i_idx] + rowsum_p[i_idx];
                sweep_tile_span(o_spans[I1], [&](auto idx1) {
                    constexpr auto i_j_idx = ck_tile::make_tuple(idx0, idx1);
                    o_acc(i_j_idx) *= tmp;
                });
            });


            auto p = ck_tile::cast_tile<scalar_t>(p_compute);
			auto p_gemm_in = ck_tile::make_static_distributed_tensor<scalar_t>(
                Policy::MakePShuffleTileDistribution(),
                p.get_thread_buffer());

            __syncthreads();
            if constexpr (Traits::GemmPVLds)
            {
                gemm_1(o_acc,
                       p,
                       v_ld_lds_window);
            }
            else
            {
                auto p_gemm_in = ck_tile::make_static_distributed_tensor<scalar_t>(
                    Policy::MakePShuffleTileDistribution(),
                    p.get_thread_buffer());
                auto v_tile = ck_tile::load_tile(v_ld_lds_window);

                __syncthreads();
                gemm_1(o_acc,
                       p_gemm_in,
                       v_tile);
            }
        }

        // Epilogue
        auto lse_acc = ck_tile::make_static_distributed_tensor<acc_t>(m.get_tile_distribution());
        constexpr auto lse_acc_spans = decltype(lse_acc)::get_distributed_spans();
        ck_tile::sweep_tile_span(lse_acc_spans[I0], [&](auto idx0) {
            constexpr auto i_idx = ck_tile::make_tuple(idx0);
            lse_acc(i_idx) = m[i_idx] * params.scale_softmax + log(l[i_idx]);
        });


        if (NoSplit)
        {
            const int32_t lse_offset = i_batch * params.size_s;
            auto lse_dram_window = Policy::MakeLSEDramTileWindow(
                reinterpret_cast<acc_t*>(params.p_softmax_lse) + lse_offset,
                params.size_s,
                i_m0);
            ck_tile::store_tile(lse_dram_window, lse_acc);
        }
        else
        {
            const int32_t split_offset = params.p_num_splits[i_batch];
            const int32_t lseacc_offset =
                ((split_offset + i_split) * params.size_h + i_nhead) *
                params.size_s + i_block_m * kBlockM;
            auto lseacc_dram_window = Policy::MakeLSEDramTileWindow(
                reinterpret_cast<acc_t*>(params.p_softmax_lseaccum) + lseacc_offset,
                params.size_s);
            ck_tile::store_tile(lseacc_dram_window, lse_acc);
        }

        __syncthreads();
        constexpr auto o_spans = decltype(o_acc)::get_distributed_spans();
        ck_tile::sweep_tile_span(o_spans[I0], [&](auto idx0) {
            constexpr auto i_idx = ck_tile::make_tuple(idx0);
            const auto tmp = [&]() {
                    return l[i_idx] == 0.f ? 0.f : 1 / l[i_idx];
            }();
            ck_tile::sweep_tile_span(o_spans[I1], [&](auto idx1) {
                constexpr auto i_j_idx = ck_tile::make_tuple(idx0, idx1);
                o_acc(i_j_idx) *= tmp;
            });
        });
        if (NoSplit)
        {
            const int32_t o_offset = i_batch * params.stride_b_o;
            auto o_dram_window = Policy::template MakeODramTileWindow<scalar_t>(
                reinterpret_cast<scalar_t*>(params.p_output) + o_offset,
                params.size_s,
                i_m0);
            ck_tile::store_tile(o_dram_window, ck_tile::cast_tile<scalar_t>(o_acc));
        }
        else
        {
            const int32_t split_offset = params.p_num_splits[i_batch];
            const int32_t oacc_offset =
                (((split_offset + i_split) * params.size_h + i_nhead) *
                params.size_s + i_block_m * kBlockM) * kSizeDV;
            auto o_acc_dram_window = Policy::template MakeODramTileWindow<acc_t>(
                reinterpret_cast<acc_t*>(params.p_output_accum) + oacc_offset,
                params.size_s);
            ck_tile::store_tile(o_acc_dram_window, o_acc);
        }
    }
}



template <typename Traits, typename scalar_t, typename acc_t, bool Is_causal>
void dispatch_fmla_fwd_splictkv(
    const FlashMlaFwdParams& params)
{
    // assert(params.page_block_size == Traits::kBlockN);
    const uint32_t num_m_block = static_cast<uint32_t>(ck_tile::integer_divide_ceil(params.size_s, Traits::kBlockM));
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    const dim3 grid {
        num_m_block,
        static_cast<uint32_t>(params.size_h),
        static_cast<uint32_t>(params.num_cu_parts)
    };

    constexpr int64_t smem_size = Traits::kLdsOffsetSum + Traits::kNumThreadsSoftmax * 4;
    auto kernel = &flash_fwd_splitkv_mla_kernel<Traits, scalar_t, acc_t, Is_causal>;
    kernel<<<grid, Traits::kNumThreads, smem_size, stream>>>(params);
}

template <typename Traits, int32_t kMaxSplits, typename scalar_t>
__global__ void kn_fmla_fwd_splictkv_combine(
    const FlashMlaFwdParams params)
{
    using Policy  = FlashMlaCombineKernelPolicy<Traits, scalar_t, float>;
    using index_t = int64_t;

    __shared__ float lds_lse_scale[kMaxSplits];

    const int32_t bidx = blockIdx.z;

    const int32_t split_offset = params.p_num_splits[bidx];
    const int32_t num_splits   = params.p_num_splits[bidx + 1] - split_offset;
    assert(num_splits <= kMaxSplits);

    if (num_splits > 1)
    {
        const int32_t lane_id          = ck_tile::get_lane_id();
        const int32_t hidx             = blockIdx.y;
        const int32_t sidx             = blockIdx.x;
        const int32_t hsidx            = hidx * params.size_s + sidx;
        const int32_t size_hs          = params.size_h * params.size_s;
        const index_t offset_lse_accum = split_offset * size_hs + hsidx;
        const index_t offset_lse       = bidx * size_hs + hsidx;

        if (ck_tile::get_warp_id() == 0)
        {
            const float* p_lse_accum = reinterpret_cast<float*>(params.p_softmax_lseaccum) + offset_lse_accum;
            float* p_lse             = reinterpret_cast<float*>(params.p_softmax_lse) + offset_lse;

            constexpr int32_t kNumLsePerThr = ck_tile::integer_divide_ceil(kMaxSplits, ck_tile::get_warp_size());
            float local_lse[kNumLsePerThr];

            // Load thread local LSE and get local max LSE
            float max_lse = -INFINITY;
            #pragma unroll
            for (int32_t i = 0; i < kNumLsePerThr; ++i)
            {
                const int32_t split_idx = i * ck_tile::get_warp_size() + lane_id;
                const float lse = (split_idx < num_splits) ? p_lse_accum[split_idx * size_hs] : -INFINITY;
                local_lse[i] = lse;
                max_lse = ck_tile::max(max_lse, lse);
            }

            // Get global max LSE
            #pragma unroll
            for (int32_t offset = ck_tile::get_warp_size() / 2; offset > 0; offset /= 2)
            {
                max_lse = ck_tile::max(max_lse, __shfl_xor(max_lse, offset));
            }

            // Get sum of LSE
            float sum_lse = 0.f;
            #pragma unroll
            for (int32_t i = 0; i < kNumLsePerThr; ++i)
            {
                sum_lse += expf(local_lse[i] - max_lse);
            }
            #pragma unroll
            for (int32_t offset = ck_tile::get_warp_size() / 2; offset > 0; offset /= 2)
            {
                sum_lse += __shfl_xor(sum_lse, offset);
            }

            // Get global LSE
            float global_lse = ((sum_lse == 0.f) || (sum_lse != sum_lse)) ? INFINITY : (logf(sum_lse) + max_lse);
            if (lane_id == 0)
            {
                *p_lse = global_lse;
            }

            // Write LSE to LDS
            #pragma unroll
            for (int32_t i = 0; i < kNumLsePerThr; ++i)
            {
                const int32_t split_idx = i * ck_tile::get_warp_size() + lane_id;
                if (split_idx < num_splits)
                {
                    lds_lse_scale[split_idx] = expf(local_lse[i] - global_lse);
                }
            }
        }

        __builtin_amdgcn_sched_barrier(0);
        ck_tile::block_sync_lds();

        static_assert(Traits::kSizeDV % Traits::kNumThreadsCombine == 0);

        auto oaccu_window =
            Policy::MakeOaccuTileWindow(params.p_output_accum, hsidx, size_hs, split_offset, num_splits);

        auto reg_out = ck_tile::make_static_distributed_tensor<float>(
            decltype(ck_tile::load_tile(oaccu_window))::get_tile_distribution());
        ck_tile::set_tile(reg_out, 0.f);

        for (int32_t split_idx = 0; split_idx < num_splits; ++split_idx)
        {
            const float lse_scale = lds_lse_scale[split_idx];
            auto oaccu = ck_tile::load_tile(oaccu_window);
            ck_tile::sweep_tile(oaccu, [&](auto idx) {
                reg_out(idx) += lse_scale * oaccu(idx);
                // const auto tile_idx = get_x_indices_from_distributed_indices(oaccu.get_tile_distribution(), idx);
                // if (blockIdx.x == 1 && blockIdx.y == 0 && blockIdx.z == 1) {
                //     printf("split_idx %d tid:%d batch_idx:%d o_acc [%d, %d]: %f scale: %f\n", split_idx, threadIdx.x, blockIdx.z, tile_idx.at(ck_tile::number<0>{}), tile_idx.at(ck_tile::number<1>{}), ck_tile::type_convert<float>(oaccu[idx]), lse_scale);
                // }
            });
            ck_tile::move_tile_window(oaccu_window, {size_hs, 0});
            // ck_tile::move_tile_window(oaccu_window, {1, 0});
        }

        auto dram_out = Policy::MakeOutputTileWindow(params.p_output,
                                                     bidx * params.stride_b_o,
                                                     hidx * params.stride_h_o,
                                                     sidx * params.stride_s_o);
        ck_tile::store_tile(dram_out, ck_tile::cast_tile<scalar_t>(reg_out));
    }
}

template <typename Traits, typename scalar_t>
void dispatch_fmla_fwd_splictkv_combine(
    const FlashMlaFwdParams& params)
{
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const dim3 grid  = dim3(params.size_s, params.size_h, params.size_b);
    const dim3 block = dim3(Traits::kNumThreadsCombine);

    if (params.num_cu_parts <= 1) return;

    if (params.num_cu_parts <= 32)
    {
        kn_fmla_fwd_splictkv_combine<Traits, 32, scalar_t><<<grid, block, 0, stream>>>(params);
    }
    else if (params.num_cu_parts <= 64)
    {
        kn_fmla_fwd_splictkv_combine<Traits, 64, scalar_t><<<grid, block, 0, stream>>>(params);
    }
    else if (params.num_cu_parts <= 96)
    {
        kn_fmla_fwd_splictkv_combine<Traits, 96, scalar_t><<<grid, block, 0, stream>>>(params);
    }
    else if (params.num_cu_parts <= 128)
    {
        kn_fmla_fwd_splictkv_combine<Traits, 128, scalar_t><<<grid, block, 0, stream>>>(params);
    }
    else
    {
        // TORCH_CHECK(false, "fmla_fwd_splictkv_combine cannot support the specified num_cu_parts ",
        //                    toString(params.num_cu_parts), ".");
        assert(false);
    }
}

#define DISPATCH_FMLA_TYPES(TYPE, IS_CAUSAL, NAME, ...) \
    switch ((TYPE))                                     \
    {                                                   \
        case at::ScalarType::BFloat16:                           \
        {                                               \
            using scalar_t = ck_tile::bf16_t;           \
            if ((IS_CAUSAL))                            \
            {                                           \
                constexpr bool Is_causal = true;        \
                __VA_ARGS__;                            \
            }                                           \
            else                                        \
            {                                           \
                constexpr bool Is_causal = false;       \
                __VA_ARGS__;                            \
            }                                           \
            break;                                      \
        }                                               \
        case at::ScalarType::Half:                               \
        {                                               \
            using scalar_t = ck_tile::fp16_t;           \
            if ((IS_CAUSAL))                            \
            {                                           \
                constexpr bool Is_causal = true;        \
                __VA_ARGS__;                            \
            }                                           \
            else                                        \
            {                                           \
                constexpr bool Is_causal = false;       \
                __VA_ARGS__;                            \
            }                                           \
            break;                                      \
        }                                               \
        default:                                        \
            TORCH_CHECK(false, NAME " does't support ", \
                        toString((TYPE)), ".");         \
    }

std::vector<torch::Tensor> flash_mla_fwd_with_kvcache_impl(
    torch::Tensor& query,
    const torch::Tensor& key_cache,
    const torch::Tensor& value_cache,
    const int32_t        head_size_v,
    const torch::Tensor& cache_seqlens,
    const torch::Tensor& block_table,
    const float          softmax_scale,
    const bool           is_causal,
    const torch::Tensor& tile_scheduler_metadata,
    const torch::Tensor& num_splits)
{
    using Traits = FlashMlaKernelTraitsInstance;

    torch::Tensor vcache = value_cache.data_ptr() ? value_cache : key_cache;

    auto opts = query.options();

    const int32_t batch_size = query.size(0);
    const int32_t seqlen_q_ori = query.size(1);
    const int32_t num_heads_q_ori = query.size(2);

    const int32_t head_size = query.size(3);


    const int32_t num_blocks = key_cache.size(0);
    const int32_t page_block_size = key_cache.size(1);
    const int32_t num_heads_k = key_cache.size(2);

    const int32_t num_groups = num_heads_q_ori / num_heads_k;
    const int32_t seqlen_q = seqlen_q_ori * num_groups;
    const int32_t num_heads = num_heads_k;
    const int32_t num_cu_parts = tile_scheduler_metadata.size(0);

    query = query.view({batch_size, seqlen_q_ori, num_heads_k, num_groups, head_size}).transpose(2, 3)
                .reshape({batch_size, seqlen_q, num_heads, head_size});

    // CHECK_SHAPE(query, batch_size, seqlen_q, num_heads, head_size);
    // CHECK_SHAPE(key_cache, num_blocks, page_block_size, num_heads, head_size);

    auto output = torch::zeros({batch_size, seqlen_q, num_heads, head_size_v}, opts);
    auto softmax_lse = torch::zeros({batch_size, num_heads, seqlen_q}, opts.dtype(torch::kFloat32));

    auto softmax_lseaccum = torch::zeros({batch_size + num_cu_parts, num_heads, seqlen_q}, opts.dtype(torch::kFloat32));
    auto output_accum = torch::zeros({batch_size + num_cu_parts, num_heads, seqlen_q, head_size_v}, opts.dtype(torch::kFloat32));

    FlashMlaFwdParams params = {};
    params.p_cu_seqlens_k            = cache_seqlens.data_ptr<int32_t>();
    params.p_block_table             = block_table.data_ptr<int32_t>();
    params.p_tile_scheduler_metadata = tile_scheduler_metadata.data_ptr<int32_t>();
    params.p_num_splits              = num_splits.data_ptr<int32_t>();

    params.p_query            = query.data_ptr();
    params.p_key              = key_cache.data_ptr();
    params.p_value            = vcache.data_ptr();
    params.p_output           = output.data_ptr();
    params.p_softmax_lse      = softmax_lse.data_ptr();
    params.p_softmax_lseaccum = softmax_lseaccum.data_ptr();
    params.p_output_accum     = output_accum.data_ptr();

    params.size_b                   = batch_size;
    params.size_s                   = seqlen_q;
    params.size_h                   = num_heads;
    params.hq_hk_ratio              = num_heads / num_heads_k;
    params.num_groups               = num_groups;
    params.num_cu_parts             = tile_scheduler_metadata.size(0);
    params.block_table_batch_stride = block_table.stride(0);
    params.page_block_size          = page_block_size;
    params.scale_softmax            = softmax_scale;
    params.scale_softmax_log2       = float(softmax_scale * M_LOG2E);
    params.is_causal                = is_causal;

    params.stride_b_q = query.stride(0);
    params.stride_s_q = query.stride(1);
    params.stride_h_q = query.stride(2);
    params.stride_b_k = key_cache.stride(0);
    params.stride_s_k = key_cache.stride(1);
    params.stride_h_k = key_cache.stride(2);
    params.stride_b_v = vcache.stride(0);
    params.stride_s_v = vcache.stride(1);
    params.stride_h_v = vcache.stride(2);
    params.stride_b_o = output.stride(0);
    params.stride_s_o = output.stride(1);
    params.stride_h_o = output.stride(2);

	using acc_t = float;

    dispatch_fmla_fwd_splictkv<Traits, ck_tile::fp16_t, float, true>(params);
    dispatch_fmla_fwd_splictkv_combine<Traits, ck_tile::fp16_t>(params);
    DISPATCH_FMLA_TYPES(
        query.scalar_type(),
        is_causal,
        "fmla_fwd",
        [&](){
            // dispatch_fmla_fwd_splictkv<Traits, scalar_t, acc_t, Is_causal>(params);
            // dispatch_fmla_fwd_splictkv_combine<Traits, scalar_t>(params);
        }();
    );
    output = output.view({batch_size, seqlen_q_ori, num_groups, num_heads_k, head_size_v}).transpose(2, 3)
            .reshape({batch_size, seqlen_q_ori, num_heads_q_ori, head_size_v});
    softmax_lse = softmax_lse.view({batch_size, num_heads_k, seqlen_q_ori, num_groups}).transpose(2, 3)
            .reshape({batch_size, num_heads_q_ori, seqlen_q_ori});
    return {output, softmax_lse};
}
