#include "hip/hip_runtime.h"
// SPDX-License-Identifier: MIT
// Copyright (C) 2025, Advanced Micro Devices, Inc. All rights reserved.

#include <ck_tile/ops/fmha.hpp>
#include <ck_tile/core/tensor/tile_scatter_gather.hpp>
#include "ck_fwd_kernels_params.hpp"
#include "ck_fwd_prefill_kernels_pipelines.hpp"

// =====================================================================================================================
// Kernel Functions
//

template <typename Traits>
CK_TILE_DEVICE static auto GetTileIndex(const int32_t num_splits)
{
    const auto f = [](int32_t dividend, int32_t divisor) {
        int32_t quotient = dividend / divisor;
        int32_t modulus  = dividend - quotient * divisor;
        return ck_tile::make_tuple(quotient, modulus);
    };

    const auto [mid, split_id] = f(blockIdx.x, num_splits);
    const int32_t hid          = blockIdx.y;
    const int32_t bid          = blockIdx.z;

    return ck_tile::make_tuple(mid, split_id, hid, bid);
}

template <typename Policy, int32_t HiddenDim, typename scalar_t = typename Policy::InOutType>
CK_TILE_DEVICE static auto MakeQDram(const scalar_t* p_data,
                                     const int32_t size_s,
                                     const int32_t stride_s,
                                     const int32_t hq_hk_ratio,
                                     const int32_t stride_h)
{
    using Traits = typename Policy::Traits;

    const auto q_dram_naive = [&] {
        if constexpr(Traits::kEnableXqa)
        {
            const auto view = ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s, hq_hk_ratio, HiddenDim),
                ck_tile::make_tuple(stride_s, stride_h, 1),
                ck_tile::number<Policy::GetAlignmentQ()>{},
                ck_tile::number<1>{});
            return ck_tile::transform_tensor_view(
                view,
                ck_tile::make_tuple(
                    ck_tile::make_merge_transform(ck_tile::make_tuple(size_s, hq_hk_ratio)),
                    ck_tile::make_pass_through_transform(HiddenDim)),
                ck_tile::make_tuple(ck_tile::sequence<0, 1>{}, ck_tile::sequence<2>{}),
                ck_tile::make_tuple(ck_tile::sequence<0>{}, ck_tile::sequence<1>{}));
        }
        else
        {
            return ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s, HiddenDim),
                ck_tile::make_tuple(stride_s, 1),
                ck_tile::number<Policy::GetAlignmentQ()>{},
                ck_tile::number<1>{});
        }
    }();

    return ck_tile::pad_tensor_view(
        q_dram_naive,
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{}, ck_tile::number<Traits::kBlockK0>{}),
        ck_tile::sequence<false, Traits::kPadHeadDimQ>{});
}

template <typename Policy, int32_t HiddenDim, typename scalar_t = typename Policy::InOutType>
CK_TILE_DEVICE static auto MakeKDram(
    const scalar_t* p_data,
    const int32_t   height,
    const int32_t   stride_s)
{
    using Traits = typename Policy::Traits;

    const auto k_dram_naive = ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
        p_data, // will update this pointer if using paged-kvcache
        ck_tile::make_tuple(height, HiddenDim),
        ck_tile::make_tuple(stride_s, 1),
        ck_tile::number<Policy::GetAlignmentK()>{},
        ck_tile::number<1>{});

    return ck_tile::pad_tensor_view(
        k_dram_naive,
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockN0>{}, ck_tile::number<Traits::kBlockK0>{}),
        ck_tile::sequence<true, Traits::kPadHeadDimQ>{});
}

template <typename Policy, typename scalar_t = typename Policy::InOutType>
CK_TILE_DEVICE static auto MakeVDram(
    const scalar_t* p_data,
    const int32_t   length,
    const int32_t   stride_s)
{
    using Traits = typename Policy::Traits;

    // Assuming Value is row-major just like Key.
    const auto v_dram_naive = ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
        p_data, // will update this pointer if using paged-kvcache
        ck_tile::make_tuple(length, Traits::kSizeDV),
        ck_tile::make_tuple(stride_s, 1),
        ck_tile::number<Policy::GetAlignmentV()>{},
        ck_tile::number<1>{});

    const auto v_dram_transposed = ck_tile::transform_tensor_view(
        v_dram_naive,
        ck_tile::make_tuple(ck_tile::make_pass_through_transform(Traits::kSizeDV),
                            ck_tile::make_pass_through_transform(length)),
        ck_tile::make_tuple(ck_tile::sequence<1>{}, ck_tile::sequence<0>{}),
        ck_tile::make_tuple(ck_tile::sequence<0>{}, ck_tile::sequence<1>{}));

    return ck_tile::pad_tensor_view(
        v_dram_transposed,
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockN1>{},
                            ck_tile::number<Traits::kBlockK1>{}),
        ck_tile::sequence<Traits::kPadHeadDimV, Traits::kPadSeqLenK>{});
}

template <typename Policy, typename Lengths, typename scalar_t>
CK_TILE_DEVICE static auto MakeLseAccDram(scalar_t* p_data,
                                          const Lengths& window_lengths,
                                          const int32_t size_s,
                                          const int32_t hq_hk_ratio,
                                          const int32_t stride_h)
{
    using Traits = typename Policy::Traits;

    const auto lse_acc_dram_naive = [&] {
        if constexpr(Traits::kEnableXqa)
        {
            // transpose + merge: (hq_hk_ratio, seqlen_q) -> (seqlenq * hq_hk_ratio)
            const auto view = ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(hq_hk_ratio, size_s),
                ck_tile::make_tuple(stride_h, 1),
                ck_tile::number<1>{},
                ck_tile::number<1>{});
            return ck_tile::transform_tensor_view(
                view,
                ck_tile::make_tuple(
                    ck_tile::make_merge_transform(ck_tile::make_tuple(size_s, hq_hk_ratio))),
                ck_tile::make_tuple(ck_tile::sequence<1, 0>{}),
                ck_tile::make_tuple(ck_tile::sequence<0>{}));
        }
        else
        {
            return ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s),
                ck_tile::make_tuple(1),
                ck_tile::number<1>{},
                ck_tile::number<1>{});
        }
    }();

    return ck_tile::pad_tensor_view(
        lse_acc_dram_naive,
        window_lengths,
        ck_tile::sequence<Traits::kPadSeqLenQ>{});
}

template <typename Policy, typename scalar_t>
CK_TILE_DEVICE static auto MakeOutAccDram(scalar_t* p_data,
                                          const int32_t size_s,
                                          const int32_t stride_s,
                                          const int32_t hq_hk_ratio,
                                          const int32_t stride_h)
{
    using Traits = typename Policy::Traits;

    const auto o_acc_dram_naive = [&] {
        if constexpr(Traits::kEnableXqa)
        {
            // merge: (seqlen_q, hq_hk_ratio, headdim) -> (seqlen_q*hq_hk_ratio, headdim)
            const auto view = ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s, hq_hk_ratio, Traits::kSizeDV),
                ck_tile::make_tuple(stride_s, stride_h, 1),
                ck_tile::number<Policy::GetAlignmentOacc()>{},
                ck_tile::number<1>{});
            return ck_tile::transform_tensor_view(
                view,
                ck_tile::make_tuple(
                    ck_tile::make_merge_transform(ck_tile::make_tuple(size_s, hq_hk_ratio)),
                    ck_tile::make_pass_through_transform(Traits::kSizeDV)),
                ck_tile::make_tuple(ck_tile::sequence<0, 1>{}, ck_tile::sequence<2>{}),
                ck_tile::make_tuple(ck_tile::sequence<0>{}, ck_tile::sequence<1>{}));
        }
        else
        {
            return ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s, Traits::kSizeDV),
                ck_tile::make_tuple(stride_s, 1),
                ck_tile::number<Policy::GetAlignmentOacc()>{},
                ck_tile::number<1>{});
        }
    }();

    return ck_tile::pad_tensor_view(
        o_acc_dram_naive,
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{}, ck_tile::number<Traits::kBlockN1>{}),
        ck_tile::sequence<Traits::kPadSeqLenQ, Traits::kPadHeadDimV>{});
}

template <typename Policy, typename Lengths, typename scalar_t>
CK_TILE_DEVICE static auto MakeLseDram(scalar_t* p_data,
                                       const Lengths& window_lenghts,
                                       const int32_t size_s,
                                       const int32_t hq_hk_ratio,
                                       const int32_t stride_h)
{
    using Traits = typename Policy::Traits;

    const auto lse_dram_naive = [&] {
        if constexpr(Traits::kEnableXqa)
        {
            // transpose + merge: (hq_hk_ratio, seqlen_q) -> (seqlenq * hq_hk_ratio)
            const auto view = ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(hq_hk_ratio, size_s),
                ck_tile::make_tuple(stride_h, 1),
                ck_tile::number<Policy::GetAlignmentLse()>{},
                ck_tile::number<1>{});
            return ck_tile::transform_tensor_view(
                view,
                ck_tile::make_tuple(
                    ck_tile::make_merge_transform(ck_tile::make_tuple(size_s, hq_hk_ratio))),
                ck_tile::make_tuple(ck_tile::sequence<1, 0>{}),
                ck_tile::make_tuple(ck_tile::sequence<0>{}));
        }
        else
        {

            return ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s),
                ck_tile::make_tuple(1),
                ck_tile::number<Policy::GetAlignmentLse()>{},
                ck_tile::number<1>{});
        }
    }();

    return ck_tile::pad_tensor_view(
        lse_dram_naive, window_lenghts, ck_tile::sequence<Traits::kPadSeqLenQ>{});
}

template <typename Policy, typename scalar_t>
CK_TILE_DEVICE static auto MakeOutDram(scalar_t* p_data,
                                       const int32_t size_s,
                                       const int32_t stride_s,
                                       const int32_t hq_hk_ratio,
                                       const int32_t stride_h)
{
    using Traits = typename Policy::Traits;

    const auto o_dram_naive = [&] {
        if constexpr(Traits::kEnableXqa)
        {
            // merge: (seqlen_q, hq_hk_ratio, headdim) -> (seqlen_q * hq_hk_ratio, headdim)
            const auto view = ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s, hq_hk_ratio, Traits::kSizeDV),
                ck_tile::make_tuple(stride_s, stride_h, 1),
                ck_tile::number<Policy::GetAlignmentO()>{},
                ck_tile::number<1>{});
            return ck_tile::transform_tensor_view(
                view,
                ck_tile::make_tuple(
                    ck_tile::make_merge_transform(ck_tile::make_tuple(size_s, hq_hk_ratio)),
                    ck_tile::make_pass_through_transform(Traits::kSizeDV)),
                ck_tile::make_tuple(ck_tile::sequence<0, 1>{}, ck_tile::sequence<2>{}),
                ck_tile::make_tuple(ck_tile::sequence<0>{}, ck_tile::sequence<1>{}));
        }
        else
        {
            return ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
                p_data,
                ck_tile::make_tuple(size_s, Traits::kSizeDV),
                ck_tile::make_tuple(stride_s, 1),
                ck_tile::number<Policy::GetAlignmentO()>{},
                ck_tile::number<1>{});
        }
    }();

    return ck_tile::pad_tensor_view(
        o_dram_naive,
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{}, ck_tile::number<Traits::kBlockN1>{}),
        ck_tile::sequence<Traits::kPadSeqLenQ, Traits::kPadHeadDimV>{});
}
// =====================================================================================================================
// Kernel Entry
//

template <typename Traits, typename scalar_t, typename acc_t, typename out_t, bool kIsCausal, bool kIsRopeSeparate, bool kDoSplit>
__launch_bounds__(Traits::kNumThreads, Traits::kWaveOccupancy)
__global__ void kn_ck_mla_fwd_splictkv_prefill(
    const CkMlaPrefillFwdParams params)
{
    using Policy = CkMlaPrefillPolicy<Traits, scalar_t, acc_t>;
    constexpr auto HiddenDimSize = kIsRopeSeparate ? Traits::kSizeNope : Traits::kSizeD;

    // allocate LDS
    __shared__ uint8_t p_smem[Policy::GetSmemSize()];

    const auto [tile_m_id, split_id, hqid, bid] =
        kDoSplit ? GetTileIndex<Traits>(params.num_splits) : GetTileIndex<Traits>(1);
    const auto    hqid_xqa = Traits::kEnableXqa ? hqid * params.hq_hk_ratio : hqid;
    const auto    hkid     = hqid_xqa / params.hq_hk_ratio;
    const int32_t mid      = __builtin_amdgcn_readfirstlane(tile_m_id * Traits::kBlockM);

    const int32_t seqlen_qo    = params.p_seqlens_qo[bid];
    const int32_t seqlen_qo_pk = Traits::kEnableXqa ? seqlen_qo * params.hq_hk_ratio : seqlen_qo;
    const int32_t seqlen_kv    = params.p_seqlens_kv[bid];

    if (mid >= seqlen_qo_pk)
    {
        return;
    }

    // Define causal mask
    using Mask = std::conditional_t<Traits::kEnableXqa,
                                    ck_tile::SimplifiedRatioAttentionMask<kIsCausal>,
                                    ck_tile::SimplifiedGenericAttentionMask<kIsCausal>>;
    Mask mask = [&] {
        if constexpr(kIsCausal)
        {
            if constexpr(Traits::kEnableXqa)
            {
                return Mask{seqlen_qo,
                            seqlen_kv - seqlen_qo + 1,
                            seqlen_qo_pk,
                            seqlen_kv,
                            params.mask_y_ratio_mdiv};
            }
            else
            {
                return Mask{seqlen_qo,
                            seqlen_kv - seqlen_qo + 1,
                            seqlen_qo,
                            seqlen_kv};
            }
        }
        else
        {
            return Mask{seqlen_qo_pk, seqlen_kv};
        }
    }();

    constexpr auto dram_nope_window_length_k = Traits::kKVLoadOnce
                                                   ? ck_tile::number<Traits::kSizeNope>{}
                                                   : ck_tile::number<Traits::kBlockK0>{};
    constexpr auto dram_rope_window_length_k = Traits::kKVLoadOnce
                                                   ? ck_tile::number<Traits::kSizeRope>{}
                                                   : ck_tile::number<Traits::kBlockK0>{};

    constexpr auto q_nope_dram_window_lengths =
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{}, dram_nope_window_length_k);
    constexpr auto q_rope_dram_window_lengths =
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{}, dram_rope_window_length_k);
    constexpr auto k_nope_dram_window_lengths =
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockN0>{}, dram_nope_window_length_k);
    constexpr auto k_rope_dram_window_lengths =
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockN0>{}, dram_rope_window_length_k);
    constexpr auto v_dram_window_lengths =
        ck_tile::make_tuple(ck_tile::number<Traits::kBlockN1>{}, ck_tile::number<Traits::kBlockK1>{});

    const scalar_t* p_query_nope = reinterpret_cast<const scalar_t*>(params.p_query_nope) +
                              int64_t(hqid_xqa) * params.stride_h_q_nope +   // head offset
                              int64_t(bid) * params.stride_b_q_nope;     // batch offset
    const scalar_t* p_key_nope   = reinterpret_cast<const scalar_t*>(params.p_key_nope) +
                              int64_t(hkid) * params.stride_h_k_nope;    // head offset
    const scalar_t* p_value = reinterpret_cast<const scalar_t*>(params.p_value) +
                              int64_t(hkid) * params.stride_h_v;    // head offset
    const int32_t*  p_block_table = params.p_block_table +
                                    int64_t(bid) * params.block_table_batch_stride; // batch offset

    const int32_t kv_cache_width = params.num_page_blocks * params.page_block_size;

    const auto q_dram_nope = MakeQDram<Policy, HiddenDimSize>(
        p_query_nope, seqlen_qo, params.stride_s_q_nope, params.hq_hk_ratio, params.stride_h_q_nope);

    const auto k_dram_nope = MakeKDram<Policy, HiddenDimSize>(p_key_nope,   kv_cache_width, params.stride_s_k_nope);
    const auto v_dram = MakeVDram<Policy>(p_value, kv_cache_width, params.stride_s_v);    

    auto q_dram_window_nope = ck_tile::make_tile_window(q_dram_nope, q_nope_dram_window_lengths, {mid, 0});
    auto q_dram_window_rope = [&] {
        if constexpr(kIsRopeSeparate)
        {
            const scalar_t* p_query_rope =
                reinterpret_cast<const scalar_t*>(params.p_query_rope) +
                int64_t(hqid_xqa) * params.stride_h_q_rope + // head offset
                int64_t(bid) * params.stride_b_q_rope;       // batch offset
            const auto q_dram_rope = MakeQDram<Policy, Traits::kSizeRope>(p_query_rope,
                                                                          seqlen_qo,
                                                                          params.stride_s_q_rope,
                                                                          params.hq_hk_ratio,
                                                                          params.stride_h_q_rope);
            return ck_tile::make_tile_window(q_dram_rope, q_rope_dram_window_lengths, {mid, 0});
        }
        else
        {
            return ck_tile::make_tile_window(q_dram_nope, q_rope_dram_window_lengths, {mid, Traits::kSizeNope});
        }
    }();

    auto k_dram_window_nope = ck_tile::make_tile_window(k_dram_nope, k_nope_dram_window_lengths, {0, 0});
    auto k_dram_window_rope = [&] {
        if constexpr(kIsRopeSeparate)
        {
            const scalar_t* p_key_rope = reinterpret_cast<const scalar_t*>(params.p_key_rope) +
                                         int64_t(hkid) * params.stride_h_k_rope; // head offset
            const auto k_dram_rope = MakeKDram<Policy, Traits::kSizeRope>(
                p_key_rope, kv_cache_width, params.stride_s_k_rope);
            return ck_tile::make_tile_window(k_dram_rope, k_rope_dram_window_lengths, {0, 0});
        }
        else
        {
            return ck_tile::make_tile_window(k_dram_nope, k_rope_dram_window_lengths, {0, Traits::kSizeNope});
        }
    }();

    auto v_dram_window = ck_tile::make_tile_window(v_dram, v_dram_window_lengths, {0, 0});

    const auto real_stride_s_k_rope = kIsRopeSeparate ? params.stride_s_k_rope : params.stride_s_k_nope;
    if constexpr (kDoSplit)
    {
        acc_t* p_lse_acc = reinterpret_cast<acc_t*>(params.p_softmax_lseaccum) +
                           int64_t(hqid_xqa) * params.stride_h_lseacc +     // head offset
                           int64_t(bid) * params.stride_b_lseacc +      // batch offset
                           int64_t(split_id) * params.stride_sp_lseacc; // split offset
        out_t* p_out_acc = reinterpret_cast<out_t*>(params.p_output_accum) +
                           int64_t(hqid_xqa) * params.stride_h_oacc +      // head offset
                           int64_t(bid) * params.stride_b_oacc +       // batch offset
                           int64_t(split_id) * params.stride_sp_oacc;  // split offset

        auto lse_acc_dram_window_lengths =
            ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{});
        auto out_acc_dram_window_lengths =
            ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{}, ck_tile::number<Traits::kBlockN1>{});

        const auto lse_acc_dram = MakeLseAccDram<Policy>(p_lse_acc,
                                                         lse_acc_dram_window_lengths,
                                                         seqlen_qo,
                                                         params.hq_hk_ratio,
                                                         params.stride_h_lseacc);
        const auto out_acc_dram = MakeOutAccDram<Policy>(p_out_acc,
                                                         seqlen_qo,
                                                         params.stride_s_oacc,
                                                         params.hq_hk_ratio,
                                                         params.stride_h_oacc);

        auto lse_acc_dram_window =
            ck_tile::make_tile_window(lse_acc_dram, lse_acc_dram_window_lengths, {mid});
        auto out_acc_dram_window =
            ck_tile::make_tile_window(out_acc_dram, out_acc_dram_window_lengths, {mid, 0});


        if constexpr (!Traits::kKVLoadOnce) {
            kn_ck_mla_fwd_splitkv_prefill_tile<Traits, scalar_t, acc_t, out_t, kIsRopeSeparate>(
                q_dram_window_nope,
                q_dram_window_rope,
                k_dram_window_nope,
                k_dram_window_rope,
                v_dram_window,
                lse_acc_dram_window,
                out_acc_dram_window,
                p_block_table,
                __builtin_amdgcn_readfirstlane(params.page_block_size),
                __builtin_amdgcn_readfirstlane(params.stride_s_k_nope),
                __builtin_amdgcn_readfirstlane(real_stride_s_k_rope),
                __builtin_amdgcn_readfirstlane(params.stride_s_v),
                seqlen_kv,
                params.num_splits,
                split_id,
                mask,
#if CK_MLA_FWD_FAST_EXP2
                static_cast<float>(params.scale_softmax * ck_tile::log2e_v<>),
#else
                params.scale_softmax,
#endif
                p_smem);
        }
        else
        {
            kn_ck_mla_fwd_splitkv_prefill_load_once_tile<Traits, scalar_t, acc_t, out_t>(
                q_dram_window_nope,
                q_dram_window_rope,
                k_dram_window_nope,
                k_dram_window_rope,
                lse_acc_dram_window,
                out_acc_dram_window,
                p_block_table,
                __builtin_amdgcn_readfirstlane(params.page_block_size),
                __builtin_amdgcn_readfirstlane(params.stride_s_k_nope),
                __builtin_amdgcn_readfirstlane(real_stride_s_k_rope),
                __builtin_amdgcn_readfirstlane(params.stride_s_v),
                seqlen_kv,
                params.num_splits,
                split_id,
                mask,
#if CK_MLA_FWD_FAST_EXP2
                static_cast<float>(params.scale_softmax * ck_tile::log2e_v<>),
#else
                params.scale_softmax,
#endif
                p_smem);
        }
    }
    else
    {
        // Assuming lse is in shape [b, h, s] and is contiguous
        acc_t* p_lse =
            reinterpret_cast<acc_t*>(params.p_softmax_lse) +
            (int64_t(bid) * params.size_h_ori + hqid_xqa) * params.max_size_s_ori; // batch+head offset
        out_t* p_out = reinterpret_cast<out_t*>(params.p_output) +
                       int64_t(hqid_xqa) * params.stride_h_o + // head offset
                       int64_t(bid) * params.stride_b_o;       // batch offset

        auto lse_dram_window_lengths =
            ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{});
        auto out_dram_window_lengths =
            ck_tile::make_tuple(ck_tile::number<Traits::kBlockM>{}, ck_tile::number<Traits::kBlockN1>{});

        const auto lse_dram = MakeLseDram<Policy>(p_lse,
                                                  lse_dram_window_lengths,
                                                  seqlen_qo,
                                                  params.hq_hk_ratio,
                                                  params.stride_h_lse);
        const auto out_dram = MakeOutDram<Policy>(
            p_out, seqlen_qo, params.stride_s_o, params.hq_hk_ratio, params.stride_h_o);

        auto lse_dram_window =
            ck_tile::make_tile_window(lse_dram, lse_dram_window_lengths, {mid});
        auto out_dram_window =
            ck_tile::make_tile_window(out_dram, out_dram_window_lengths, {mid, 0});

        if constexpr (!Traits::kKVLoadOnce)
        {
            kn_ck_mla_fwd_splitkv_prefill_tile<Traits, scalar_t, acc_t, out_t, kIsRopeSeparate>(
                q_dram_window_nope,
                q_dram_window_rope,
                k_dram_window_nope,
                k_dram_window_rope,
                v_dram_window,
                lse_dram_window,
                out_dram_window,
                p_block_table,
                __builtin_amdgcn_readfirstlane(params.page_block_size),
                __builtin_amdgcn_readfirstlane(params.stride_s_k_nope),
                __builtin_amdgcn_readfirstlane(real_stride_s_k_rope),
                __builtin_amdgcn_readfirstlane(params.stride_s_v),
                seqlen_kv,
                1, // num_splits
                0, // split_id
                mask,
#if CK_MLA_FWD_FAST_EXP2
                static_cast<float>(params.scale_softmax * ck_tile::log2e_v<>),
#else
                params.scale_softmax,
#endif
                p_smem);
        }
        else
        {
            kn_ck_mla_fwd_splitkv_prefill_load_once_tile<Traits, scalar_t, acc_t, out_t>(
                q_dram_window_nope,
                q_dram_window_rope,
                k_dram_window_nope,
                k_dram_window_rope,
                lse_dram_window,
                out_dram_window,
                p_block_table,
                __builtin_amdgcn_readfirstlane(params.page_block_size),
                __builtin_amdgcn_readfirstlane(params.stride_s_k_nope),
                __builtin_amdgcn_readfirstlane(real_stride_s_k_rope),
                __builtin_amdgcn_readfirstlane(params.stride_s_v),
                seqlen_kv,
                1, // num_splits
                0, // split_id
                mask,
#if CK_MLA_FWD_FAST_EXP2
                static_cast<float>(params.scale_softmax * ck_tile::log2e_v<>),
#else
                params.scale_softmax,
#endif
                p_smem);
        }
    }
}

template <typename Traits, int32_t kMaxSplits, typename out_t, typename in_t>
__global__ void kn_ck_mla_fwd_splictkv_prefill_combine(
    const CkMlaPrefillFwdParams params)
{
    using Policy  = CkMlaCombineKernelPolicy<Traits, out_t, in_t>;
    using index_t = int64_t;

    __shared__ in_t lds_lse_scale[kMaxSplits];

    const int32_t bidx = blockIdx.z;

    const int32_t num_splits   = params.num_splits;
    const int32_t split_offset = bidx * params.num_splits;
    assert((num_splits > 1) && (num_splits <= kMaxSplits));

    const int32_t lane_id          = ck_tile::get_lane_id();
    const int32_t hidx             = blockIdx.y;
    const int32_t sidx             = blockIdx.x;
    const int32_t hsidx            = hidx * params.max_size_s_ori + sidx;
    const int32_t shidx            = hidx + sidx * params.size_h_ori;
    const int32_t size_hs          = params.size_h_ori * params.max_size_s_ori;
    const index_t offset_lse_accum = split_offset * size_hs + hsidx; // offset to split 0
    const index_t offset_lse       = bidx * size_hs + hsidx;

    if (ck_tile::get_warp_id() == 0)
    {
        const in_t* p_lse_accum = reinterpret_cast<in_t*>(params.p_softmax_lseaccum) + offset_lse_accum;
        in_t* p_lse             = reinterpret_cast<in_t*>(params.p_softmax_lse) + offset_lse;

        constexpr int32_t kNumLsePerThr = ck_tile::integer_divide_ceil(kMaxSplits, ck_tile::get_warp_size());
        in_t local_lse[kNumLsePerThr];

        // Load thread local LSE and get local max LSE
        in_t max_lse = -ck_tile::numeric<in_t>::infinity();
        #pragma unroll
        for (int32_t i = 0; i < kNumLsePerThr; ++i)
        {
            const int32_t split_idx = i * ck_tile::get_warp_size() + lane_id;
            const in_t lse =
                (split_idx < num_splits) ? p_lse_accum[split_idx * size_hs] : -ck_tile::numeric<in_t>::infinity();
            local_lse[i] = lse;
            max_lse = ck_tile::max(max_lse, lse);
        }

        // Get global max LSE
        #pragma unroll
        for (int32_t offset = ck_tile::get_warp_size() / 2; offset > 0; offset /= 2)
        {
            max_lse = ck_tile::max(max_lse, __shfl_xor(max_lse, offset));
        }

        // Get sum of LSE
        in_t sum_lse = 0.f;
        #pragma unroll
        for (int32_t i = 0; i < kNumLsePerThr; ++i)
        {
#ifdef CK_MLA_FWD_FAST_EXP2
            static_assert(0, "have not figured out if need exp2 here");
#endif
            sum_lse += ck_tile::exp(local_lse[i] - max_lse);
        }
        #pragma unroll
        for (int32_t offset = ck_tile::get_warp_size() / 2; offset > 0; offset /= 2)
        {
            sum_lse += __shfl_xor(sum_lse, offset);
        }

        // Get global LSE
        const auto [global_lse, output_lse] = [&]() {
            if ((sum_lse == 0.f) || (sum_lse != sum_lse))
            {
                return ck_tile::make_tuple(ck_tile::numeric<in_t>::infinity(), -ck_tile::numeric<in_t>::infinity());
            }
            else
            {
                const in_t lse = ck_tile::log(sum_lse) + max_lse;
                return ck_tile::make_tuple(lse, lse);
            }
        } ();

        if (lane_id == 0)
        {
            *p_lse = output_lse;
        }

        // Write LSE to LDS
        #pragma unroll
        for (int32_t i = 0; i < kNumLsePerThr; ++i)
        {
            const int32_t split_idx = i * ck_tile::get_warp_size() + lane_id;
            if (split_idx < num_splits)
            {
                lds_lse_scale[split_idx] = ck_tile::exp(local_lse[i] - global_lse);
            }
        }
    }

    __builtin_amdgcn_sched_barrier(0);
    ck_tile::block_sync_lds();

    static_assert(Traits::kSizeDV % Traits::kNumThreadsCombine == 0);

    auto oaccu_window =
        Policy::MakeOaccuTileWindow(params.p_output_accum, shidx, size_hs, split_offset, num_splits);

    auto reg_out = ck_tile::make_static_distributed_tensor<in_t>(
        decltype(ck_tile::load_tile(oaccu_window))::get_tile_distribution());
    ck_tile::set_tile(reg_out, 0.f);

    for (int32_t split_idx = 0; split_idx < num_splits; ++split_idx)
    {
        const in_t lse_scale = lds_lse_scale[split_idx];
        auto oaccu = ck_tile::load_tile(oaccu_window);
        ck_tile::sweep_tile(oaccu, [&](auto idx) {
            reg_out(idx) += lse_scale * oaccu(idx);
        });
        ck_tile::move_tile_window(oaccu_window, {size_hs, 0});
    }

    auto dram_out = Policy::MakeOutputTileWindow(
        static_cast<out_t*>(params.p_output) +
        bidx * params.stride_b_o + hidx * params.stride_h_o + sidx * params.stride_s_o);
    ck_tile::store_tile(dram_out, ck_tile::cast_tile<out_t>(reg_out));
}

// =====================================================================================================================
// Dispatch
//

template <typename Traits, typename scalar_t, typename acc_t, typename out_t, bool kIsCausal, bool kIsRopeSeparate>
void dispatch_ck_mla_fwd_splictkv_prefill(
    const CkMlaPrefillFwdParams& params)
{
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const int32_t num_blk =
        ck_tile::integer_divide_ceil(params.max_size_s_pk, Traits::kBlockM) * params.num_splits;
    const dim3 grid_attn = dim3(num_blk, params.size_h_pk, params.size_b);
    const dim3 grid_comb = dim3(params.max_size_s_ori, params.size_h_ori, params.size_b);


    if (params.num_splits > 1)
    {
        // out_t is not take into consideration when doing splits because combine shader is always expected to do
        // the final output type conversion.
        auto kn_attn = &kn_ck_mla_fwd_splictkv_prefill<Traits, scalar_t, acc_t, acc_t, kIsCausal, kIsRopeSeparate, true>;
        auto kn_comb =
            (params.num_splits <= 32)  ? &kn_ck_mla_fwd_splictkv_prefill_combine<Traits, 32,  scalar_t, acc_t> :
            // (params.num_splits <= 64)  ? &kn_ck_mla_fwd_splictkv_prefill_combine<Traits, 64,  scalar_t, acc_t> :
            // (params.num_splits <= 96)  ? &kn_ck_mla_fwd_splictkv_prefill_combine<Traits, 96,  scalar_t, acc_t> :
            // (params.num_splits <= 128) ? &kn_ck_mla_fwd_splictkv_prefill_combine<Traits, 128, scalar_t, acc_t> :
            static_cast<decltype(kn_ck_mla_fwd_splictkv_prefill_combine<Traits, 32, scalar_t, acc_t>)*>(nullptr);
        TORCH_CHECK(kn_comb != nullptr, "num_splits is larger than expected (<=128) !");
        kn_attn<<<grid_attn, Traits::kNumThreads, 0, stream>>>(params);
        kn_comb<<<grid_comb, Traits::kNumThreadsCombine, 0, stream>>>(params);
    }
    else
    {
        auto kn_attn = &kn_ck_mla_fwd_splictkv_prefill<Traits, scalar_t, acc_t, out_t, kIsCausal, kIsRopeSeparate, false>;
        kn_attn<<<grid_attn, Traits::kNumThreads, 0, stream>>>(params);
    }
}

// =====================================================================================================================
// Interfaces
//
#define CK_MLA_CASE(IS_CAUSAL, IS_ROPE_SEPARATE, ...)                  \
    if(is_causal == IS_CAUSAL && is_rope_separate == IS_ROPE_SEPARATE) \
    {                                                                  \
        constexpr bool Is_causal        = IS_CAUSAL;                   \
        constexpr bool Is_rope_separate = IS_ROPE_SEPARATE;            \
        __VA_ARGS__;                                                   \
    }

#define DISPATCH_CK_MLA_TYPES(TYPE, NAME, ...)                                   \
    switch((TYPE))                                                               \
    {                                                                            \
    case at::ScalarType::BFloat16: {                                             \
        using scalar_t = ck_tile::bf16_t;                                        \
        using out_t    = std::conditional_t<kForceOutAcc, acc_t, scalar_t>;      \
        CK_MLA_CASE(true, true, __VA_ARGS__)                                     \
        CK_MLA_CASE(true, false, __VA_ARGS__)                                    \
        CK_MLA_CASE(false, true, __VA_ARGS__)                                    \
        CK_MLA_CASE(false, false, __VA_ARGS__)                                   \
        break;                                                                   \
    }                                                                            \
    case at::ScalarType::Half: {                                                 \
        using scalar_t = ck_tile::fp16_t;                                        \
        using out_t    = std::conditional_t<kForceOutAcc, acc_t, scalar_t>;      \
        CK_MLA_CASE(true, true, __VA_ARGS__)                                     \
        CK_MLA_CASE(true, false, __VA_ARGS__)                                    \
        CK_MLA_CASE(false, true, __VA_ARGS__)                                    \
        CK_MLA_CASE(false, false, __VA_ARGS__)                                   \
        break;                                                                   \
    }                                                                            \
    default: TORCH_CHECK(false, NAME " does't support ", toString((TYPE)), "."); \
    }

int num_splits_heuristic(int batch_nhead_mblocks, int num_SMs, int num_n_blocks, int max_splits)
{
    int32_t result = 1;

    if (batch_nhead_mblocks < 0.8f * num_SMs)
    {
        max_splits = std::min(max_splits, std::min(num_SMs, num_n_blocks));
        float max_efficiency = 0.f;
        std::vector<float> efficiency;
        efficiency.reserve(max_splits);

        // Some splits are not eligible. For example, if we have 64 blocks and choose 11 splits,
        // we'll have 6 * 10 + 4 blocks. If we choose 12 splits, we'll have 6 * 11 + (-2) blocks
        // (i.e. it's 11 splits anyway).
        // So we check if the number of blocks per split is the same as the previous num_splits.
        auto is_split_eligible = [&num_n_blocks](int num_splits) {
            return (num_splits == 1) ||
                (ck_tile::integer_divide_ceil(num_n_blocks, num_splits) !=
                 ck_tile::integer_divide_ceil(num_n_blocks, num_splits - 1));
        };

        for(int num_splits = 1; num_splits <= max_splits; num_splits++)
        {
            if(!is_split_eligible(num_splits))
            {
                efficiency.push_back(0.f);
            }
            else
            {
                float n_waves = float(batch_nhead_mblocks * num_splits) / num_SMs;
                float eff     = n_waves / ceil(n_waves);
                if(eff > max_efficiency)
                {
                    max_efficiency = eff;
                }
                efficiency.push_back(eff);
            }
        }

        for(int num_splits = 1; num_splits <= max_splits; num_splits++)
        {
            if(!is_split_eligible(num_splits))
            {
                continue;
            }

            if(efficiency[num_splits - 1] >= 0.85 * max_efficiency)
            {
                result = num_splits;
                break;
            }
        }
    }

    return result;
}

template <typename Traits>
int32_t calculate_num_splits(
    const int32_t size_b,
    const int32_t size_h,
    const int32_t size_s)
{
    hipDevice_t dev;
    hipDeviceProp_t dev_prop;
    ck_tile::hip_check_error(hipGetDevice(&dev));
    ck_tile::hip_check_error(hipGetDeviceProperties(&dev_prop, dev));
    const int32_t cu_count = dev_prop.multiProcessorCount;

    const int32_t num_m_blocks = ck_tile::integer_divide_ceil(size_s, Traits::kBlockM);
    const int32_t num_n_blocks = ck_tile::integer_divide_ceil(Traits::kSizeDV, Traits::kBlockN1);

    return num_splits_heuristic(size_b * size_h * num_m_blocks, cu_count * Traits::kCuReuse, num_n_blocks, 128);
}

std::vector<torch::Tensor>
ck_mla_fwd_prefill_with_kvcache_impl(
    torch::Tensor& query_nope,
    const torch::Tensor& key_nope_cache,
    const torch::Tensor& value_cache,
    const int32_t head_size_v,
    const torch::Tensor& seqlens_qo,
    const torch::Tensor& seqlens_kv,
    const torch::Tensor& block_table,
    const float softmax_scale,
    const bool is_causal,
    std::optional<torch::Tensor>& query_rope,
    const std::optional<torch::Tensor>& key_rope_cache)
{
    const bool is_rope_separate = query_rope.has_value() && key_rope_cache.has_value();

    constexpr bool kKVLoadOnce = false;
    constexpr bool kEnableXqa  = true;
    //TODO:
    // cases need maintenance:
    //     warp4 + load_once=false + occ=2
    //     warp8 + load_once=true  + occ=1
    // targe case:
    // warp4 + load_once=true + occ=1
    //                             dqk  dv   m0  n0  n1   #warp  wave_occu
    using Traits = std::conditional_t<kKVLoadOnce,
        CkMlaPrefillKernelTrait<576, 512, 64, 16, 512, 8,     1,   kKVLoadOnce, kEnableXqa>,
        CkMlaPrefillKernelTrait<576, 512, 64, 64, 256, 4,     2,   kKVLoadOnce, kEnableXqa>>;
    constexpr bool kForceOutAcc = false;
    using acc_t                 = float;

    torch::Tensor vcache = value_cache.data_ptr() ? value_cache : key_nope_cache;

    auto opts = query_nope.options();
    static_assert(std::is_same_v<acc_t, float>);
    auto opts_acc = opts.dtype(torch::kFloat32);

    const int32_t batch_size        = query_nope.size(0);
    const int32_t max_seqlen_qo_ori = query_nope.size(1);
    const int32_t num_heads_q_ori   = query_nope.size(2);
    int32_t max_seqlen_qo           = max_seqlen_qo_ori;
    int32_t num_heads_q             = num_heads_q_ori;

    const int32_t head_size_nope = query_nope.size(3);
    const int32_t head_size_rope = is_rope_separate ? query_rope.value().size(3) : 0;
    const int32_t head_size = head_size_nope + head_size_rope;
    TORCH_CHECK((head_size == 576) && (head_size_v == 512), "Only support QK head dim 576 and V head dim 512!");

    const int32_t num_blocks      = key_nope_cache.size(0);
    const int32_t page_block_size = key_nope_cache.size(1);
    const int32_t num_heads_k     = key_nope_cache.size(2);

    TORCH_CHECK(num_heads_q % num_heads_k == 0,
                "Number of heads in key/value must divide number of heads in query");

    const int32_t hq_hk_ratio = num_heads_q_ori / num_heads_k;
    int32_t mask_y_ratio = 1;

    if constexpr(kEnableXqa)
    {
        max_seqlen_qo = max_seqlen_qo_ori * hq_hk_ratio;
        num_heads_q   = num_heads_k;
        mask_y_ratio  = hq_hk_ratio;
    }

    const int32_t num_splits = calculate_num_splits<Traits>(batch_size, num_heads_q, max_seqlen_qo);
    const bool    do_splits = num_splits > 1;

    // Combine shader, which only exists when num_splits > 1, will conduct type convert by default and force.
    // Thus, kForceOutAcc doesn't work in this case.
    auto output = torch::empty({batch_size, max_seqlen_qo_ori, num_heads_q_ori, head_size_v},
                               (kForceOutAcc && !do_splits) ? opts_acc : opts);
    auto softmax_lse = torch::empty({batch_size, num_heads_q_ori, max_seqlen_qo_ori}, opts_acc);

    CkMlaPrefillFwdParams params = {};

    params.num_splits    = num_splits;
    params.p_seqlens_qo  = seqlens_qo.data_ptr<int32_t>();
    params.p_seqlens_kv  = seqlens_kv.data_ptr<int32_t>();
    params.p_block_table = block_table.data_ptr<int32_t>();

    params.p_query_nope  = query_nope.data_ptr();
    params.p_key_nope    = key_nope_cache.data_ptr();
    params.p_value       = vcache.data_ptr();
    params.p_output      = output.data_ptr();
    params.p_softmax_lse = softmax_lse.data_ptr();

    params.size_b                   = batch_size;
    params.max_size_s_pk            = max_seqlen_qo;
    params.max_size_s_ori           = max_seqlen_qo_ori;
    params.size_h_pk                = num_heads_q;
    params.size_h_ori               = num_heads_q_ori;
    params.hq_hk_ratio              = hq_hk_ratio;
    params.block_table_batch_stride = block_table.stride(0);
    params.num_page_blocks          = num_blocks;
    params.page_block_size          = page_block_size;
    params.scale_softmax            = softmax_scale;

    params.mask_y_ratio_mdiv = ck_tile::mdiv{static_cast<uint32_t>(mask_y_ratio)};

    params.stride_b_q_nope = query_nope.stride(0);
    params.stride_s_q_nope = query_nope.stride(1);
    params.stride_h_q_nope = query_nope.stride(2);
    params.stride_b_k_nope = key_nope_cache.stride(0);
    params.stride_s_k_nope = key_nope_cache.stride(1); // size_hk * size_d
    params.stride_h_k_nope = key_nope_cache.stride(2);
    params.stride_b_v      = vcache.stride(0);
    params.stride_s_v      = vcache.stride(1); // size_hk * size_d
    params.stride_h_v      = vcache.stride(2);
    params.stride_b_o      = output.stride(0);
    params.stride_s_o      = output.stride(1);
    params.stride_h_o      = output.stride(2);
    params.stride_h_lse    = softmax_lse.stride(1);
    if (is_rope_separate)
    {
        params.p_query_rope    = query_rope.value().data_ptr();
        params.p_key_rope      = key_rope_cache.value().data_ptr();
        params.stride_b_q_rope = query_rope.value().stride(0);
        params.stride_s_q_rope = query_rope.value().stride(1);
        params.stride_h_q_rope = query_rope.value().stride(2);
        params.stride_b_k_rope = key_rope_cache.value().stride(0);
        params.stride_s_k_rope = key_rope_cache.value().stride(1); // size_hk * size_d
        params.stride_h_k_rope = key_rope_cache.value().stride(2);
    }

    if(num_splits > 1)
    {
        auto output_accum =
            torch::empty({batch_size, num_splits, max_seqlen_qo_ori, num_heads_q_ori, head_size_v}, opts_acc);
        auto softmax_lseaccum =
            torch::empty({batch_size, num_splits, num_heads_q_ori, max_seqlen_qo_ori}, opts_acc);

        params.p_softmax_lseaccum = softmax_lseaccum.data_ptr();
        params.p_output_accum     = output_accum.data_ptr();
        params.stride_b_oacc      = output_accum.stride(0);
        params.stride_h_oacc      = output_accum.stride(3);
        params.stride_sp_oacc     = output_accum.stride(1);
        params.stride_s_oacc      = output_accum.stride(2);
        params.stride_b_lseacc    = softmax_lseaccum.stride(0);
        params.stride_h_lseacc    = softmax_lseaccum.stride(2);
        params.stride_sp_lseacc   = softmax_lseaccum.stride(1);
    }

    DISPATCH_CK_MLA_TYPES(
        query_nope.scalar_type(),
        "ck_mla_fwd",
        [&](){
            dispatch_ck_mla_fwd_splictkv_prefill<Traits, scalar_t, acc_t, out_t, Is_causal, Is_rope_separate>(params);
        }();
    );
    // assert(is_causal == false);
    // assert(query_nope.scalar_type() == at::ScalarType::BFloat16);
    // assert(query_rope.value().scalar_type() == at::ScalarType::BFloat16);
    // assert(is_rope_separate == false);
    // using scalar_t = ck_tile::bf16_t;
    // using out_t = std::conditional_t<kForceOutAcc, acc_t, scalar_t>;
    // dispatch_ck_mla_fwd_splictkv_prefill<Traits, scalar_t, acc_t, out_t, false, false>(params);


    return {output.to(opts), softmax_lse};
}
