#include "hip/hip_runtime.h"
// SPDX-License-Identifier: MIT
// Copyright (C) 2025, Advanced Micro Devices, Inc. All rights reserved.

#include <sstream>
#include <torch/python.h>
#include <c10/cuda/CUDAGuard.h>
#include "aiter_hip_common.h"
#include "mla.h"

template <int32_t kSizeDV_,
          int32_t kNumHeadQ_,
          bool    kOutputLse_>
struct MlaReduceKernelV1Traits
{
    static constexpr int32_t kSizeDV          = kSizeDV_;       // hidden dimension size of value/output
    static constexpr int32_t kNumHeadQ        = kNumHeadQ_;     // head count of q
    static constexpr int32_t kNumWarps        = 2;
    static constexpr int32_t kNumThreads      = kNumWarps * ck_tile::get_warp_size();
    static constexpr int32_t kMaxVgprLocalLse = 16;             // scratch buffer will be used with larger value
    static constexpr bool    kOutputLse       = kOutputLse_;
};

struct MlaReduceKernelV1Params
{
    const int32_t*            p_reduce_indptr;
    const MlaPartialTileInfo* p_reduce_final_map;
    const int32_t*            p_reduce_partial_map;

    void* __restrict__ p_final_lse;
    void* __restrict__ p_final_output;
    void* __restrict__ p_partial_lse;
    void* __restrict__ p_partial_output;

    int32_t stride_s_o;
    int32_t stride_h_o;
    int32_t max_splits;
};

// Returns count of warps which don't contain any idle thread.
template <int32_t NumWarps, int32_t M, int32_t N>
CK_TILE_HOST_DEVICE static constexpr auto GetMaxNumWarpsForTile()
{
    static_assert(NumWarps == 1 || NumWarps == 2 || NumWarps == 4);
    constexpr int32_t ElemPerThread = (M * N) / (NumWarps * ck_tile::get_warp_size());
    if constexpr(0 < ElemPerThread)
    {
        return NumWarps;
    }
    else
    {
        return GetMaxNumWarpsForTile<NumWarps / 2, M, N>();
    }
}

// Returns vector size for given warp count for handing the specified matrix.
template <int32_t NumWarps, int32_t M, int32_t N, typename scalar_t>
CK_TILE_HOST_DEVICE static constexpr auto GetVectorSizeForTile()
{
    constexpr int32_t MaxNumWarps = GetMaxNumWarpsForTile<NumWarps, M, N>();
    constexpr int32_t ElemPerThread = (M * N) / (MaxNumWarps * ck_tile::get_warp_size());
    constexpr int32_t MaxNPerThread = 16 / sizeof(scalar_t);
    return ck_tile::min(MaxNPerThread, ElemPerThread);
}

template <typename Traits, typename scalar_t>
CK_TILE_DEVICE static constexpr auto MakeOutputTileDistribution()
{
    constexpr int32_t kVectorN     = GetVectorSizeForTile<Traits::kNumWarps, 1, Traits::kSizeDV, scalar_t>();
    constexpr int32_t kThrPerWarpN = ck_tile::get_warp_size();
    constexpr int32_t kNumWarpN    = Traits::kNumWarps;
    constexpr int32_t kNumRepeat   = ck_tile::max(1, Traits::kSizeDV / kThrPerWarpN / kNumWarpN / kVectorN);

    return ck_tile::make_static_tile_distribution(
        ck_tile::tile_distribution_encoding<
            ck_tile::sequence<>,    // no replicate
            ck_tile::tuple<ck_tile::sequence<1>,
                           ck_tile::sequence<kNumRepeat, kNumWarpN, kThrPerWarpN, kVectorN>>,
            ck_tile::tuple<ck_tile::sequence<2>, ck_tile::sequence<2>>,
            ck_tile::tuple<ck_tile::sequence<1>, ck_tile::sequence<2>>,
            ck_tile::sequence<2, 1, 2>,
            ck_tile::sequence<0, 0, 3>>{});
}

template <typename Traits, typename scalar_t>
CK_TILE_DEVICE static auto MakeTileWindow(
    scalar_t* p_tile)
{
    const auto naive_view =
        ck_tile::make_naive_tensor_view<ck_tile::address_space_enum::global>(
            p_tile,
            ck_tile::make_tuple(1, Traits::kSizeDV),    // lengths
            ck_tile::make_tuple(Traits::kSizeDV, 1),    // strides
            ck_tile::number<Traits::kSizeDV>{},         // last dim alignment
            ck_tile::number<1>{});                      // last dim stride

    const auto tile_window = ck_tile::make_tile_window(
        naive_view,
        ck_tile::make_tuple(ck_tile::number<1>{},               // window size
                            ck_tile::number<Traits::kSizeDV>{}),
        {0, 0});                                                // origin

    return tile_window;
}

template <typename T>
class LocalLseLds
{
public:
    CK_TILE_DEVICE LocalLseLds(T* p_local_lse, const int32_t group_size, const int32_t idx_in_group) :
        p_local_lse_(p_local_lse), group_size_(group_size), idx_in_group_(idx_in_group) {}
    CK_TILE_DEVICE T& operator[](int32_t idx) { return p_local_lse_[idx * group_size_ + idx_in_group_]; }
    CK_TILE_DEVICE T operator[](int32_t idx) const { return p_local_lse_[idx * group_size_ + idx_in_group_]; }

private:
    T* p_local_lse_;
    int32_t group_size_;
    int32_t idx_in_group_;
};

template <typename Traits,
          bool kFastMode,
          typename LocalLse,
          typename lse_t>
CK_TILE_DEVICE void reduce_lse(
    const MlaReduceKernelV1Params& params,
    const int32_t                  seq_idx,
    const int32_t                  reduce_tile_start,
    const int32_t                  reduce_tile_end,
    const int32_t                  num_lse_per_thr,
    const int32_t                  q_len,
    const float*                   p_partial_lse_seq_base,
    LocalLse&                      local_lse,
    float*                         p_lds_lse_scale,
    lse_t*                         p_final_lse_base)
{
    if (ck_tile::get_warp_id() == 0)
    {
        const int32_t lane_idx = ck_tile::get_lane_id();

        // Load thread local LSE and get local max LSE
        float max_lse = -INFINITY;

        #pragma unroll 2
        for (int32_t i = 0; i < num_lse_per_thr; ++i)
        {
            const int32_t split_idx = i * ck_tile::get_warp_size() + lane_idx;
            const int32_t tile_idx = reduce_tile_start + split_idx;
            if (tile_idx < reduce_tile_end)
            {
                const int32_t q_loc = [&]() {
                    if constexpr (kFastMode)
                    {
                        return tile_idx * q_len;
                    }
                    else
                    {
                        return params.p_reduce_partial_map[tile_idx];
                    }
                }();
                const int64_t reduce_tile_pos = q_loc * int64_t(Traits::kNumHeadQ);
                const float lse = p_partial_lse_seq_base[reduce_tile_pos];
                local_lse[i] = lse;
                max_lse = ck_tile::max(max_lse, lse);
            }
            else
            {
                local_lse[i] = -INFINITY;
            }
        }

        // Get global max LSE
        #pragma unroll
        for (int32_t offset = ck_tile::get_warp_size() / 2; offset > 0; offset /= 2)
        {
            const int32_t srd_lane = (offset ^ ck_tile::get_warp_size()) ^ ck_tile::get_lane_id();
            max_lse = ck_tile::max(max_lse, ck_tile::warp_shuffle(max_lse, srd_lane));
        }

        // Get sum of LSE
        float sum_lse = 0.f;
        #pragma unroll 2
        for (int32_t i = 0; i < num_lse_per_thr; ++i)
        {
            sum_lse += expf(local_lse[i] - max_lse);
        }
        #pragma unroll
        for (int32_t offset = ck_tile::get_warp_size() / 2; offset > 0; offset /= 2)
        {
            const int32_t srd_lane = (offset ^ ck_tile::get_warp_size()) ^ ck_tile::get_lane_id();
            sum_lse += ck_tile::warp_shuffle(sum_lse, srd_lane);
        }

        // Get global LSE
        float global_lse = ((sum_lse == 0.f) || (sum_lse != sum_lse)) ? INFINITY : (logf(sum_lse) + max_lse);
        if constexpr (Traits::kOutputLse)
        {
            if (lane_idx == 0)
            {
                lse_t* p_final_lse = p_final_lse_base + seq_idx * Traits::kNumHeadQ;
                *p_final_lse = ck_tile::type_convert<lse_t>(global_lse);
            }
        }

        // Write LSE to LDS
        #pragma unroll 2
        for (int32_t i = 0; i < num_lse_per_thr; ++i)
        {
            const int32_t split_idx = i * ck_tile::get_warp_size() + lane_idx;
            if ((reduce_tile_start + split_idx) < reduce_tile_end)
            {
                p_lds_lse_scale[split_idx] = expf(local_lse[i] - global_lse);
            }
        }
    }
}

template <typename Traits,
          bool kFastMode,
          typename out_t>
CK_TILE_DEVICE void reduce_output(
    const MlaReduceKernelV1Params& params,
    const int32_t                  seq_idx,
    const int32_t                  reduce_tile_start,
    const int32_t                  reduce_tile_end,
    const int32_t                  q_len,
    const float*                   p_lds_lse_scale,
    const float*                   p_partial_output_seq_base,
    out_t*                         p_final_out_base)
{
    auto oaccu_window = ck_tile::make_tile_window(MakeTileWindow<Traits, const float>(nullptr),
                                                  MakeOutputTileDistribution<Traits, const float>());
    auto reg_out = ck_tile::make_static_distributed_tensor<float>(
        decltype(ck_tile::load_tile(oaccu_window))::get_tile_distribution());
    ck_tile::set_tile(reg_out, 0.f);

    for (int32_t tile_idx = reduce_tile_start; tile_idx < reduce_tile_end; ++tile_idx)
    {
        const int32_t split_idx = tile_idx - reduce_tile_start;
        const int32_t q_loc = [&]() {
            if constexpr (kFastMode)
            {
                return tile_idx * q_len;
            }
            else
            {
                return params.p_reduce_partial_map[tile_idx];
            }
        }();
        const int64_t reduce_tile_pos = q_loc * int64_t(Traits::kNumHeadQ * Traits::kSizeDV);
        const float* p_partial_output = p_partial_output_seq_base + reduce_tile_pos;
        oaccu_window.set_bottom_tensor_view_data_ptr(p_partial_output);

        const float lse_scale = p_lds_lse_scale[split_idx];
        auto oaccu = ck_tile::load_tile(oaccu_window);
        ck_tile::sweep_tile(oaccu, [&](auto idx) {
            reg_out(idx) += lse_scale * oaccu(idx);
        });
    }

    out_t* p_final_out = p_final_out_base + seq_idx * params.stride_s_o;
    auto dram_out = MakeTileWindow<Traits, out_t>(p_final_out);
    ck_tile::store_tile(dram_out, ck_tile::cast_tile<out_t>(reg_out));
}

template <typename Traits, typename lse_t, typename out_t>
__global__ void kn_mla_reduce_v1(
    const MlaReduceKernelV1Params params)
{
    extern __shared__ float p_lds_lse_scale[];

    const int32_t head_idx = blockIdx.x;
    const int32_t work_idx = blockIdx.y;

    const int32_t reduce_tile_start = params.p_reduce_indptr[work_idx];
    const int32_t reduce_tile_end = params.p_reduce_indptr[work_idx + 1];


    if (reduce_tile_start < reduce_tile_end)
    {
        int32_t q_len = 0;
        MlaPartialTileInfo final_loc{};
        const int32_t fast_mode = params.p_reduce_partial_map[0];
        if (fast_mode == -1)
        {
            q_len = params.p_reduce_partial_map[1];
            final_loc.q_start = q_len * work_idx;
            final_loc.q_end   = final_loc.q_start + q_len;
        }
        else
        {
            final_loc = params.p_reduce_final_map[work_idx];
        }

        // Assuming that the layout of LSE final output is in [bs, h].
        // Thus, stride of head is 1 and stride of b/s is #heads.
        lse_t* p_final_lse_base = reinterpret_cast<lse_t*>(params.p_final_lse) + head_idx;
        const float* p_partial_lse_base =
            reinterpret_cast<const float*>(params.p_partial_lse) + head_idx;

        // Assuming that the layout of partial output is in [bs, h, d].
        // Thus, stride of hidden dim is 1, head is Traits::kSizeDV and b/s is Traits::kSizeDV * #heads
        // while the strides are 1, params.stride_h_o and params.stride_s_o for final output.
        out_t* p_final_out_base = reinterpret_cast<out_t*>(params.p_final_output) + head_idx * params.stride_h_o;
        const float* p_partial_output_base =
            reinterpret_cast<float*>(params.p_partial_output) + head_idx * Traits::kSizeDV;

        const int32_t num_lse_per_thr =
            ck_tile::integer_divide_ceil(params.max_splits, ck_tile::get_warp_size());

        if (fast_mode == -1)
        {
            for (int32_t seq_idx = final_loc.q_start; seq_idx < final_loc.q_end; ++seq_idx)
            {
                const int32_t local_seqlen_idx = seq_idx - final_loc.q_start;
                const float* p_partial_lse_seq_base = p_partial_lse_base + local_seqlen_idx * Traits::kNumHeadQ;
                const float* p_partial_output_seq_base =
                    p_partial_output_base + local_seqlen_idx * Traits::kNumHeadQ * Traits::kSizeDV;

                float* p_local_lse = p_lds_lse_scale + params.max_splits;
                LocalLseLds<float> local_lse(p_local_lse, ck_tile::get_warp_size(), ck_tile::get_lane_id());
                reduce_lse<Traits, true>(
                    params,
                    seq_idx,
                    reduce_tile_start,
                    reduce_tile_end,
                    num_lse_per_thr,
                    q_len,
                    p_partial_lse_seq_base,
                    local_lse,
                    p_lds_lse_scale,
                    p_final_lse_base);

                __builtin_amdgcn_sched_barrier(0);
                ck_tile::block_sync_lds();

                reduce_output<Traits, true>(
                    params,
                    seq_idx,
                    reduce_tile_start,
                    reduce_tile_end,
                    q_len,
                    p_lds_lse_scale,
                    p_partial_output_seq_base,
                    p_final_out_base);
            }
        }
        else
        {
            for (int32_t seq_idx = final_loc.q_start; seq_idx < final_loc.q_end; ++seq_idx)
            {
                const int32_t local_seqlen_idx = seq_idx - final_loc.q_start;
                const float* p_partial_lse_seq_base = p_partial_lse_base + local_seqlen_idx * Traits::kNumHeadQ;
                const float* p_partial_output_seq_base =
                    p_partial_output_base + local_seqlen_idx * Traits::kNumHeadQ * Traits::kSizeDV;

                float* p_local_lse = p_lds_lse_scale + params.max_splits;
                LocalLseLds<float> local_lse(p_local_lse, ck_tile::get_warp_size(), ck_tile::get_lane_id());
                reduce_lse<Traits, false>(
                    params,
                    seq_idx,
                    reduce_tile_start,
                    reduce_tile_end,
                    num_lse_per_thr,
                    q_len,
                    p_partial_lse_seq_base,
                    local_lse,
                    p_lds_lse_scale,
                    p_final_lse_base);

                __builtin_amdgcn_sched_barrier(0);
                ck_tile::block_sync_lds();

                reduce_output<Traits, false>(
                    params,
                    seq_idx,
                    reduce_tile_start,
                    reduce_tile_end,
                    q_len,
                    p_lds_lse_scale,
                    p_partial_output_seq_base,
                    p_final_out_base);
            }
        }
    }
}

#define MLA_MERGE_CASE(NUM_HEAD_C, OUTPUT_LSE_C, NAME, ...)                                                 \
    constexpr int32_t NumHeads  = (NUM_HEAD_C);                                                             \
    constexpr bool    OutputLse = (OUTPUT_LSE_C);                                                           \
    using Traits = MlaReduceKernelV1Traits<512, NumHeads, OutputLse>;                                       \
    __VA_ARGS__;

#define MLA_MERGE_CASE_IF(NUM_HEAD, NUM_HEAD_C, OUTPUT_LSE, OUTPUT_LSE_C, NAME, ...)                        \
    if (((NUM_HEAD) == (NUM_HEAD_C)) && ((OUTPUT_LSE) == (OUTPUT_LSE_C)))                                   \
    {                                                                                                       \
        MLA_MERGE_CASE(NUM_HEAD_C, OUTPUT_LSE_C, NAME, __VA_ARGS__)                                         \
    }

#define MLA_MERGE_CASE_EF(NUM_HEAD, NUM_HEAD_C, OUTPUT_LSE, OUTPUT_LSE_C, NAME, ...)                        \
    else if (((NUM_HEAD) == (NUM_HEAD_C)) && ((OUTPUT_LSE) == (OUTPUT_LSE_C)))                              \
    {                                                                                                       \
        MLA_MERGE_CASE(NUM_HEAD_C, OUTPUT_LSE_C, NAME, __VA_ARGS__)                                         \
    }

#define MLA_MERGE_ERROR(NUM_HEAD, OUTPUT_LSE, NAME)                                                         \
    {                                                                                                       \
        std::stringstream ss;                                                                               \
        ss << "#heads: " << (NUM_HEAD) << ", Output LSE: " << (OUTPUT_LSE);                                 \
        TORCH_CHECK(false, NAME " doesn't support the specified settings: ", ss.str().c_str(), ".");        \
    }

#define DISPATCH_MLA_MERGE_KERNEL(LSE_TYPE, OUT_TYPE, NUM_HEAD, OUTPUT_LSE, NAME, ...)                      \
    switch ((LSE_TYPE))                                                                                     \
    {                                                                                                       \
        case at::ScalarType::Float:                                                                         \
        {                                                                                                   \
            using lse_t = float;                                                                            \
            switch ((OUT_TYPE))                                                                             \
            {                                                                                               \
                case at::ScalarType::BFloat16:                                                              \
                {                                                                                           \
                    using out_t = ck_tile::bf16_t;                                                          \
                    MLA_MERGE_CASE_IF(NUM_HEAD,  16, OUTPUT_LSE, true,  NAME, __VA_ARGS__)                  \
                    MLA_MERGE_CASE_EF(NUM_HEAD,  16, OUTPUT_LSE, false, NAME, __VA_ARGS__)                  \
                    MLA_MERGE_CASE_EF(NUM_HEAD, 128, OUTPUT_LSE, true,  NAME, __VA_ARGS__)                  \
                    MLA_MERGE_CASE_EF(NUM_HEAD, 128, OUTPUT_LSE, false, NAME, __VA_ARGS__)                  \
                    else MLA_MERGE_ERROR(NUM_HEAD, OUTPUT_LSE, NAME);                                       \
                }                                                                                           \
                break;                                                                                      \
                case at::ScalarType::Half:                                                                  \
                {                                                                                           \
                    using out_t = ck_tile::fp16_t;                                                          \
                    MLA_MERGE_CASE_IF(NUM_HEAD,  16, OUTPUT_LSE, true,  NAME, __VA_ARGS__)                  \
                    MLA_MERGE_CASE_EF(NUM_HEAD,  16, OUTPUT_LSE, false, NAME, __VA_ARGS__)                  \
                    MLA_MERGE_CASE_EF(NUM_HEAD, 128, OUTPUT_LSE, true,  NAME, __VA_ARGS__)                  \
                    MLA_MERGE_CASE_EF(NUM_HEAD, 128, OUTPUT_LSE, false, NAME, __VA_ARGS__)                  \
                    else MLA_MERGE_ERROR(NUM_HEAD, OUTPUT_LSE, NAME);                                       \
                }                                                                                           \
                break;                                                                                      \
                default:                                                                                    \
                    TORCH_CHECK(false, NAME " doesn't support output type ", toString((OUT_TYPE)), ".");    \
            }                                                                                               \
        }                                                                                                   \
        break;                                                                                              \
        default:                                                                                            \
            TORCH_CHECK(false, NAME " doesn't support LSE type ", toString((LSE_TYPE)), ".");               \
    }

template <typename Traits, typename lse_t, typename out_t>
void dispatch_mla_reduce_v1(
    const MlaReduceKernelV1Params& params,
    const int32_t                  num_reduce_tile,
    const hipStream_t&            stream)
{
    hipDevice_t dev;
    hipDeviceProp_t dev_prop;
    HIP_CALL(hipGetDevice(&dev));
    HIP_CALL(hipGetDeviceProperties(&dev_prop, dev));

    const int32_t lds_size = params.max_splits * sizeof(float) * 2;
    if (lds_size <= dev_prop.maxSharedMemoryPerMultiProcessor)
    {
        const dim3 grid = dim3(Traits::kNumHeadQ, num_reduce_tile);
        kn_mla_reduce_v1<Traits, lse_t, out_t><<<grid, Traits::kNumThreads, lds_size, stream>>>(params);
    }
    else
    {
        TORCH_CHECK(false, "kn_mla_reduce_v1: There are too much splits. We cannot handle them.");
    }
}

void mla_reduce_v1(
    const torch::Tensor& partial_output,        // contiguous [max(reduce_partial_map)+s, h, dv]
    const torch::Tensor& partial_lse,           // contiguous [max(reduce_partial_map)+s, h]
    const torch::Tensor& reduce_indptr,         // contiguous [#work + 1]
    const torch::Tensor& reduce_final_map,      // contiguous [#work, 2]
    const torch::Tensor& reduce_partial_map,    // contiguous [reduce_indptr[-1]]
    torch::Tensor&       final_output,          //            [bs, h, dv]
    std::optional<torch::Tensor>&       final_lse)             // contiguous [bs, h]
{
    const at::cuda::OptionalCUDAGuard device_guard(device_of(final_output));
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    hipDevice_t dev;
    hipDeviceProp_t dev_prop;
    HIP_CALL(hipGetDevice(&dev));
    HIP_CALL(hipGetDeviceProperties(&dev_prop, dev));

    const bool output_lse = final_lse.has_value();
    const int32_t num_reduce_tile = reduce_indptr.size(0) - 1;
    const int32_t num_heads = partial_output.size(-2);

    if (num_reduce_tile > 0)
    {
        MlaReduceKernelV1Params params = {};
        params.p_reduce_indptr = reduce_indptr.data_ptr<int32_t>();
        params.p_reduce_final_map =
            reinterpret_cast<const MlaPartialTileInfo*>(reduce_final_map.data_ptr());
        params.p_reduce_partial_map = reduce_partial_map.data_ptr<int32_t>();
        params.p_final_lse = output_lse ? final_lse.value().data_ptr() : nullptr;
        params.p_final_output = final_output.data_ptr();
        params.p_partial_lse = partial_lse.data_ptr();
        params.p_partial_output = partial_output.data_ptr();
        params.stride_s_o = final_output.stride(-3);
        params.stride_h_o = final_output.stride(-2);
        params.max_splits = dev_prop.multiProcessorCount;

        DISPATCH_MLA_MERGE_KERNEL(
            output_lse ? final_lse.value().scalar_type() : at::ScalarType::Float,
            final_output.scalar_type(),
            num_heads,
            output_lse,
            "kn_mla_reduce_v1",
            dispatch_mla_reduce_v1<Traits, lse_t, out_t>(params, num_reduce_tile, stream)
        );
    }
}
