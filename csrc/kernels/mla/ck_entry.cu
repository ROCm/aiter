#include "hip/hip_runtime.h"
#include <torch/python.h>
#include <c10/cuda/CUDAGuard.h>
#include "ck_mla.h"

// clang-format off
std::vector<torch::Tensor> get_mla_metadata(
    const torch::Tensor& p_seqlens_kv,          // [batch size]
    const int32_t        num_heads_per_head_k,
    const int32_t        num_heads_k)
{
    assert(false);

    auto opts = p_seqlens_kv.options();
    auto ret  = torch::empty({4}, opts);
    return {ret};
}

std::vector<torch::Tensor> ck_mla_fwd_with_kvcache_impl(
    torch::Tensor&                      query_nope,               // [batch size,  seqlen of q, head count of q,  head dim of qk]
    const torch::Tensor&                key_nope_cache,           // [block count, block size,  head count of kv, head dim of qk]
    const torch::Tensor&                value_cache,              // [block count, block size,  head count of kv, head dim of v ]
    const int32_t                       head_size_v,
    const torch::Tensor&                seqlens_qo,
    const torch::Tensor&                seqlens_kv,               // [batch size]
    const torch::Tensor&                block_table,              // [batch size, max blocks per seq]
    const float                         softmax_scale,
    const bool                          is_causal,
    const torch::Tensor&                tile_scheduler_metadata,  // [num cu parts, metadata size]
    const torch::Tensor&                num_splits,               // [batch size + 1]
    std::optional<torch::Tensor>&       query_rope,
    const std::optional<torch::Tensor>& key_rope_cache)
{
    const int32_t seqlen_q      = query_nope.size(1);
    const bool is_rope_separate = query_rope.has_value() && key_rope_cache.has_value();
    if(seqlen_q < 32 && !is_rope_separate) // TODO: not support nope/rope split
    {
        return ck_mla_fwd_decode_with_kvcache_impl(
            query_nope,
            key_nope_cache,
            value_cache,
            head_size_v,
            seqlens_kv,
            block_table,
            softmax_scale,
            is_causal,
            tile_scheduler_metadata,
            num_splits
        );
    }
    else
    {
        return ck_mla_fwd_prefill_with_kvcache_impl(
            query_nope,
            key_nope_cache,
            value_cache,
            head_size_v,
            seqlens_qo,
            seqlens_kv,
            block_table,
            softmax_scale,
            is_causal,
            query_rope,
            key_rope_cache
        );
    }
}
// clang-format on
