#include "mha_common.h"
#include "py_itfs_common.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>

namespace aiter {
namespace torch_itfs {

std::vector<at::Tensor> poyenc_mha_v3_fwd(at::Tensor& q,       // [b, sq, hq, d]
                                          const at::Tensor& k, // [b, sk, hk, d]
                                          const at::Tensor& v  // [b, sk, hk, d_v]
)
{
    auto q_dtype = q.dtype();
    TORCH_CHECK(q_dtype == torch::kFloat16 || q_dtype == torch::kBFloat16,
                "FlashAttention only support fp16 and bf16 data type");

    TORCH_CHECK(k.dtype() == q_dtype, "query and key must have the same dtype");
    TORCH_CHECK(v.dtype() == q_dtype, "query and value must have the same dtype");

    std::string q_dtype_str = q_dtype == torch::kFloat16 ? "fp16" : "bf16";

    CHECK_DEVICE(q);
    CHECK_DEVICE(k);
    CHECK_DEVICE(v);

    TORCH_CHECK(q.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    TORCH_CHECK(k.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    TORCH_CHECK(v.stride(-1) == 1, "Input tensor must have contiguous last dimension");

    const auto sizes = q.sizes();

    const int batch_size  = sizes[0];
    int seqlen_q          = sizes[1];
    int num_heads         = sizes[2];
    const int head_size_q = sizes[3];
    const int head_size_v = v.sizes()[3];
    const int seqlen_k    = k.size(1);
    const int num_heads_k = k.size(2);
    TORCH_CHECK(batch_size > 0, "batch size must be positive");
    TORCH_CHECK(head_size_q <= 256, "CK only supports head dimension at most 256");
    TORCH_CHECK(head_size_v <= 256, "CK only supports head dimension at most 256");
    TORCH_CHECK(head_size_q % 8 == 0,
                "query, key, value, and out_ must have a head_size_q that is a multiple of 8");
    TORCH_CHECK(head_size_v % 8 == 0,
                "query, key, value, and out_ must have a head_size_q that is a multiple of 8");
    TORCH_CHECK(num_heads % num_heads_k == 0,
                "Number of heads in key/value must divide number of heads in query");

    CHECK_SHAPE(q, batch_size, seqlen_q, num_heads, head_size_q);
    CHECK_SHAPE(k, batch_size, seqlen_k, num_heads_k, head_size_q);
    CHECK_SHAPE(v, batch_size, seqlen_k, num_heads_k, head_size_v);

    auto opts = q.options();
    at::Tensor out =
        torch::empty({batch_size, seqlen_q, num_heads, head_size_v}, opts.dtype(q_dtype));

    return {out};
}

} // namespace torch_itfs
} // namespace aiter
