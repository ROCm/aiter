#include "hip/hip_runtime.h"
#include "mha_common.h"
#include "py_itfs_common.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>

#include "ck_tile/core.hpp"
#include "ck_tile/core/utility/functional.hpp"
#include "ck_tile/ops/common.hpp"
#include "ck_tile/ops/fmha/block/block_attention_bias_enum.hpp"
#include "ck_tile/ops/fmha/block/block_dropout.hpp"
#include "ck_tile/ops/fmha/block/variants.hpp"
#include "ck_tile/ops/fmha/pipeline/block_fmha_pipeline_qx_ks_vs_custom_policy.hpp"
#include "ck_tile/ops/reduce/block/block_reduce.hpp"

#include <string>
#include <type_traits>
#include <utility>
#include <variant>

#include "fmha_fwd.hpp"
#include "mask.hpp"

#define ASM_MARKER(marker)               \
    __builtin_amdgcn_sched_barrier(0);   \
    asm volatile("; [POYENC] " #marker); \
    __builtin_amdgcn_sched_barrier(0);

namespace aiter {

struct BlockFmhaPipelineQRKSVSDefaultPolicy
    : ck_tile::BlockFmhaPipelineQXKSVSCustomPolicy</* QLoadOnce = */ true,
                                                   /* AsyncCopy = */ false,
                                                   /* NumPrefetchK = */ 1,
                                                   /* NumPrefetchV = */ 1>

{
    template <typename Problem>
    CK_TILE_HOST_DEVICE static constexpr auto MakeKDramTileDistribution()
    {
        using namespace ck_tile;

        using KDataType = remove_cvref_t<typename Problem::KDataType>;

        constexpr index_t NumWarpGroups = 2;

        // make distribution for a single warp-group and duplicate content in all groups
        constexpr index_t kBlockSize = Problem::kBlockSize / NumWarpGroups;
        constexpr index_t kNPerBlock = Problem::BlockFmhaShape::kN0;
        constexpr index_t kKPerBlock = Problem::BlockFmhaShape::kK0;

        constexpr index_t MaxVectorSize = 16 / sizeof(KDataType);
        constexpr index_t ElemPerThread = (kNPerBlock * kKPerBlock) / kBlockSize;

        constexpr index_t KPerThread     = ck_tile::min(MaxVectorSize, ElemPerThread);
        constexpr index_t KThreads       = kKPerBlock / KPerThread;
        constexpr index_t NThreadPerWarp = get_warp_size() / KThreads;
        constexpr index_t NumWarps       = kBlockSize / get_warp_size();

        constexpr index_t NPerThread = kNPerBlock / (NumWarps * NThreadPerWarp);

        // 2 warp-groups share the same data
        return make_static_tile_distribution(
            tile_distribution_encoding<sequence<NumWarpGroups>,
                                       tuple<sequence<NPerThread, NumWarps, NThreadPerWarp>,
                                             sequence<KThreads, KPerThread>>,
                                       tuple<sequence<0, 1>, sequence<1, 2>>,
                                       tuple<sequence<0, 1>, sequence<2, 0>>,
                                       sequence<1, 2>,
                                       sequence<0, 1>>{});
    }

    template <typename Problem>
    CK_TILE_DEVICE static constexpr auto MakeVDramTileDistribution()
    {
        using namespace ck_tile;

        using VLayout = remove_cvref_t<typename Problem::BlockFmhaShape::VLayout>;

        constexpr index_t NumWarpGroups = 2;

        // make distribution for a single warp-group and duplicate content in all groups
        constexpr index_t kBlockSize = Problem::kBlockSize / NumWarpGroups;
        constexpr index_t kNPerBlock = Problem::BlockFmhaShape::kN1;
        constexpr index_t kKPerBlock = Problem::BlockFmhaShape::kK1;

        static_assert(std::is_same_v<VLayout, ck_tile::tensor_layout::gemm::RowMajor>);

        constexpr index_t NPerThread = GetAlignmentV<Problem>();
        constexpr index_t NThreads   = kNPerBlock / NPerThread; // P

        constexpr index_t total_pixels = kNPerBlock * kKPerBlock / kBlockSize;
        static_assert(total_pixels % NPerThread == 0); // TODO: this is not always true?
        constexpr index_t KPerThread = total_pixels / NPerThread;
        constexpr index_t kKPack     = GetSmemKPackV<Problem>();
        static_assert(kKPack % KPerThread == 0);
        constexpr index_t K2 =
            kKPack / KPerThread; // TODO: this dimention could be outside single wave
        if constexpr(get_warp_size() % (K2 * NThreads) == 0)
        {
            constexpr index_t K1       = get_warp_size() / (K2 * NThreads);
            constexpr index_t NumWarps = kBlockSize / get_warp_size();
            static_assert(kKPerBlock == NumWarps * K1 * K2 * KPerThread);
            // 2 warp-groups share the same data
            return make_static_tile_distribution(
                tile_distribution_encoding<
                    sequence<NumWarpGroups>,
                    tuple<sequence<NThreads, NPerThread>, sequence<NumWarps, K1, K2, KPerThread>>,
                    tuple<sequence<0, 2>, sequence<2, 1, 2>>,
                    tuple<sequence<0, 0>, sequence<1, 0, 2>>,
                    sequence<2, 1>,
                    sequence<3, 1>>{});
        }
        else
        {
            constexpr index_t K1   = (K2 * NThreads) / get_warp_size();
            constexpr index_t K2_m = K2 / K1;
            constexpr index_t K0   = kBlockSize / get_warp_size() / K1;
            static_assert(kKPerBlock == K0 * K1 * K2_m * KPerThread);
            // 2 warp-groups share the same data
            return make_static_tile_distribution(
                tile_distribution_encoding<
                    sequence<NumWarpGroups>,
                    tuple<sequence<NThreads, NPerThread>, sequence<K0, K1, K2_m, KPerThread>>,
                    tuple<sequence<0, 2, 2>, sequence<1, 2>>,
                    tuple<sequence<0, 0, 1>, sequence<0, 2>>,
                    sequence<2, 1>,
                    sequence<3, 1>>{});
        }
    }

    // this function should match the MakeVDramTileDistribution()
    template <typename Problem>
    CK_TILE_HOST_DEVICE static constexpr auto MakeShuffledVRegBlockDescriptor()
    {
        using namespace ck_tile;

        // This descriptor only used when V layout is seqlen * hdim
        using VLayout = remove_cvref_t<typename Problem::BlockFmhaShape::VLayout>;
        static_assert(std::is_same_v<VLayout, ck_tile::tensor_layout::gemm::RowMajor>);

        constexpr index_t NumWarpGroups = 2;

        // make distribution for a single warp-group and duplicate content in all groups
        constexpr index_t kBlockSize = Problem::kBlockSize / 2;
        constexpr index_t kNPerBlock = Problem::BlockFmhaShape::kN1;
        constexpr index_t kKPerBlock = Problem::BlockFmhaShape::kK1;

        constexpr index_t NPerThread = GetAlignmentV<Problem>();
        constexpr index_t NThreads   = kNPerBlock / NPerThread;

        constexpr index_t total_pixels = kNPerBlock * kKPerBlock / kBlockSize;
        static_assert(total_pixels % NPerThread == 0); // TODO: this is not always true?
        constexpr index_t KPerThread = total_pixels / NPerThread;
        constexpr index_t kKPack     = GetSmemKPackV<Problem>();
        static_assert(kKPack % KPerThread == 0);
        constexpr index_t K2 =
            kKPack / KPerThread; // TODO: this dimention could be outside single wave
        if constexpr(get_warp_size() % (K2 * NThreads) == 0)
        {
            constexpr index_t K1       = get_warp_size() / (K2 * NThreads);
            constexpr index_t NumWarps = kBlockSize / get_warp_size();
            // 2 warp-groups share the same data
            return make_static_tile_distribution(
                tile_distribution_encoding<
                    sequence<NumWarpGroups>,
                    tuple<sequence<NThreads, NPerThread>, sequence<NumWarps, K1, K2, KPerThread>>,
                    tuple<sequence<0, 2>, sequence<2, 1, 2>>,
                    tuple<sequence<0, 0>, sequence<1, 0, 2>>,
                    sequence<1, 2>,
                    sequence<1, 3>>{});
        }
        else
        {
            constexpr index_t K1   = (K2 * NThreads) / get_warp_size();
            constexpr index_t K2_m = K2 / K1;
            constexpr index_t K0   = kBlockSize / get_warp_size() / K1;
            static_assert(kKPerBlock == K0 * K1 * K2_m * KPerThread);
            // 2 warp-groups share the same data
            return make_static_tile_distribution(
                tile_distribution_encoding<
                    sequence<NumWarpGroups>,
                    tuple<sequence<NThreads, NPerThread>, sequence<K0, K1, K2_m, KPerThread>>,
                    tuple<sequence<0, 2, 2>, sequence<1, 2>>,
                    tuple<sequence<0, 0, 1>, sequence<0, 2>>,
                    sequence<1, 2>,
                    sequence<1, 3>>{});
        }
    }

    template <typename Problem>
    CK_TILE_HOST_DEVICE static constexpr ck_tile::index_t GetSmemSize()
    {
        return 2 * GetSmemSizeKV<Problem>();
    }
};

// This pipeline is qkv all located in LDS
template <typename Problem_, typename Policy_ = BlockFmhaPipelineQRKSVSDefaultPolicy>
struct BlockFmhaPipelineQRKSVS
{
    using Problem               = ck_tile::remove_cvref_t<Problem_>;
    using Policy                = ck_tile::remove_cvref_t<Policy_>;
    using QDataType             = ck_tile::remove_cvref_t<typename Problem::QDataType>;
    using KDataType             = ck_tile::remove_cvref_t<typename Problem::KDataType>;
    using VDataType             = ck_tile::remove_cvref_t<typename Problem::VDataType>;
    using SaccDataType          = ck_tile::remove_cvref_t<typename Problem::SaccDataType>;
    using SMPLComputeDataType   = ck_tile::remove_cvref_t<typename Problem::SMPLComputeDataType>;
    using BiasDataType          = ck_tile::remove_cvref_t<typename Problem::BiasDataType>;
    using RandValOutputDataType = ck_tile::remove_cvref_t<typename Problem::RandValOutputDataType>;
    using LSEDataType           = ck_tile::remove_cvref_t<typename Problem::LSEDataType>;
    using PDataType             = ck_tile::remove_cvref_t<typename Problem::PDataType>;
    using OaccDataType          = ck_tile::remove_cvref_t<typename Problem::OaccDataType>;
    using ODataType             = ck_tile::remove_cvref_t<typename Problem::ODataType>;
    using AttentionVariant      = ck_tile::remove_cvref_t<typename Problem::AttentionVariant>;
    using FmhaMask              = ck_tile::remove_cvref_t<typename Problem::FmhaMask>;

    using BlockFmhaShape             = ck_tile::remove_cvref_t<typename Problem::BlockFmhaShape>;
    using VLayout                    = ck_tile::remove_cvref_t<typename BlockFmhaShape::VLayout>;
    static constexpr bool kQLoadOnce = true; // if q_tile load whole block length (hdim) at once
    static_assert(kQLoadOnce == Policy::QLoadOnce);

    static constexpr ck_tile::index_t kBlockSize = Problem::kBlockSize;

    static constexpr ck_tile::index_t kM0           = BlockFmhaShape::kM0;
    static constexpr ck_tile::index_t kN0           = BlockFmhaShape::kN0;
    static constexpr ck_tile::index_t kK0           = BlockFmhaShape::kK0;
    static constexpr ck_tile::index_t kN1           = BlockFmhaShape::kN1;
    static constexpr ck_tile::index_t kK1           = BlockFmhaShape::kK1;
    static constexpr ck_tile::index_t kQKHeaddim    = BlockFmhaShape::kQKHeaddim;
    static constexpr ck_tile::index_t kSubQKHeaddim = BlockFmhaShape::kSubQKHeaddim;

    static_assert(kSubQKHeaddim <= 256, "hdim bigger than 256 is not suitable for this pipeline!");

    static constexpr bool kIsGroupMode      = Problem::kIsGroupMode;
    static constexpr bool kPadSeqLenQ       = Problem::kPadSeqLenQ;
    static constexpr bool kPadSeqLenK       = Problem::kPadSeqLenK;
    static constexpr bool kPadHeadDimQ      = Problem::kPadHeadDimQ;
    static constexpr bool kPadHeadDimV      = Problem::kPadHeadDimV;
    static constexpr bool kHasLogitsSoftCap = Problem::kHasLogitsSoftCap;
    static constexpr auto BiasEnum          = Problem::BiasEnum;
    static constexpr bool kStoreLSE         = Problem::kStoreLSE;
    static constexpr bool kHasDropout       = Problem::kHasDropout;

    static_assert(!kHasLogitsSoftCap &&
                  Problem::BiasEnum == ck_tile::BlockAttentionBiasEnum::NO_BIAS && !kHasDropout);

    // last dimension vector length used to create tensor view(and decide buffer_load vector length)
    // ... together with tensor distribution. tensor dist should able to overwrite this
    static constexpr ck_tile::index_t kAlignmentQ =
        kPadHeadDimQ ? 1 : Policy::template GetAlignmentQ<Problem>();
    static constexpr ck_tile::index_t kAlignmentK =
        kPadHeadDimQ ? 1 : Policy::template GetAlignmentK<Problem>();
    static constexpr ck_tile::index_t kAlignmentV = []() {
        if constexpr(std::is_same_v<VLayout, ck_tile::tensor_layout::gemm::RowMajor>)
            return kPadHeadDimV ? 1 : Policy::template GetAlignmentV<Problem>();
        else
            return kPadSeqLenK ? 1 : Policy::template GetAlignmentV<Problem>();
    }();

    static constexpr ck_tile::index_t kAlignmentO =
        kPadHeadDimV ? 1 : Policy::template GetAlignmentO<Problem>();
    static constexpr ck_tile::index_t kAlignmentBias =
        kPadSeqLenK ? 1 : Policy::template GetAlignmentBias<Problem>();

    static constexpr ck_tile::index_t kBlockPerCu = []() {
        if constexpr(Problem::kBlockPerCu != -1)
            return Problem::kBlockPerCu;
        else
        {
            if constexpr(kQKHeaddim <= 32)
            {
                return 2;
            }
            else if constexpr(kQKHeaddim <= 64)
            {
                return 3;
            }
            else if constexpr(kQKHeaddim <= 128)
            {
                if constexpr(BiasEnum == ck_tile::BlockAttentionBiasEnum::ELEMENTWISE_BIAS)
                    return 1;
                else
                    return 2;
            }
            else if constexpr(kQKHeaddim <= 256)
            {
                return 1;
            }
            else
            {
                return 1;
            }
        }
    }();

    static constexpr const char* name = "qr";

    using DropoutType =
        std::conditional_t<kHasDropout, ck_tile::BlockDropout, ck_tile::NullBlockDropout>;

    CK_TILE_HOST_DEVICE static constexpr ck_tile::index_t GetSmemSize()
    {
        return Policy::template GetSmemSize<Problem>();
    }

    template <typename QDramBlockWindowTmp,
              typename KDramBlockWindowTmp,
              typename VDramBlockWindowTmp,
              typename BiasDramBlockWindowTmp,
              typename RandValDramBlockWindowTmp,
              typename LSEDramBlockWindowTmp,
              typename QElementFunction,
              typename KElementFunction,
              typename VElementFunction,
              typename BiasElementFunction,
              typename LSEElementFunction,
              typename SAccElementFunction,
              typename PComputeElementFunction,
              typename OAccElementFunction,
              typename PositionEncoding,
              typename AttentionVariantParams,
              typename BlockIndices>
    CK_TILE_HOST_DEVICE auto
    operator()(const QDramBlockWindowTmp& q_dram_block_window_tmp, // M0*K0 tile
               const QElementFunction& q_element_func,
               const KDramBlockWindowTmp& k_dram_block_window_tmp, // N0*K0 tile
               const KElementFunction& k_element_func,
               const VDramBlockWindowTmp& v_dram_block_window_tmp, // N1*K1 tile
               const VElementFunction& v_element_func,
               const BiasDramBlockWindowTmp& bias_dram_block_window_tmp, // M0*N0 tile
               const BiasElementFunction& bias_element_func,
               RandValDramBlockWindowTmp& randval_dram_block_window_tmp,
               LSEDramBlockWindowTmp& lse_dram_window_tmp, // M0*1 tile
               const LSEElementFunction& lse_element_func,
               const SAccElementFunction& s_acc_element_func,
               const PComputeElementFunction& p_compute_element_func,
               const OAccElementFunction& o_acc_element_func,
               FmhaMask mask,
               PositionEncoding position_encoding,
               float scale_s,
               const AttentionVariant& variant,
               const AttentionVariantParams& variant_params,
               const BlockIndices& block_indices,
               void* smem_ptr,
               DropoutType& dropout) const
    {
        using namespace ck_tile;

        static_assert(
            std::is_same_v<QDataType, remove_cvref_t<typename QDramBlockWindowTmp::DataType>> &&
                std::is_same_v<KDataType, remove_cvref_t<typename KDramBlockWindowTmp::DataType>> &&
                std::is_same_v<VDataType, remove_cvref_t<typename VDramBlockWindowTmp::DataType>>,
            "wrong!");

        static_assert(kM0 == QDramBlockWindowTmp{}.get_window_lengths()[number<0>{}] &&
                          kN0 == KDramBlockWindowTmp{}.get_window_lengths()[number<0>{}] &&
                          kK0 == KDramBlockWindowTmp{}.get_window_lengths()[number<1>{}] &&
                          kN1 == VDramBlockWindowTmp{}.get_window_lengths()[number<0>{}] &&
                          kK1 == VDramBlockWindowTmp{}.get_window_lengths()[number<1>{}] &&
                          kM0 == BiasDramBlockWindowTmp{}.get_window_lengths()[number<0>{}] &&
                          kN0 == BiasDramBlockWindowTmp{}.get_window_lengths()[number<1>{}],
                      "wrong!");

        const index_t warp_group_id = get_warp_id() / 4;

        // K tile in LDS
        const auto* k_lds_ptr = reinterpret_cast<const KDataType*>(smem_ptr);
        auto k_lds            = make_tensor_view<address_space_enum::lds>(
            k_lds_ptr, Policy::template MakeKLdsBlockDescriptor<Problem>());
        auto k_lds_window = make_tile_window(
            k_lds, Policy::template MakeKLdsBlockDescriptor<Problem>().get_lengths(), {0, 0});

        // V tile in LDS
        const auto* v_lds_ptr = reinterpret_cast<const VDataType*>(
            static_cast<char*>(smem_ptr) + Policy::template GetSmemSizeKV<Problem>());
        auto v_lds = make_tensor_view<address_space_enum::lds>(
            v_lds_ptr, Policy::template MakeVLdsBlockDescriptor<Problem>());
        auto v_lds_window = make_tile_window(
            v_lds, Policy::template MakeVLdsBlockDescriptor<Problem>().get_lengths(), {0, 0});

        // Block GEMM
        constexpr auto gemm_0 = Policy::template GetQKBlockGemm<Problem>();
        constexpr auto gemm_1 = Policy::template GetKVBlockGemm<Problem>();

        auto q_dram_window = make_tile_window(q_dram_block_window_tmp.get_bottom_tensor_view(),
                                              q_dram_block_window_tmp.get_window_lengths(),
                                              q_dram_block_window_tmp.get_window_origin(),
                                              Policy::template MakeQRegTileDistribution<Problem>());

        auto q = load_tile(q_dram_window);

        using SaccBlockTileType = decltype(gemm_0.MakeCBlockTile());
        auto s_acc              = SaccBlockTileType{};

        // reduction function for softmax
        const auto f_max = [](auto e0, auto e1) { return max(e0, e1); };
        const auto f_sum = [](auto e0, auto e1) { return e0 + e1; };

        // infer Sacc, S, P, M, L, Oacc type
        using SBlockTileType = decltype(cast_tile<SMPLComputeDataType>(s_acc));

        using MLBlockTileType = decltype(block_tile_reduce<SMPLComputeDataType>(
            SBlockTileType{}, sequence<1>{}, f_max, SMPLComputeDataType{0}));

        using OaccBlockTileType = decltype(gemm_1.MakeCBlockTile());

        // init Oacc, M, L
        auto o_acc = OaccBlockTileType{};
        auto m     = MLBlockTileType{};
        auto l     = MLBlockTileType{};

        clear_tile(o_acc);
        set_tile(m, -numeric<SMPLComputeDataType>::infinity());
        clear_tile(l);

        const auto q_origin = q_dram_window.get_window_origin();
        const auto [seqlen_k_start, seqlen_k_end] =
            mask.GetTileRangeAlongX(q_origin.at(number<0>{}), number<kM0>{}, number<kN0>{});

        const auto num_total_loop = integer_divide_ceil(seqlen_k_end - seqlen_k_start, kN0);

        // check early exit if no work to do
        if constexpr(FmhaMask::IsMasking || kPadSeqLenK)
        {
            if(num_total_loop <= 0)
            {
                if constexpr(kStoreLSE)
                {
                    auto lse =
                        make_static_distributed_tensor<LSEDataType>(m.get_tile_distribution());

                    set_tile(lse, -numeric<SMPLComputeDataType>::infinity());

                    store_tile(lse_dram_window_tmp, tile_elementwise_in(lse_element_func, lse));
                }

                // Note: here occ are all cleard, return it
                // Note: q loaded but no fence, ignore it.
                return o_acc;
            }
        }

        auto k_dram_block_window =
            make_tile_window(k_dram_block_window_tmp.get_bottom_tensor_view(),
                             k_dram_block_window_tmp.get_window_lengths(),
                             {seqlen_k_start, 0});

        const auto bias_origin = bias_dram_block_window_tmp.get_window_origin();
        auto bias_dram_window =
            make_tile_window(bias_dram_block_window_tmp.get_bottom_tensor_view(),
                             bias_dram_block_window_tmp.get_window_lengths(),
                             {bias_origin.at(number<0>{}), seqlen_k_start}, // M/N
                             Policy::template MakeBiasDramTileDistribution<decltype(gemm_0)>());

        auto randval_dram_window = dropout.template MakeRandvalDramWindow<decltype(gemm_0)>(
            randval_dram_block_window_tmp, seqlen_k_start);

        auto v_dram_window =
            make_tile_window(v_dram_block_window_tmp.get_bottom_tensor_view(),
                             v_dram_block_window_tmp.get_window_lengths(),
                             {0, seqlen_k_start}, // TODO: hdim split?
                             Policy::template MakeVDramTileDistribution<Problem>());

        auto q_tile = tile_elementwise_in(q_element_func, q);

        // prefetch K tile
        index_t i_total_loops      = 0;
        constexpr index_t k0_loops = kQKHeaddim / kK0;
        constexpr index_t k1_loops = kN0 / kK1;

#define ENABLE_PINGPONG_SCHED 1

#if ENABLE_PINGPONG_SCHED
        if(warp_group_id == 1)
        {
            __builtin_amdgcn_s_barrier();
        }
#endif

        static_assert(1 == k0_loops);
        static_assert(1 == k1_loops);
        do
        {
            clear_tile(s_acc); // initialize C

            // (1) load & store K =============================================
            auto k_dram_window = make_tile_window(
                k_dram_block_window, Policy::template MakeKDramTileDistribution<Problem>());
            auto k_block_tile = load_tile(k_dram_window); // global read i
            __builtin_amdgcn_sched_barrier(0);

            store_tile(k_lds_window, tile_elementwise_in(k_element_func, k_block_tile));

            __builtin_amdgcn_s_waitcnt(0xc07f);

            __builtin_amdgcn_sched_barrier(0);
            __builtin_amdgcn_s_barrier();
            __builtin_amdgcn_sched_barrier(0);
            // (2) mfma + softmax =============================================
            {
                gemm_0(s_acc,
                       get_slice_tile(q_tile,
                                      sequence<0, (k0_loops - 1) * kK0>{},
                                      sequence<kM0, k0_loops * kK0>{}),
                       k_lds_window);
            }

            // scale_s, mask, softmax
            {
                s_acc = tile_elementwise_in(s_acc_element_func, s_acc);
#if !CK_TILE_FMHA_FWD_FAST_EXP2
                tile_elementwise_inout([&scale_s](auto& x) { x = x * scale_s; }, s_acc);
#endif
            }

            if constexpr(kPadSeqLenK || FmhaMask::IsMasking)
            {
                const auto k_origin      = k_dram_block_window.get_window_origin();
                bool need_perpixel_check = mask.IsEdgeTile(q_origin.at(number<0>{}),
                                                           k_origin.at(number<0>{}),
                                                           number<kM0>{},
                                                           number<kN0>{});
                if(need_perpixel_check)
                {
                    set_tile_if(
                        s_acc, -numeric<SMPLComputeDataType>::infinity(), [&](auto tile_idx) {
                            const auto row = q_origin.at(number<0>{}) + tile_idx.at(number<0>{});
                            const auto col = k_origin.at(number<0>{}) + tile_idx.at(number<1>{});
                            return !variant.LogitsMask(variant_params,
                                                       block_indices.batch_idx,
                                                       row,
                                                       col,
                                                       block_indices.qo_head_idx,
                                                       block_indices.kv_head_idx);
                        });
                }
            }

            const auto s = cast_tile<SMPLComputeDataType>(s_acc); // S{j}
            auto m_local = block_tile_reduce<SMPLComputeDataType>(
                s,
                sequence<1>{},
                f_max,
                -numeric<SMPLComputeDataType>::infinity()); // m_local = rowmax(S{j})
            block_tile_reduce_sync(m_local, f_max, bool_constant<false>{});

            const auto m_old = m; // m{j-1}
            tile_elementwise_inout(
                [](auto& e0, auto e1, auto e2) { e0 = max(e1, e2); }, m, m_old, m_local); // m{j}

            auto p_compute = make_static_distributed_tensor<SMPLComputeDataType>(
                s.get_tile_distribution()); // Pcompute{j}

            static const auto get_validated_m = [](SMPLComputeDataType raw_m) {
                /// NOTICE: bias might be materialized mask including -inf values, need
                /// consideration
                {
                    return raw_m;
                }
            };

            constexpr auto p_spans = decltype(p_compute)::get_distributed_spans();
            sweep_tile_span(p_spans[number<0>{}], [&](auto idx0) {
                constexpr auto i_idx = make_tuple(idx0);
#if CK_TILE_FMHA_FWD_FAST_EXP2
                auto row_max = scale_s * get_validated_m(m[i_idx]);
#endif
                sweep_tile_span(p_spans[number<1>{}], [&](auto idx1) {
                    constexpr auto i_j_idx = make_tuple(idx0, idx1);
#if CK_TILE_FMHA_FWD_FAST_EXP2
                    p_compute(i_j_idx) = ck_tile::exp2(scale_s * s[i_j_idx] - row_max);
#else
                    p_compute(i_j_idx) = exp(s[i_j_idx] - get_validated_m(m[i_idx]));
#endif
                });
            });

            __builtin_amdgcn_sched_barrier(0);
            __builtin_amdgcn_s_barrier();
            __builtin_amdgcn_sched_barrier(0);
            // (3) load & store V =============================================
            const auto v_prefetch = load_tile(v_dram_window);
            __builtin_amdgcn_sched_barrier(0);

            if constexpr(std::is_same_v<VLayout, ck_tile::tensor_layout::gemm::RowMajor>)
            {
                auto v_shuffle_tmp = make_static_distributed_tensor<VDataType>(
                    Policy::template MakeShuffledVRegBlockDescriptor<Problem>());
                shuffle_tile(v_shuffle_tmp, v_prefetch);
                store_tile(
                    v_lds_window,
                    tile_elementwise_in(v_element_func, v_shuffle_tmp)); // store the prefetch
            }
            else
            {
                store_tile(v_lds_window,
                           tile_elementwise_in(v_element_func, v_prefetch)); // store the prefetch
            }
            move_tile_window(v_dram_window, {0, kK1});

            __builtin_amdgcn_s_waitcnt(0xc07f);

            __builtin_amdgcn_sched_barrier(0);
            __builtin_amdgcn_s_barrier();
            __builtin_amdgcn_sched_barrier(0);
            // (4) softmax + mfma =============================================
            auto rowsum_p = block_tile_reduce<SMPLComputeDataType>(
                p_compute, sequence<1>{}, f_sum, SMPLComputeDataType{0}); // rowsum(Pcompute{j})

            block_tile_reduce_sync(rowsum_p, f_sum, bool_constant<false>{});
            // l{j}, Oacc{j}
            constexpr auto o_spans = decltype(o_acc)::get_distributed_spans();
            sweep_tile_span(o_spans[number<0>{}], [&](auto idx0) {
                constexpr auto i_idx = make_tuple(idx0);
#if CK_TILE_FMHA_FWD_FAST_EXP2
                const auto tmp = [&]() {
                    auto row_max = scale_s * get_validated_m(m[i_idx]);
                    return ck_tile::exp2(scale_s * m_old[i_idx] - row_max);
                }();
#else
                const auto tmp       = exp(m_old[i_idx] - get_validated_m(m[i_idx]));
#endif
                l(i_idx) = tmp * l[i_idx] + rowsum_p[i_idx];
                sweep_tile_span(o_spans[number<1>{}], [&](auto idx1) {
                    constexpr auto i_j_idx = make_tuple(idx0, idx1);
                    // FIXME: this use different equation from FA v2 paper,
                    // but produce correc result.
                    // Is the equation wrong?
                    o_acc(i_j_idx) *= tmp;
                });
            });

            const auto p =
                cast_tile<PDataType>(tile_elementwise_in(p_compute_element_func, p_compute));

            {
                gemm_1(o_acc,
                       get_slice_tile(p, sequence<0, (k1_loops - 1) * kK1>{}, sequence<kM0, kN0>{}),
                       v_lds_window);
            }

            // move K tile windows
            move_tile_window(k_dram_block_window, {kN0, 0});

            __builtin_amdgcn_sched_barrier(0);
            __builtin_amdgcn_s_barrier();
            __builtin_amdgcn_sched_barrier(0);
        } while(++i_total_loops < num_total_loop);

#if ENABLE_PINGPONG_SCHED
        if(warp_group_id == 0)
        {
            __builtin_amdgcn_s_barrier();
        }
#endif

        // store lse
        if constexpr(kStoreLSE)
        {
            auto lse = make_static_distributed_tensor<LSEDataType>(m.get_tile_distribution());

            constexpr auto lse_spans = decltype(lse)::get_distributed_spans();
            sweep_tile_span(lse_spans[number<0>{}], [&, m_ = m, l_ = l](auto idx0) {
                constexpr auto i_idx = make_tuple(idx0);
#if CK_TILE_FMHA_FWD_FAST_EXP2
                lse(i_idx) = m_[i_idx] * scale_s / C_LOG2E + log(l_[i_idx]);
#else
                lse(i_idx) = m_[i_idx] + log(l_[i_idx]);
#endif
            });

            store_tile(lse_dram_window_tmp, tile_elementwise_in(lse_element_func, lse));
        }

        // finally, O
        constexpr auto o_spans = decltype(o_acc)::get_distributed_spans();

        sweep_tile_span(o_spans[number<0>{}], [&](auto idx0) {
            constexpr auto i_idx = make_tuple(idx0);
            const auto tmp       = [&]() {
                if constexpr(FmhaMask::IsMasking)
                {
                    return l[i_idx] == 0.f ? 0.f : 1 / l[i_idx];
                }
                else
                    return 1 / l[i_idx];
            }();
            sweep_tile_span(o_spans[number<1>{}], [&](auto idx1) {
                constexpr auto i_j_idx = make_tuple(idx0, idx1);
                o_acc(i_j_idx) *= tmp;
            });
        });

        o_acc = tile_elementwise_in(o_acc_element_func, o_acc);

        return o_acc;
    }

    template <typename QDramBlockWindowTmp,
              typename KDramBlockWindowTmp,
              typename VDramBlockWindowTmp,
              typename BiasDramBlockWindowTmp,
              typename RandValDramBlockWindowTmp,
              typename LSEDramBlockWindowTmp,
              typename PositionEncoding,
              typename AttentionVariantParams,
              typename BlockIndices>
    CK_TILE_HOST_DEVICE auto
    operator()(const QDramBlockWindowTmp& q_dram_block_window_tmp,       // M0*K0 tile
               const KDramBlockWindowTmp& k_dram_block_window_tmp,       // N0*K0 tile
               const VDramBlockWindowTmp& v_dram_block_window_tmp,       // N1*K1 tile
               const BiasDramBlockWindowTmp& bias_dram_block_window_tmp, // M0*N0 tile
               RandValDramBlockWindowTmp& randval_dram_block_window_tmp, // M0*N0 tile
               LSEDramBlockWindowTmp& lse_dram_block_window_tmp,         // M0*1 tile
               FmhaMask mask,
               PositionEncoding position_encoding,
               float scale_s,
               const AttentionVariant& variant,
               const AttentionVariantParams& variant_params,
               const BlockIndices& block_indices,
               void* smem_ptr,
               DropoutType& dropout) const
    {
        using namespace ck_tile;

        return operator()(q_dram_block_window_tmp,
                          identity{},
                          k_dram_block_window_tmp,
                          identity{},
                          v_dram_block_window_tmp,
                          identity{},
                          bias_dram_block_window_tmp,
                          identity{},
                          randval_dram_block_window_tmp,
                          lse_dram_block_window_tmp,
                          identity{},
                          identity{},
                          identity{},
                          identity{},
                          mask,
                          position_encoding,
                          scale_s,
                          variant,
                          variant_params,
                          block_indices,
                          smem_ptr,
                          dropout);
    }
};

template <typename FmhaPipeline_, typename EpiloguePipeline_>
struct FmhaFwdKernel
{
    using FmhaPipeline                            = ck_tile::remove_cvref_t<FmhaPipeline_>;
    using EpiloguePipeline                        = ck_tile::remove_cvref_t<EpiloguePipeline_>;
    static constexpr ck_tile::index_t kBlockSize  = FmhaPipeline::kBlockSize;
    static constexpr ck_tile::index_t kBlockPerCu = FmhaPipeline::kBlockPerCu;
    static_assert(kBlockPerCu > 0);
    static constexpr ck_tile::index_t kBlockPerCuInput = FmhaPipeline::Problem::kBlockPerCu;

    using QDataType    = ck_tile::remove_cvref_t<typename FmhaPipeline::QDataType>;
    using KDataType    = ck_tile::remove_cvref_t<typename FmhaPipeline::KDataType>;
    using VDataType    = ck_tile::remove_cvref_t<typename FmhaPipeline::VDataType>;
    using BiasDataType = ck_tile::remove_cvref_t<typename FmhaPipeline::BiasDataType>;
    using RandValOutputDataType =
        ck_tile::remove_cvref_t<typename FmhaPipeline::RandValOutputDataType>;
    using LSEDataType  = ck_tile::remove_cvref_t<typename FmhaPipeline::LSEDataType>;
    using ODataType    = ck_tile::remove_cvref_t<typename FmhaPipeline::ODataType>;
    using SaccDataType = ck_tile::remove_cvref_t<typename FmhaPipeline::SaccDataType>;

    using VLayout = ck_tile::remove_cvref_t<typename FmhaPipeline::VLayout>;

    static constexpr bool kIsGroupMode      = FmhaPipeline::kIsGroupMode;
    static constexpr bool kPadSeqLenQ       = FmhaPipeline::kPadSeqLenQ;
    static constexpr bool kPadSeqLenK       = FmhaPipeline::kPadSeqLenK;
    static constexpr bool kPadHeadDimQ      = FmhaPipeline::kPadHeadDimQ;
    static constexpr bool kPadHeadDimV      = FmhaPipeline::kPadHeadDimV;
    static constexpr bool kHasLogitsSoftCap = FmhaPipeline::kHasLogitsSoftCap;
    static constexpr auto BiasEnum          = FmhaPipeline::BiasEnum;
    static constexpr bool kStoreLSE         = FmhaPipeline::kStoreLSE;
    static constexpr bool kHasDropout       = FmhaPipeline::kHasDropout;
    static constexpr bool kDoFp8StaticQuant = FmhaPipeline::Problem::kDoFp8StaticQuant;
    static constexpr bool kSkipMinSeqlenQ   = FmhaPipeline::Problem::kSkipMinSeqlenQ;

    using AttentionVariant = ck_tile::remove_cvref_t<typename FmhaPipeline::AttentionVariant>;
    using FmhaMask         = ck_tile::remove_cvref_t<typename FmhaPipeline::FmhaMask>;
    static constexpr bool kHasMask = FmhaMask::IsMasking;

    static constexpr bool kUseAsyncCopy = FmhaPipeline::Policy::AsyncCopy;

    // clang-format off
    template <typename T> struct t2s;
    template <> struct t2s<float> { static constexpr const char * name = "fp32"; };
    template <> struct t2s<ck_tile::fp16_t> { static constexpr const char * name = "fp16"; };
    template <> struct t2s<ck_tile::bf16_t> { static constexpr const char * name = "bf16"; };
    template <> struct t2s<ck_tile::fp8_t> { static constexpr const char * name = "fp8"; };
    template <> struct t2s<ck_tile::bf8_t> { static constexpr const char * name = "bf8"; };
    // clang-format on

    CK_TILE_HOST static std::string GetName()
    {
        using namespace ck_tile;

        // sync with generate.py
        // clang-format off
        using bfs = typename FmhaPipeline::BlockFmhaShape;
        using g0br = typename bfs::Gemm0BlockWarps;
        using g1br = typename bfs::Gemm1BlockWarps;
        using g0wt = typename bfs::Gemm0WarpTile;
        using g1wt = typename bfs::Gemm1WarpTile;
        #define _SS_  std::string
        #define _TS_  std::to_string
        auto pn = [&] () {
            std::string n;
            if (kPadSeqLenQ) n += "s";
            if (kPadSeqLenK) n += "sk";
            if (kPadHeadDimQ) n += "d";
            if (kPadHeadDimV) n += "dv";
            return n.empty() ? n : std::string("p") + n; }();
        return
            _SS_("fmha_fwd_d") + _TS_(bfs::kQKHeaddim) + "_" + _SS_(t2s<QDataType>::name) +
            "_" + (kIsGroupMode ? "group" : "batch") + "_"
            "b" + _TS_(bfs::kM0) + "x" + _TS_(bfs::kN0) + "x" + _TS_(bfs::kK0) + "x" +
                    _TS_(bfs::kN1) + "x" + _TS_(bfs::kK1) + "x" + _TS_(bfs::kQKHeaddim) + "_" +
            "r" + _TS_(g0br::at(ck_tile::number<0>{})) + "x" + _TS_(g0br::at(ck_tile::number<1>{})) + "x" + _TS_(g0br::at(ck_tile::number<2>{})) + "_" +
            "r" + _TS_(g1br::at(ck_tile::number<0>{})) + "x" + _TS_(g1br::at(ck_tile::number<1>{})) + "x" + _TS_(g1br::at(ck_tile::number<2>{})) + "_" +
            "w" + _TS_(g0wt::at(ck_tile::number<0>{})) + "x" + _TS_(g0wt::at(ck_tile::number<1>{})) + "x" + _TS_(g0wt::at(ck_tile::number<2>{})) + "_" +
            "w" + _TS_(g1wt::at(ck_tile::number<0>{})) + "x" + _TS_(g1wt::at(ck_tile::number<1>{})) + "x" + _TS_(g1wt::at(ck_tile::number<2>{})) + "_" +
            (kBlockPerCuInput == -1 ? "" : ("o" + _TS_(kBlockPerCu) + "_")) + _SS_(FmhaPipeline::name) + "_" +
            "v" + (std::is_same_v<VLayout, ck_tile::tensor_layout::gemm::RowMajor> ? "r" : "c") + (pn.empty() ? "_npad" : "_" + pn) +
            (kHasLogitsSoftCap ? "_logits" : "_nlogits" ) + (BiasEnum == BlockAttentionBiasEnum::NO_BIAS ? _SS_("_nbias") : (_SS_("_") + BlockAttentionBiasEnumToStr<BiasEnum>::name)) +
            (kHasMask ? "_" + _SS_(FmhaMask::name) : "_nmask") + (kStoreLSE ? "_lse" : "_nlse" ) + (kHasDropout ? "_dropout" : "_ndropout" ) + (kDoFp8StaticQuant ? "_squant" : "_nsquant" );
        #undef _SS_
        #undef _TS_
        // clang-format on
    }

    template <ck_tile::index_t I> // to avoid duplicated base class prblem, introduce an template
                                  // arg
    struct FmhaFwdEmptyKargs
    {
    };

    // kargs use aggregate initializer, so no constructor will provided
    // use inheritance to minimize karg size
    // user need to use MakeKargs() function to create kargs.
    struct FmhaFwdCommonKargs
    {
        const void* q_ptr;
        const void* k_ptr;
        const void* v_ptr;
        void* o_ptr;

        ck_tile::index_t seqlen_q;
        ck_tile::index_t seqlen_k;
        ck_tile::index_t hdim_q;
        ck_tile::index_t hdim_v;

        ck_tile::index_t num_head_q;
        // for MQA/GQA, nhead could be different. This parameter is nhead_q / nhead_k
        // if this param is larger than 1, indicate MQA/GQA case
        ck_tile::index_t nhead_ratio_qk;
        float scale_s;

        ck_tile::index_t stride_q;
        ck_tile::index_t stride_k;
        ck_tile::index_t stride_v;
        ck_tile::index_t stride_o;

        ck_tile::index_t nhead_stride_q;
        ck_tile::index_t nhead_stride_k;
        ck_tile::index_t nhead_stride_v;
        ck_tile::index_t nhead_stride_o;
    };

    struct FmhaFwdLogitsSoftCapKargs
    {
        FmhaFwdLogitsSoftCapKargs() = default;

        void init_logits_soft_cap(float logits_soft_cap_)
        {
            if(0 < logits_soft_cap_)
            {
                logits_soft_cap     = logits_soft_cap_;
                logits_soft_cap_rcp = 1.f / logits_soft_cap;
            }
            else
            {
                logits_soft_cap     = 0.f;
                logits_soft_cap_rcp = 0.f;
            }
        }

        float logits_soft_cap;
        float logits_soft_cap_rcp;
    };

    struct FmhaFwdCommonBiasKargs
    {
        const void* bias_ptr               = nullptr;
        ck_tile::index_t stride_bias       = 0;
        ck_tile::index_t nhead_stride_bias = 0;
    };

    struct FmhaFwdBatchModeBiasKargs : FmhaFwdCommonBiasKargs
    {
        ck_tile::index_t batch_stride_bias = 0;
    };

    struct FmhaFwdAlibiKargs
    {
        // alibi is batch*nhead*1, no matter in batch/group mode, they are the same
        const void* alibi_slope_ptr;
        ck_tile::index_t alibi_slope_stride; // stride in batch, or 0 for all batch share same slope
    };

    struct FmhaFwdMaskKargs
    {
        // ck_tile::index_t window_size_left, window_size_right;
        ck_tile::index_t window_size_left, window_size_right;
        ck_tile::GenericAttentionMaskEnum mask_type;
    };

    struct FmhaFwdFp8StaticQuantKargs
    {
        float scale_p;
        float scale_o;
    };

    struct FmhaFwdCommonLSEKargs
    {
        void* lse_ptr                     = nullptr;
        ck_tile::index_t nhead_stride_lse = 0;
        ck_tile::index_t batch_stride_lse = 0;
    };

    struct FmhaFwdDropoutSeedOffset
    {
        template <typename T>
        union ValueOrPointer
        {
            T val;
            const T* ptr;
        };

        ValueOrPointer<uint64_t> drop_seed;
        ValueOrPointer<uint64_t> drop_offset;
        bool is_drop_seed_offset_from_host;
    };

    struct FmhaFwdCommonDropoutKargs : FmhaFwdDropoutSeedOffset
    {
        void init_dropout(float p_drop, uint64_t seed, uint64_t offset)
        {
            float p_undrop = 1.0 - p_drop;
            p_undrop_in_uint8_t =
                uint8_t(std::floor(p_undrop * std::numeric_limits<uint8_t>::max()));
            rp_undrop = 1.0 / p_undrop;

            this->drop_seed.val                 = seed;
            this->drop_offset.val               = offset;
            this->is_drop_seed_offset_from_host = true;
        }

        void init_dropout(float p_drop, const uint64_t* seed_ptr, const uint64_t* offset_ptr)
        {
            float p_undrop = 1.0 - p_drop;
            p_undrop_in_uint8_t =
                uint8_t(std::floor(p_undrop * std::numeric_limits<uint8_t>::max()));
            rp_undrop = 1.0 / p_undrop;

            this->drop_seed.ptr                 = seed_ptr;
            this->drop_offset.ptr               = offset_ptr;
            this->is_drop_seed_offset_from_host = false;
        }

        float rp_undrop             = 1;
        uint8_t p_undrop_in_uint8_t = std::numeric_limits<uint8_t>::max();
        bool is_store_randval       = false;
        void* rand_val_ptr          = nullptr;

        ck_tile::index_t stride_randval       = 0;
        ck_tile::index_t nhead_stride_randval = 0;
    };

    struct FmhaFwdBatchModeDropoutKargs : FmhaFwdCommonDropoutKargs
    {
        ck_tile::index_t batch_stride_randval = 0;
    };

    struct FmhaFwdSkipMinSeqlenQKargs
    {
        ck_tile::index_t min_seqlen_q = 0;
    };

    struct FmhaFwdBatchModeKargs
        : FmhaFwdCommonKargs,
          std::conditional_t<BiasEnum == ck_tile::BlockAttentionBiasEnum::ELEMENTWISE_BIAS,
                             FmhaFwdBatchModeBiasKargs,
                             std::conditional_t<BiasEnum == ck_tile::BlockAttentionBiasEnum::ALIBI,
                                                FmhaFwdAlibiKargs,
                                                FmhaFwdEmptyKargs<0>>>,
          std::conditional_t<kHasMask, FmhaFwdMaskKargs, FmhaFwdEmptyKargs<1>>,
          std::conditional_t<kStoreLSE, FmhaFwdCommonLSEKargs, FmhaFwdEmptyKargs<2>>,
          std::conditional_t<kDoFp8StaticQuant, FmhaFwdFp8StaticQuantKargs, FmhaFwdEmptyKargs<3>>,
          std::conditional_t<kHasDropout, FmhaFwdBatchModeDropoutKargs, FmhaFwdEmptyKargs<4>>,
          std::conditional_t<kHasLogitsSoftCap, FmhaFwdLogitsSoftCapKargs, FmhaFwdEmptyKargs<5>>
    {
        ck_tile::index_t batch_stride_q;
        ck_tile::index_t batch_stride_k;
        ck_tile::index_t batch_stride_v;
        ck_tile::index_t batch_stride_o;
    };

    struct FmhaFwdGroupModeKargs
        : FmhaFwdCommonKargs,
          std::conditional_t<BiasEnum == ck_tile::BlockAttentionBiasEnum::ELEMENTWISE_BIAS,
                             FmhaFwdCommonBiasKargs,
                             std::conditional_t<BiasEnum == ck_tile::BlockAttentionBiasEnum::ALIBI,
                                                FmhaFwdAlibiKargs,
                                                FmhaFwdEmptyKargs<0>>>,
          std::conditional_t<kHasMask, FmhaFwdMaskKargs, FmhaFwdEmptyKargs<1>>,
          std::conditional_t<kStoreLSE, FmhaFwdCommonLSEKargs, FmhaFwdEmptyKargs<2>>,
          std::conditional_t<kDoFp8StaticQuant, FmhaFwdFp8StaticQuantKargs, FmhaFwdEmptyKargs<3>>,
          std::conditional_t<kHasDropout, FmhaFwdCommonDropoutKargs, FmhaFwdEmptyKargs<4>>,
          std::conditional_t<kHasLogitsSoftCap, FmhaFwdLogitsSoftCapKargs, FmhaFwdEmptyKargs<5>>,
          std::conditional_t<kSkipMinSeqlenQ, FmhaFwdSkipMinSeqlenQKargs, FmhaFwdEmptyKargs<6>>
    {
        const int32_t* seqstart_q_ptr;
        const int32_t* seqstart_k_ptr;
        const int32_t* seqlen_k_ptr;
    };

    using Kargs = std::conditional_t<kIsGroupMode, FmhaFwdGroupModeKargs, FmhaFwdBatchModeKargs>;

    struct BlockIndices
    {
        ck_tile::index_t batch_idx;
        ck_tile::index_t qo_head_idx;
        ck_tile::index_t kv_head_idx;
    };

    template <bool Cond = !kIsGroupMode>
    CK_TILE_HOST static constexpr std::enable_if_t<Cond, Kargs>
    MakeKargsImpl(const void* q_ptr,
                  const void* k_ptr,
                  const void* v_ptr,
                  const void* bias_ptr,
                  void* rand_val_ptr,
                  void* lse_ptr,
                  void* o_ptr,
                  ck_tile::index_t seqlen_q,
                  ck_tile::index_t seqlen_k,
                  ck_tile::index_t hdim_q,
                  ck_tile::index_t hdim_v,
                  ck_tile::index_t num_head_q,
                  ck_tile::index_t nhead_ratio_qk,
                  float scale_s,
                  float scale_p,
                  float scale_o,
                  float logits_soft_cap,
                  ck_tile::index_t stride_q,
                  ck_tile::index_t stride_k,
                  ck_tile::index_t stride_v,
                  ck_tile::index_t stride_bias,
                  ck_tile::index_t stride_randval,
                  ck_tile::index_t stride_o,
                  ck_tile::index_t nhead_stride_q,
                  ck_tile::index_t nhead_stride_k,
                  ck_tile::index_t nhead_stride_v,
                  ck_tile::index_t nhead_stride_bias,
                  ck_tile::index_t nhead_stride_randval,
                  ck_tile::index_t nhead_stride_lse,
                  ck_tile::index_t nhead_stride_o,
                  ck_tile::index_t batch_stride_q,
                  ck_tile::index_t batch_stride_k,
                  ck_tile::index_t batch_stride_v,
                  ck_tile::index_t batch_stride_bias,
                  ck_tile::index_t batch_stride_randval,
                  ck_tile::index_t batch_stride_lse,
                  ck_tile::index_t batch_stride_o,
                  ck_tile::index_t window_size_left,
                  ck_tile::index_t window_size_right,
                  ck_tile::index_t mask_type,
                  float p_drop,
                  bool s_randval,
                  std::variant<std::pair<uint64_t, uint64_t>, std::pair<const void*, const void*>>
                      drop_seed_offset)
    {
        Kargs kargs{{q_ptr,
                     k_ptr,
                     v_ptr,
                     o_ptr,
                     seqlen_q,
                     seqlen_k,
                     hdim_q,
                     hdim_v,
                     num_head_q,
                     nhead_ratio_qk,
#if CK_TILE_FMHA_FWD_FAST_EXP2
                     static_cast<float>(scale_s * ck_tile::log2e_v<>),
#else
                     scale_s,
#endif
                     stride_q,
                     stride_k,
                     stride_v,
                     stride_o,
                     nhead_stride_q,
                     nhead_stride_k,
                     nhead_stride_v,
                     nhead_stride_o}, // args for common karg
                    {},               // placeholder for bias
                    {},               // placeholder for mask
                    {},               // placeholder for lse
                    {},               // placeholder for fp8_static_quant args
                    {},               // placeholder for dropout
                    {},               // placeholder for logits_soft_cap
                    batch_stride_q,
                    batch_stride_k,
                    batch_stride_v,
                    batch_stride_o};

        if constexpr(BiasEnum == ck_tile::BlockAttentionBiasEnum::ELEMENTWISE_BIAS)
        {
            kargs.bias_ptr          = bias_ptr;
            kargs.stride_bias       = stride_bias;
            kargs.nhead_stride_bias = nhead_stride_bias;
            kargs.batch_stride_bias = batch_stride_bias;
        }
        else if constexpr(BiasEnum == ck_tile::BlockAttentionBiasEnum::ALIBI)
        {
            kargs.alibi_slope_ptr    = bias_ptr;
            kargs.alibi_slope_stride = stride_bias;
        }
        if constexpr(kHasMask)
        {
            kargs.window_size_left  = window_size_left;
            kargs.window_size_right = window_size_right;
            kargs.mask_type         = static_cast<ck_tile::GenericAttentionMaskEnum>(mask_type);
        }
        if constexpr(kStoreLSE)
        {
            kargs.lse_ptr          = lse_ptr;
            kargs.nhead_stride_lse = nhead_stride_lse;
            kargs.batch_stride_lse = batch_stride_lse;
        }
        if constexpr(kDoFp8StaticQuant)
        {
            kargs.scale_p = scale_p;
            kargs.scale_o = scale_o;
        }
        if constexpr(kHasDropout)
        {
            if(drop_seed_offset.index() == 0) // seed & offset come from host
            {
                const auto& [seed, offset] = std::get<0>(drop_seed_offset);
                kargs.init_dropout(p_drop, seed, offset);
            }
            else // seed & offset come from device
            {
                const auto& [seed_ptr, offset_ptr] = std::get<1>(drop_seed_offset);
                kargs.init_dropout(p_drop,
                                   reinterpret_cast<const uint64_t*>(seed_ptr),
                                   reinterpret_cast<const uint64_t*>(offset_ptr));
            }

            kargs.rand_val_ptr         = rand_val_ptr;
            kargs.stride_randval       = stride_randval;
            kargs.nhead_stride_randval = nhead_stride_randval;
            kargs.batch_stride_randval = batch_stride_randval;
            kargs.is_store_randval     = s_randval;
        }
        if constexpr(kHasLogitsSoftCap)
        {
            kargs.init_logits_soft_cap(logits_soft_cap);
        }

        return kargs;
    }

    // std::variant<> can't take in a list initializer, overload for backward compatibility
    template <bool Cond = !kIsGroupMode>
    CK_TILE_HOST static constexpr std::enable_if_t<Cond, Kargs>
    MakeKargs(const void* q_ptr,
              const void* k_ptr,
              const void* v_ptr,
              const void* bias_ptr,
              void* rand_val_ptr,
              void* lse_ptr,
              void* o_ptr,
              ck_tile::index_t seqlen_q,
              ck_tile::index_t seqlen_k,
              ck_tile::index_t hdim_q,
              ck_tile::index_t hdim_v,
              ck_tile::index_t num_head_q,
              ck_tile::index_t nhead_ratio_qk,
              float scale_s,
              float scale_p,
              float scale_o,
              float logits_soft_cap,
              ck_tile::index_t stride_q,
              ck_tile::index_t stride_k,
              ck_tile::index_t stride_v,
              ck_tile::index_t stride_bias,
              ck_tile::index_t stride_randval,
              ck_tile::index_t stride_o,
              ck_tile::index_t nhead_stride_q,
              ck_tile::index_t nhead_stride_k,
              ck_tile::index_t nhead_stride_v,
              ck_tile::index_t nhead_stride_bias,
              ck_tile::index_t nhead_stride_randval,
              ck_tile::index_t nhead_stride_lse,
              ck_tile::index_t nhead_stride_o,
              ck_tile::index_t batch_stride_q,
              ck_tile::index_t batch_stride_k,
              ck_tile::index_t batch_stride_v,
              ck_tile::index_t batch_stride_bias,
              ck_tile::index_t batch_stride_randval,
              ck_tile::index_t batch_stride_lse,
              ck_tile::index_t batch_stride_o,
              ck_tile::index_t window_size_left,
              ck_tile::index_t window_size_right,
              ck_tile::index_t mask_type,
              float p_drop,
              bool s_randval,
              const std::tuple<uint64_t, uint64_t>& drop_seed_offset)
    {
        return MakeKargsImpl(
            q_ptr,
            k_ptr,
            v_ptr,
            bias_ptr,
            rand_val_ptr,
            lse_ptr,
            o_ptr,
            seqlen_q,
            seqlen_k,
            hdim_q,
            hdim_v,
            num_head_q,
            nhead_ratio_qk,
            scale_s,
            scale_p,
            scale_o,
            logits_soft_cap,
            stride_q,
            stride_k,
            stride_v,
            stride_bias,
            stride_randval,
            stride_o,
            nhead_stride_q,
            nhead_stride_k,
            nhead_stride_v,
            nhead_stride_bias,
            nhead_stride_randval,
            nhead_stride_lse,
            nhead_stride_o,
            batch_stride_q,
            batch_stride_k,
            batch_stride_v,
            batch_stride_bias,
            batch_stride_randval,
            batch_stride_lse,
            batch_stride_o,
            window_size_left,
            window_size_right,
            mask_type,
            p_drop,
            s_randval,
            std::make_pair(std::get<0>(drop_seed_offset), std::get<1>(drop_seed_offset)));
    }

    // std::variant<> can't take in a list initializer, overload for backward compatibility
    template <bool Cond = !kIsGroupMode>
    CK_TILE_HOST static constexpr std::enable_if_t<Cond, Kargs>
    MakeKargs(const void* q_ptr,
              const void* k_ptr,
              const void* v_ptr,
              const void* bias_ptr,
              void* rand_val_ptr,
              void* lse_ptr,
              void* o_ptr,
              ck_tile::index_t seqlen_q,
              ck_tile::index_t seqlen_k,
              ck_tile::index_t hdim_q,
              ck_tile::index_t hdim_v,
              ck_tile::index_t num_head_q,
              ck_tile::index_t nhead_ratio_qk,
              float scale_s,
              float scale_p,
              float scale_o,
              float logits_soft_cap,
              ck_tile::index_t stride_q,
              ck_tile::index_t stride_k,
              ck_tile::index_t stride_v,
              ck_tile::index_t stride_bias,
              ck_tile::index_t stride_randval,
              ck_tile::index_t stride_o,
              ck_tile::index_t nhead_stride_q,
              ck_tile::index_t nhead_stride_k,
              ck_tile::index_t nhead_stride_v,
              ck_tile::index_t nhead_stride_bias,
              ck_tile::index_t nhead_stride_randval,
              ck_tile::index_t nhead_stride_lse,
              ck_tile::index_t nhead_stride_o,
              ck_tile::index_t batch_stride_q,
              ck_tile::index_t batch_stride_k,
              ck_tile::index_t batch_stride_v,
              ck_tile::index_t batch_stride_bias,
              ck_tile::index_t batch_stride_randval,
              ck_tile::index_t batch_stride_lse,
              ck_tile::index_t batch_stride_o,
              ck_tile::index_t window_size_left,
              ck_tile::index_t window_size_right,
              ck_tile::index_t mask_type,
              float p_drop,
              bool s_randval,
              const std::tuple<const void*, const void*>& drop_seed_offset)
    {
        return MakeKargsImpl(
            q_ptr,
            k_ptr,
            v_ptr,
            bias_ptr,
            rand_val_ptr,
            lse_ptr,
            o_ptr,
            seqlen_q,
            seqlen_k,
            hdim_q,
            hdim_v,
            num_head_q,
            nhead_ratio_qk,
            scale_s,
            scale_p,
            scale_o,
            logits_soft_cap,
            stride_q,
            stride_k,
            stride_v,
            stride_bias,
            stride_randval,
            stride_o,
            nhead_stride_q,
            nhead_stride_k,
            nhead_stride_v,
            nhead_stride_bias,
            nhead_stride_randval,
            nhead_stride_lse,
            nhead_stride_o,
            batch_stride_q,
            batch_stride_k,
            batch_stride_v,
            batch_stride_bias,
            batch_stride_randval,
            batch_stride_lse,
            batch_stride_o,
            window_size_left,
            window_size_right,
            mask_type,
            p_drop,
            s_randval,
            std::make_pair(std::get<0>(drop_seed_offset), std::get<1>(drop_seed_offset)));
    }

    template <bool Cond = kIsGroupMode>
    CK_TILE_HOST static constexpr std::enable_if_t<Cond, Kargs>
    MakeKargsImpl(const void* q_ptr,
                  const void* k_ptr,
                  const void* v_ptr,
                  const void* bias_ptr,
                  void* rand_val_ptr,
                  void* lse_ptr,
                  void* o_ptr,
                  const void* seqstart_q_ptr,
                  const void* seqstart_k_ptr,
                  const void* seqlen_k_ptr,
                  ck_tile::index_t hdim_q,
                  ck_tile::index_t hdim_v,
                  ck_tile::index_t num_head_q,
                  ck_tile::index_t nhead_ratio_qk,
                  float scale_s,
                  float scale_p,
                  float scale_o,
                  float logits_soft_cap,
                  ck_tile::index_t stride_q,
                  ck_tile::index_t stride_k,
                  ck_tile::index_t stride_v,
                  ck_tile::index_t stride_bias,
                  ck_tile::index_t stride_randval,
                  ck_tile::index_t stride_o,
                  ck_tile::index_t nhead_stride_q,
                  ck_tile::index_t nhead_stride_k,
                  ck_tile::index_t nhead_stride_v,
                  ck_tile::index_t nhead_stride_bias,
                  ck_tile::index_t nhead_stride_randval,
                  ck_tile::index_t nhead_stride_lse,
                  ck_tile::index_t nhead_stride_o,
                  ck_tile::index_t window_size_left,
                  ck_tile::index_t window_size_right,
                  ck_tile::index_t mask_type,
                  ck_tile::index_t min_seqlen_q,
                  float p_drop,
                  bool s_randval,
                  std::variant<std::pair<uint64_t, uint64_t>, std::pair<const void*, const void*>>
                      drop_seed_offset)
    {
        Kargs kargs{{q_ptr,
                     k_ptr,
                     v_ptr,
                     o_ptr,
                     -1, // seqlen will be updated by another pointer
                     -1, //
                     hdim_q,
                     hdim_v,
                     num_head_q,
                     nhead_ratio_qk,
#if CK_TILE_FMHA_FWD_FAST_EXP2
                     static_cast<float>(scale_s * ck_tile::log2e_v<>),
#else
                     scale_s,
#endif
                     stride_q,
                     stride_k,
                     stride_v,
                     stride_o,
                     nhead_stride_q,
                     nhead_stride_k,
                     nhead_stride_v,
                     nhead_stride_o}, // args for common karg
                    {},               // placeholder for bias
                    {},               // placeholder for mask
                    {},               // placeholder for lse
                    {},               // placeholder for fp8_static_quant args
                    {},               // placeholder for dropout
                    {},               // placeholder for logits_soft_cap
                    {},               // placeholder for min_seqlen_q
                    reinterpret_cast<const int32_t*>(seqstart_q_ptr),
                    reinterpret_cast<const int32_t*>(seqstart_k_ptr),
                    reinterpret_cast<const int32_t*>(seqlen_k_ptr)};

        if constexpr(BiasEnum == ck_tile::BlockAttentionBiasEnum::ELEMENTWISE_BIAS)
        {
            kargs.bias_ptr          = bias_ptr;
            kargs.stride_bias       = stride_bias;
            kargs.nhead_stride_bias = nhead_stride_bias;
        }
        else if constexpr(BiasEnum == ck_tile::BlockAttentionBiasEnum::ALIBI)
        {
            kargs.alibi_slope_ptr    = bias_ptr;
            kargs.alibi_slope_stride = stride_bias;
        }
        if constexpr(kHasMask)
        {
            kargs.window_size_left  = window_size_left;
            kargs.window_size_right = window_size_right;
            kargs.mask_type         = static_cast<ck_tile::GenericAttentionMaskEnum>(mask_type);
        }
        if constexpr(kStoreLSE)
        {
            kargs.lse_ptr          = lse_ptr;
            kargs.nhead_stride_lse = nhead_stride_lse;
        }
        if constexpr(kDoFp8StaticQuant)
        {
            kargs.scale_p = scale_p;
            kargs.scale_o = scale_o;
        }
        if constexpr(kHasDropout)
        {
            if(drop_seed_offset.index() == 0) // seed & offset come from host
            {
                const auto& [seed, offset] = std::get<0>(drop_seed_offset);
                kargs.init_dropout(p_drop, seed, offset);
            }
            else // seed & offset come from device
            {
                const auto& [seed_ptr, offset_ptr] = std::get<1>(drop_seed_offset);
                kargs.init_dropout(p_drop,
                                   reinterpret_cast<const uint64_t*>(seed_ptr),
                                   reinterpret_cast<const uint64_t*>(offset_ptr));
            }

            kargs.rand_val_ptr         = rand_val_ptr;
            kargs.stride_randval       = stride_randval;
            kargs.nhead_stride_randval = nhead_stride_randval;
            kargs.is_store_randval     = s_randval;
        }
        if constexpr(kHasLogitsSoftCap)
        {
            kargs.init_logits_soft_cap(logits_soft_cap);
        }
        if constexpr(kSkipMinSeqlenQ)
        {
            kargs.min_seqlen_q = min_seqlen_q;
        }

        return kargs;
    }

    // std::variant<> can't take in a list initializer, overload for backward compatibility
    template <bool Cond = kIsGroupMode>
    CK_TILE_HOST static constexpr std::enable_if_t<Cond, Kargs>
    MakeKargs(const void* q_ptr,
              const void* k_ptr,
              const void* v_ptr,
              const void* bias_ptr,
              void* rand_val_ptr,
              void* lse_ptr,
              void* o_ptr,
              const void* seqstart_q_ptr,
              const void* seqstart_k_ptr,
              const void* seqlen_k_ptr,
              ck_tile::index_t hdim_q,
              ck_tile::index_t hdim_v,
              ck_tile::index_t num_head_q,
              ck_tile::index_t nhead_ratio_qk,
              float scale_s,
              float scale_p,
              float scale_o,
              float logits_soft_cap,
              ck_tile::index_t stride_q,
              ck_tile::index_t stride_k,
              ck_tile::index_t stride_v,
              ck_tile::index_t stride_bias,
              ck_tile::index_t stride_randval,
              ck_tile::index_t stride_o,
              ck_tile::index_t nhead_stride_q,
              ck_tile::index_t nhead_stride_k,
              ck_tile::index_t nhead_stride_v,
              ck_tile::index_t nhead_stride_bias,
              ck_tile::index_t nhead_stride_randval,
              ck_tile::index_t nhead_stride_lse,
              ck_tile::index_t nhead_stride_o,
              ck_tile::index_t window_size_left,
              ck_tile::index_t window_size_right,
              ck_tile::index_t mask_type,
              float p_drop,
              bool s_randval,
              const std::tuple<uint64_t, uint64_t>& drop_seed_offset)
    {
        return MakeKargsImpl(
            q_ptr,
            k_ptr,
            v_ptr,
            bias_ptr,
            rand_val_ptr,
            lse_ptr,
            o_ptr,
            seqstart_q_ptr,
            seqstart_k_ptr,
            seqlen_k_ptr,
            hdim_q,
            hdim_v,
            num_head_q,
            nhead_ratio_qk,
            scale_s,
            scale_p,
            scale_o,
            logits_soft_cap,
            stride_q,
            stride_k,
            stride_v,
            stride_bias,
            stride_randval,
            stride_o,
            nhead_stride_q,
            nhead_stride_k,
            nhead_stride_v,
            nhead_stride_bias,
            nhead_stride_randval,
            nhead_stride_lse,
            nhead_stride_o,
            window_size_left,
            window_size_right,
            mask_type,
            p_drop,
            s_randval,
            std::make_pair(std::get<0>(drop_seed_offset), std::get<1>(drop_seed_offset)));
    }

    // std::variant<> can't take in a list initializer, overload for backward compatibility
    template <bool Cond = kIsGroupMode>
    CK_TILE_HOST static constexpr std::enable_if_t<Cond, Kargs>
    MakeKargs(const void* q_ptr,
              const void* k_ptr,
              const void* v_ptr,
              const void* bias_ptr,
              void* rand_val_ptr,
              void* lse_ptr,
              void* o_ptr,
              const void* seqstart_q_ptr,
              const void* seqstart_k_ptr,
              const void* seqlen_k_ptr,
              ck_tile::index_t hdim_q,
              ck_tile::index_t hdim_v,
              ck_tile::index_t num_head_q,
              ck_tile::index_t nhead_ratio_qk,
              float scale_s,
              float scale_p,
              float scale_o,
              float logits_soft_cap,
              ck_tile::index_t stride_q,
              ck_tile::index_t stride_k,
              ck_tile::index_t stride_v,
              ck_tile::index_t stride_bias,
              ck_tile::index_t stride_randval,
              ck_tile::index_t stride_o,
              ck_tile::index_t nhead_stride_q,
              ck_tile::index_t nhead_stride_k,
              ck_tile::index_t nhead_stride_v,
              ck_tile::index_t nhead_stride_bias,
              ck_tile::index_t nhead_stride_randval,
              ck_tile::index_t nhead_stride_lse,
              ck_tile::index_t nhead_stride_o,
              ck_tile::index_t window_size_left,
              ck_tile::index_t window_size_right,
              ck_tile::index_t mask_type,
              float p_drop,
              bool s_randval,
              const std::tuple<const void*, const void*>& drop_seed_offset)
    {
        return MakeKargsImpl(
            q_ptr,
            k_ptr,
            v_ptr,
            bias_ptr,
            rand_val_ptr,
            lse_ptr,
            o_ptr,
            seqstart_q_ptr,
            seqstart_k_ptr,
            seqlen_k_ptr,
            hdim_q,
            hdim_v,
            num_head_q,
            nhead_ratio_qk,
            scale_s,
            scale_p,
            scale_o,
            logits_soft_cap,
            stride_q,
            stride_k,
            stride_v,
            stride_bias,
            stride_randval,
            stride_o,
            nhead_stride_q,
            nhead_stride_k,
            nhead_stride_v,
            nhead_stride_bias,
            nhead_stride_randval,
            nhead_stride_lse,
            nhead_stride_o,
            window_size_left,
            window_size_right,
            mask_type,
            p_drop,
            s_randval,
            std::make_pair(std::get<0>(drop_seed_offset), std::get<1>(drop_seed_offset)));
    }

    CK_TILE_HOST static constexpr auto GridSize(ck_tile::index_t batch_size_,
                                                ck_tile::index_t nhead_,
                                                ck_tile::index_t seqlen_q_,
                                                ck_tile::index_t hdim_v_,
                                                bool has_padded_seqlen_k = false)
    {
        // has_padded_seqlen_k is determined by checking (seqlen_k_ptr != nullptr)
        if(has_padded_seqlen_k)
        {
            // TODO: this may need tuning
            return dim3(nhead_,
                        batch_size_,
                        ck_tile::integer_divide_ceil(seqlen_q_, FmhaPipeline::kM0) *
                            ck_tile::integer_divide_ceil(hdim_v_, FmhaPipeline::kN1));
        }
        else
        {
            // TODO: this may need tuning
            return dim3(ck_tile::integer_divide_ceil(seqlen_q_, FmhaPipeline::kM0) *
                            ck_tile::integer_divide_ceil(hdim_v_, FmhaPipeline::kN1),
                        nhead_,
                        batch_size_);
        }
    }

    CK_TILE_DEVICE static constexpr auto GetTileIndex(const Kargs& kargs)
    {
        using namespace ck_tile;

        bool has_padded_seqlen_k = false;

        if constexpr(kIsGroupMode)
            has_padded_seqlen_k = (kargs.seqlen_k_ptr != nullptr);

        if(has_padded_seqlen_k)
        {
            // const index_t num_tile_m0 = seqlen_q / kM0;
            const index_t num_tile_n1 =
                ck_tile::integer_divide_ceil(kargs.hdim_v, FmhaPipeline::kN1);

            const index_t i_block = blockIdx.z;
            const index_t i_nhead = blockIdx.x;
            const index_t i_batch = blockIdx.y;

            const auto f = [](index_t dividend, index_t divisor) {
                index_t quotient = dividend / divisor;
                index_t modulus  = dividend - quotient * divisor;
                return ck_tile::make_tuple(quotient, modulus);
            };

            const auto [i_tile_m, i_tile_n] = f(i_block, num_tile_n1);

            if constexpr(kHasMask)
            {
                // assume that num_tile_n1 is always 1
                return ck_tile::make_tuple(gridDim.z - 1 - i_tile_m, i_tile_n, i_nhead, i_batch);
            }
            else
            {
                return ck_tile::make_tuple(i_tile_m, i_tile_n, i_nhead, i_batch);
            }
        }
        else
        {
            // const index_t num_tile_m0 = seqlen_q / kM0;
            const index_t num_tile_n1 =
                ck_tile::integer_divide_ceil(kargs.hdim_v, FmhaPipeline::kN1);

            const index_t i_block = blockIdx.x;
            const index_t i_nhead = blockIdx.y;
            const index_t i_batch = blockIdx.z;

            const auto f = [](index_t dividend, index_t divisor) {
                index_t quotient = dividend / divisor;
                index_t modulus  = dividend - quotient * divisor;
                return ck_tile::make_tuple(quotient, modulus);
            };

            const auto [i_tile_m, i_tile_n] = f(i_block, num_tile_n1);

            if constexpr(kHasMask)
            {
                // assume that num_tile_n1 is always 1
                return ck_tile::make_tuple(gridDim.x - 1 - i_tile_m, i_tile_n, i_nhead, i_batch);
            }
            else
            {
                return ck_tile::make_tuple(i_tile_m, i_tile_n, i_nhead, i_batch);
            }
        }
    }

    CK_TILE_HOST static constexpr auto BlockSize() { return dim3(kBlockSize); }

    CK_TILE_HOST_DEVICE static constexpr ck_tile::index_t GetSmemSize()
    {
        return ck_tile::max(FmhaPipeline::GetSmemSize(), EpiloguePipeline::GetSmemSize());
    }

    CK_TILE_DEVICE void operator()(Kargs kargs) const
    {
        using namespace ck_tile;

        // allocate LDS
        __shared__ char smem_ptr[GetSmemSize()];

        // divide problem
        const auto [i_tile_m, i_tile_n, i_nhead, i_batch] = GetTileIndex(kargs);

        const index_t i_m0 = __builtin_amdgcn_readfirstlane(i_tile_m * FmhaPipeline::kM0);
        const index_t i_n1 = __builtin_amdgcn_readfirstlane(i_tile_n * FmhaPipeline::kN1);

        long_index_t batch_offset_q       = 0;
        long_index_t batch_offset_k       = 0;
        long_index_t batch_offset_v       = 0;
        long_index_t batch_offset_bias    = 0;
        long_index_t batch_offset_randval = 0;
        long_index_t batch_offset_lse     = 0;
        long_index_t batch_offset_o       = 0;

        if constexpr(kIsGroupMode)
        {
            // get starting offset for each batch
            const long_index_t query_start = kargs.seqstart_q_ptr[i_batch];
            const long_index_t key_start   = kargs.seqstart_k_ptr[i_batch];

            batch_offset_q = query_start * kargs.stride_q;
            batch_offset_k = key_start * kargs.stride_k;
            if constexpr(std::is_same_v<VLayout, ck_tile::tensor_layout::gemm::RowMajor>)
            {
                batch_offset_v = key_start * kargs.stride_v;
            }
            else
            {
                batch_offset_v = key_start;
            }
            if constexpr(BiasEnum == BlockAttentionBiasEnum::ELEMENTWISE_BIAS)
            {
                batch_offset_bias = query_start * kargs.stride_bias;
            }
            if constexpr(kStoreLSE)
            {
                batch_offset_lse = query_start;
            }
            if constexpr(kHasDropout)
            {
                batch_offset_randval = query_start * kargs.stride_randval;
            }
            batch_offset_o = query_start * kargs.stride_o;

            // get real # queries & # keys under group mode
            const auto adjusted_seqstart_q_ptr = kargs.seqstart_q_ptr + i_batch;
            kargs.seqlen_q = adjusted_seqstart_q_ptr[1] - adjusted_seqstart_q_ptr[0];

            if constexpr(kSkipMinSeqlenQ)
            {
                if(kargs.seqlen_q <= kargs.min_seqlen_q)
                {
                    return;
                }
            }

            // # of required blocks is different in each groups, terminate unnecessary blocks
            // earlier
            if(kargs.seqlen_q <= i_m0)
            {
                return;
            }

            if(kargs.seqlen_k_ptr != nullptr)
            {
                kargs.seqlen_k = kargs.seqlen_k_ptr[i_batch];
            }
            else
            {
                const auto adjusted_seqstart_k_ptr = kargs.seqstart_k_ptr + i_batch;
                kargs.seqlen_k = adjusted_seqstart_k_ptr[1] - adjusted_seqstart_k_ptr[0];
            }
        }
        else
        {
            batch_offset_q = static_cast<long_index_t>(i_batch) * kargs.batch_stride_q;
            batch_offset_k = static_cast<long_index_t>(i_batch) * kargs.batch_stride_k;
            batch_offset_v = static_cast<long_index_t>(i_batch) * kargs.batch_stride_v;
            if constexpr(BiasEnum == BlockAttentionBiasEnum::ELEMENTWISE_BIAS)
            {
                batch_offset_bias = static_cast<long_index_t>(i_batch) * kargs.batch_stride_bias;
            }
            if constexpr(kStoreLSE)
            {
                batch_offset_lse = static_cast<long_index_t>(i_batch) * kargs.batch_stride_lse;
            }
            if constexpr(kHasDropout)
            {
                batch_offset_randval =
                    static_cast<long_index_t>(i_batch) * kargs.batch_stride_randval;
            }
            batch_offset_o = static_cast<long_index_t>(i_batch) * kargs.batch_stride_o;
        }

        // for simplicity, batch stride we just modify the pointer
        const QDataType* q_ptr = reinterpret_cast<const QDataType*>(kargs.q_ptr) +
                                 static_cast<long_index_t>(i_nhead) * kargs.nhead_stride_q +
                                 batch_offset_q;
        const KDataType* k_ptr =
            reinterpret_cast<const KDataType*>(kargs.k_ptr) +
            static_cast<long_index_t>(i_nhead / kargs.nhead_ratio_qk) * kargs.nhead_stride_k +
            batch_offset_k;
        const VDataType* v_ptr =
            reinterpret_cast<const VDataType*>(kargs.v_ptr) +
            static_cast<long_index_t>(i_nhead / kargs.nhead_ratio_qk) * kargs.nhead_stride_v +
            batch_offset_v;
        ODataType* o_ptr = reinterpret_cast<ODataType*>(kargs.o_ptr) +
                           static_cast<long_index_t>(i_nhead) * kargs.nhead_stride_o +
                           batch_offset_o;

        // Q/K/V DRAM and DRAM window
        const auto q_dram = [&]() {
            const auto q_dram_naive = make_naive_tensor_view<address_space_enum::global>(
                q_ptr,
                make_tuple(kargs.seqlen_q, kargs.hdim_q),
                make_tuple(kargs.stride_q, 1),
                number<FmhaPipeline::kAlignmentQ>{},
                number<1>{});
            if constexpr(FmhaPipeline::kQLoadOnce)
            {
                return pad_tensor_view(
                    q_dram_naive,
                    make_tuple(number<FmhaPipeline::kM0>{}, number<FmhaPipeline::kSubQKHeaddim>{}),
                    sequence<kPadSeqLenQ, kPadHeadDimQ>{});
            }
            else
            {
                return pad_tensor_view(
                    q_dram_naive,
                    make_tuple(number<FmhaPipeline::kM0>{}, number<FmhaPipeline::kK0>{}),
                    sequence<kPadSeqLenQ, kPadHeadDimQ>{});
            }
        }();
        const auto k_dram = [&]() {
            const auto k_dram_naive = make_naive_tensor_view<address_space_enum::global>(
                k_ptr,
                make_tuple(kargs.seqlen_k, kargs.hdim_q),
                make_tuple(kargs.stride_k, 1),
                number<FmhaPipeline::kAlignmentK>{},
                number<1>{});

            constexpr bool kPadSeqLenK_ = kUseAsyncCopy ? kPadSeqLenK : false;
            return pad_tensor_view(
                k_dram_naive,
                make_tuple(number<FmhaPipeline::kN0>{}, number<FmhaPipeline::kK0>{}),
                sequence<kPadSeqLenK_, kPadHeadDimQ>{});
        }();
        const auto v_dram = [&]() {
            if constexpr(std::is_same_v<VLayout, ck_tile::tensor_layout::gemm::RowMajor>)
            {
                const auto v_dram_naive = make_naive_tensor_view<address_space_enum::global>(
                    v_ptr,
                    make_tuple(kargs.seqlen_k, kargs.hdim_v),
                    make_tuple(kargs.stride_v, 1),
                    number<FmhaPipeline::kAlignmentV>{},
                    number<1>{});

                const auto v_dram_transposed =
                    transform_tensor_view(v_dram_naive,
                                          make_tuple(make_pass_through_transform(kargs.hdim_v),
                                                     make_pass_through_transform(kargs.seqlen_k)),
                                          make_tuple(sequence<1>{}, sequence<0>{}),
                                          make_tuple(sequence<0>{}, sequence<1>{}));

                constexpr bool kPadSeqLenK_ = kUseAsyncCopy ? kPadSeqLenK : false;
                return pad_tensor_view(
                    v_dram_transposed,
                    make_tuple(number<FmhaPipeline::kN1>{}, number<FmhaPipeline::kK1>{}),
                    sequence<kPadHeadDimV, kPadSeqLenK_>{});
            }
            else
            {
                const auto v_dram_naive = make_naive_tensor_view<address_space_enum::global>(
                    v_ptr,
                    make_tuple(kargs.hdim_v, kargs.seqlen_k),
                    make_tuple(kargs.stride_v, 1),
                    number<FmhaPipeline::kAlignmentV>{},
                    number<1>{});

                constexpr bool kPadHeadDimV_ = kUseAsyncCopy ? kPadHeadDimV : false;
                return pad_tensor_view(
                    v_dram_naive,
                    make_tuple(number<FmhaPipeline::kN1>{}, number<FmhaPipeline::kK1>{}),
                    sequence<kPadHeadDimV_, kPadSeqLenK>{});
            }
        }();

        auto q_dram_window = make_tile_window(
            q_dram,
            [&]() {
                if constexpr(FmhaPipeline::kQLoadOnce)
                    return make_tuple(number<FmhaPipeline::kM0>{},
                                      number<FmhaPipeline::kSubQKHeaddim>{});
                else
                    return make_tuple(number<FmhaPipeline::kM0>{}, number<FmhaPipeline::kK0>{});
            }(),
            {i_m0, 0});

        auto k_dram_window = make_tile_window(
            k_dram, make_tuple(number<FmhaPipeline::kN0>{}, number<FmhaPipeline::kK0>{}), {0, 0});

        auto v_dram_window =
            make_tile_window(v_dram,
                             make_tuple(number<FmhaPipeline::kN1>{}, number<FmhaPipeline::kK1>{}),
                             {i_n1, 0});
        /// FIXME: Before C++20, capturing structured binding variables are not supported. Remove
        /// following copy capture of the 'i_nhead' if in C++20
        const auto bias_dram_window = [&, i_nhead_ = i_nhead]() {
            constexpr auto bias_dram_window_lengths =
                make_tuple(number<FmhaPipeline::kM0>{}, number<FmhaPipeline::kN0>{});
            if constexpr(BiasEnum == BlockAttentionBiasEnum::ELEMENTWISE_BIAS)
            {
                const BiasDataType* bias_ptr =
                    reinterpret_cast<const BiasDataType*>(kargs.bias_ptr) +
                    static_cast<long_index_t>(i_nhead_) * kargs.nhead_stride_bias +
                    batch_offset_bias;

                const auto bias_dram = [&]() {
                    const auto bias_dram_naive = make_naive_tensor_view<address_space_enum::global>(
                        bias_ptr,
                        make_tuple(kargs.seqlen_q, kargs.seqlen_k),
                        make_tuple(kargs.stride_bias, 1),
                        number<FmhaPipeline::kAlignmentBias>{},
                        number<1>{});

                    return pad_tensor_view(bias_dram_naive,
                                           bias_dram_window_lengths,
                                           sequence<kPadSeqLenQ, kPadSeqLenK>{});
                }();

                return make_tile_window(bias_dram, bias_dram_window_lengths, {i_m0, 0});
            }
            else
            {
                return make_null_tile_window(bias_dram_window_lengths);
            }
        }();

        // lse
        auto lse_dram_window = [&, i_nhead_ = i_nhead]() {
            constexpr auto lse_dram_window_lengths = make_tuple(number<FmhaPipeline::kM0>{});
            if constexpr(kStoreLSE)
            {
                LSEDataType* lse_ptr =
                    reinterpret_cast<LSEDataType*>(kargs.lse_ptr) +
                    static_cast<long_index_t>(i_nhead_) * kargs.nhead_stride_lse + batch_offset_lse;

                const auto lse_dram = [&]() {
                    const auto lse_dram_naive = make_naive_tensor_view<address_space_enum::global>(
                        lse_ptr,
                        make_tuple(kargs.seqlen_q),
                        make_tuple(1),
                        number<1>{},
                        number<1>{});

                    return pad_tensor_view(
                        lse_dram_naive, lse_dram_window_lengths, sequence<kPadSeqLenQ>{});
                }();

                return make_tile_window(lse_dram, lse_dram_window_lengths, {i_m0});
            }
            else
            {
                return make_null_tile_window(lse_dram_window_lengths);
            }
        }();

        auto dropout = [&, i_nhead_ = i_nhead, i_batch_ = i_batch]() {
            if constexpr(kHasDropout)
            {
                return BlockDropout{i_batch_,
                                    i_nhead_,
                                    kargs.num_head_q,
                                    kargs.is_drop_seed_offset_from_host ? kargs.drop_seed.val
                                                                        : *kargs.drop_seed.ptr,
                                    kargs.is_drop_seed_offset_from_host ? kargs.drop_offset.val
                                                                        : *kargs.drop_offset.ptr,
                                    kargs.rp_undrop,
                                    kargs.p_undrop_in_uint8_t,
                                    kargs.is_store_randval};
            }
            else
            {
                return NullBlockDropout{};
            };
        }();

        auto randval_dram_window = [&, i_nhead_ = i_nhead]() {
            constexpr auto randval_dram_window_lengths =
                make_tuple(number<FmhaPipeline::kM0>{}, number<FmhaPipeline::kN0>{});
            if constexpr(kHasDropout)
            {
                RandValOutputDataType* rand_val_ptr =
                    reinterpret_cast<RandValOutputDataType*>(kargs.rand_val_ptr) +
                    static_cast<long_index_t>(i_nhead_) * kargs.nhead_stride_randval +
                    batch_offset_randval;

                const auto randval_dram = [&]() {
                    const auto randval_dram_naive =
                        make_naive_tensor_view<address_space_enum::global>(
                            rand_val_ptr,
                            make_tuple(kargs.seqlen_q, kargs.seqlen_k),
                            make_tuple(kargs.stride_randval, 1),
                            number<1>{},
                            number<1>{});

                    return pad_tensor_view(randval_dram_naive,
                                           randval_dram_window_lengths,
                                           sequence<kPadSeqLenQ, kPadSeqLenK>{});
                }();

                return make_tile_window(randval_dram, randval_dram_window_lengths, {i_m0, 0});
            }
            else
            {
                return make_null_tile_window(randval_dram_window_lengths);
            }
        }();

        FmhaMask mask = [&]() {
            if constexpr(kHasMask)
                return ck_tile::make_generic_attention_mask_from_lr_window<FmhaMask>(
                    kargs.window_size_left,
                    kargs.window_size_right,
                    kargs.seqlen_q,
                    kargs.seqlen_k,
                    kargs.mask_type == GenericAttentionMaskEnum::MASK_FROM_TOP_LEFT);
            else
                return FmhaMask{kargs.seqlen_q, kargs.seqlen_k};
        }();

        // WA i_batch capture structure binding before c++20
        auto position_encoding = [&, i_batch_ = i_batch, i_nhead_ = i_nhead]() {
            if constexpr(BiasEnum == BlockAttentionBiasEnum::ALIBI)
            {
                // data loading, shared by entire wg
                // TODO: how to use s_read?
                SaccDataType slope =
                    *(reinterpret_cast<const SaccDataType*>(kargs.alibi_slope_ptr) +
                      i_batch_ * kargs.alibi_slope_stride + i_nhead_);
#if CK_TILE_FMHA_FWD_FAST_EXP2
                slope *= ck_tile::log2e_v<>;
#endif
                if constexpr(kHasMask)
                {
                    return make_alibi_from_lr_mask<SaccDataType, true>(slope,
                                                                       kargs.window_size_left,
                                                                       kargs.window_size_right,
                                                                       kargs.seqlen_q,
                                                                       kargs.seqlen_k,
                                                                       kargs.mask_type);
                }
                else
                {
                    return Alibi<SaccDataType, true>{
                        slope, kargs.seqlen_q, kargs.seqlen_k, AlibiMode::FROM_BOTTOM_RIGHT};
                }
            }
            else
            {
                return EmptyPositionEncoding<SaccDataType>{};
            }
        }();

        AttentionVariant variant;
        const auto variant_params = [&] {
            if constexpr(kHasLogitsSoftCap)
            {
                return ck_tile::LogitsSoftCapParams<FmhaMask, CK_TILE_FMHA_FWD_FAST_EXP2>{
                    mask, kargs.scale_s, kargs.logits_soft_cap, kargs.logits_soft_cap_rcp};
            }
            else
            {
                return ck_tile::StandardAttentionParams<FmhaMask>{mask, kargs.scale_s};
            }
        }();

        BlockIndices block_indices{i_batch, i_nhead, i_nhead / kargs.nhead_ratio_qk};

        auto o_acc_tile = [&]() {
            if constexpr(kDoFp8StaticQuant)
            {
                return FmhaPipeline{}(
                    q_dram_window,
                    identity{}, // q_element_func
                    k_dram_window,
                    identity{}, // k_element_func
                    v_dram_window,
                    identity{}, // v_element_func
                    bias_dram_window,
                    identity{}, // bias_element_func
                    randval_dram_window,
                    lse_dram_window,
                    identity{},                                          // lse_element_func
                    identity{},                                          // s_acc_element_func
                    scales{kargs.scale_p},                               // p_compute_element_func
                    composes(saturates<fp8_t>{}, scales{kargs.scale_o}), // o_acc_element_func
                    mask,
                    position_encoding,
                    kargs.scale_s,
                    variant,
                    variant_params,
                    block_indices,
                    smem_ptr,
                    dropout);
            }
            else
            {
                return FmhaPipeline{}(q_dram_window,
                                      k_dram_window,
                                      v_dram_window,
                                      bias_dram_window,
                                      randval_dram_window,
                                      lse_dram_window,
                                      mask,
                                      position_encoding,
                                      kargs.scale_s,
                                      variant,
                                      variant_params,
                                      block_indices,
                                      smem_ptr,
                                      dropout);
            }
        }();

        // O DRAM and O DRAM window
        auto o_dram = [&]() {
            const auto o_dram_naive = make_naive_tensor_view<address_space_enum::global>(
                o_ptr,
                make_tuple(kargs.seqlen_q, kargs.hdim_v),
                make_tuple(kargs.stride_o, 1),
                number<FmhaPipeline::kAlignmentO>{},
                number<1>{});

            return pad_tensor_view(
                o_dram_naive,
                make_tuple(number<FmhaPipeline::kM0>{}, number<FmhaPipeline::kN1>{}),
                sequence<kPadSeqLenQ, kPadHeadDimV>{});
        }();

        auto o_dram_window =
            make_tile_window(o_dram,
                             make_tuple(number<FmhaPipeline::kM0>{}, number<FmhaPipeline::kN1>{}),
                             {i_m0, i_n1});

        EpiloguePipeline{}(o_dram_window, o_acc_tile);
    }
};

namespace torch_itfs {
namespace {
struct host_args
{
    ck_tile::index_t batch;
    ck_tile::index_t seqlen_q;
    ck_tile::index_t seqlen_k;
    ck_tile::index_t hdim_q;
    ck_tile::index_t hdim_v;
    ck_tile::index_t nhead_q;
    ck_tile::index_t nhead_k;

    float scale_s;

    const void* q_ptr;
    ck_tile::index_t stride_q;
    ck_tile::index_t nhead_stride_q;
    ck_tile::index_t batch_stride_q;

    const void* k_ptr;
    ck_tile::index_t stride_k;
    ck_tile::index_t nhead_stride_k;
    ck_tile::index_t batch_stride_k;

    const void* v_ptr;
    ck_tile::index_t stride_v;
    ck_tile::index_t nhead_stride_v;
    ck_tile::index_t batch_stride_v;

    void* o_ptr;
    ck_tile::index_t stride_o;
    ck_tile::index_t nhead_stride_o;
    ck_tile::index_t batch_stride_o;
};

//////////////////////////////////////////////////////////////////////////////////////
template <typename DataType>
struct get_kernel
{
    using fmha_dtype = DataType;
    //                                        M0   N0  K0   N1   K1
    using fmha_block_tile = ck_tile::sequence<256, 32, 128, 128, 32, 128>;

    using fmha_warp_gemm_shape = ck_tile::sequence<32, 32, 16>;

    using fmha_block_warps = ck_tile::sequence<8, 1, 1>;

    using fmha_shape = ck_tile::TileFmhaShape<fmha_block_tile,
                                              fmha_block_warps,
                                              fmha_warp_gemm_shape,
                                              fmha_block_warps,
                                              fmha_warp_gemm_shape,
                                              true // IsVLayoutRowMajor
                                              >;

    using fmha_traits = ck_tile::TileFmhaTraits<true,  // kPadSeqLenQ
                                                true,  // kPadSeqLenK
                                                false, // kPadHeadDimQ
                                                false, // kPadHeadDimV
                                                false, // kHasLogitsSoftCap
                                                ck_tile::BlockAttentionBiasEnum::NO_BIAS,
                                                false, // kHasBiasGrad
                                                false, // kStoreLSE
                                                false, // kHasDropout
                                                false, // kDoFp8StaticQuant
                                                -1,    // kBlockPerCu
                                                false  // kSkipMinSeqlenQ
                                                >;

    using fmha_variant =
        ck_tile::ComposedAttention<false * ck_tile::LOGITS_SOFT_CAP, // VARIANT_CODE
                                   CK_TILE_FMHA_FWD_FAST_EXP2        // UseExp2
                                   >;

    using fmha_mask = ck_tile::SimplifiedGenericAttentionMask<false // IsMasking
                                                              >;

    using fmha_problem = ck_tile::BlockFmhaPipelineProblem<
        typename FmhaFwdTypeConfig<fmha_dtype>::QDataType,
        typename FmhaFwdTypeConfig<fmha_dtype>::KDataType,
        typename FmhaFwdTypeConfig<fmha_dtype>::VDataType,
        typename FmhaFwdTypeConfig<fmha_dtype>::SaccDataType,
        typename FmhaFwdTypeConfig<fmha_dtype>::SMPLComputeDataType,
        typename FmhaFwdTypeConfig<fmha_dtype>::BiasDataType,
        typename FmhaFwdTypeConfig<fmha_dtype>::RandValOutputDataType,
        typename FmhaFwdTypeConfig<fmha_dtype>::LSEDataType,
        typename FmhaFwdTypeConfig<fmha_dtype>::PDataType,
        typename FmhaFwdTypeConfig<fmha_dtype>::OaccDataType,
        typename FmhaFwdTypeConfig<fmha_dtype>::ODataType,
        fmha_shape,
        false, // kIsGroupMode
        fmha_variant,
        fmha_mask,
        fmha_traits>;

    using fmha_pipeline = aiter::BlockFmhaPipelineQRKSVS<fmha_problem>;

    using fmha_epilogue = ck_tile::Default2DEpilogue<
        ck_tile::Default2DEpilogueProblem<typename FmhaFwdTypeConfig<fmha_dtype>::OaccDataType,
                                          typename FmhaFwdTypeConfig<fmha_dtype>::ODataType,
                                          true, // kPadM
                                          true  // kPadM
                                          >>;

    using type = aiter::FmhaFwdKernel<fmha_pipeline, fmha_epilogue>;
};

template <typename DataType>
using get_kernel_t = typename get_kernel<DataType>::type;

template <typename Kernel>
void launch(const host_args& args)
{
    auto kargs = Kernel::MakeKargsImpl(args.q_ptr,
                                       args.k_ptr,
                                       args.v_ptr,
                                       nullptr, // bias_ptr
                                       nullptr, // rand_val_ptr
                                       nullptr, // lse_ptr
                                       args.o_ptr,
                                       args.seqlen_q,
                                       args.seqlen_k,
                                       args.hdim_q,
                                       args.hdim_v,
                                       args.nhead_q,
                                       args.nhead_q / args.nhead_k,
                                       args.scale_s,
                                       1.0f, // scale_p
                                       1.0f, // scale_o
                                       0.0f, // logits_soft_cap
                                       args.stride_q,
                                       args.stride_k,
                                       args.stride_v,
                                       0, // stride_bias
                                       0, // stride_randval
                                       args.stride_o,
                                       args.nhead_stride_q,
                                       args.nhead_stride_k,
                                       args.nhead_stride_v,
                                       0, // nhead_stride_bias
                                       0, // nhead_stride_randval
                                       0, // nhead_stride_lse
                                       args.nhead_stride_o,
                                       args.batch_stride_q,
                                       args.batch_stride_k,
                                       args.batch_stride_v,
                                       0, // batch_stride_bias
                                       0, // batch_stride_randval
                                       0, // batch_stride_lse
                                       args.batch_stride_o,
                                       0,                         // window_size_left
                                       0,                         // window_size_right
                                       0,                         // mask_type
                                       0.0f,                      // p_drop
                                       false,                     // s_randval
                                       std::make_pair(0UL, 0UL)); // drop_seed_offset

    dim3 grids = Kernel::GridSize(args.batch, args.nhead_q, args.seqlen_q, args.hdim_v, false);
    constexpr dim3 blocks                  = Kernel::BlockSize();
    constexpr ck_tile::index_t kBlockPerCu = Kernel::kBlockPerCu;

    auto stream = at::cuda::getCurrentHIPStream().stream();
    ck_tile::stream_config stream_config{stream};

    [[maybe_unused]] const float time = ck_tile::launch_kernel(
        stream_config,
        ck_tile::make_kernel<blocks.x, kBlockPerCu>(Kernel{}, grids, blocks, 0, kargs));
}
} // namespace
//////////////////////////////////////////////////////////////////////////////////////

std::vector<at::Tensor> poyenc_mha_v3_fwd(const at::Tensor& q, // [b, sq, hq, d]
                                          const at::Tensor& k, // [b, sk, hk, d]
                                          const at::Tensor& v, // [b, sk, hk, d_v]
                                          float softmax_scale)
{
    auto q_dtype = q.dtype();
    TORCH_CHECK(q_dtype == at::ScalarType::Half || q_dtype == at::ScalarType::BFloat16,
                "FlashAttention only support fp16 and bf16 data type");

    TORCH_CHECK(k.dtype() == q_dtype, "query and key must have the same dtype");
    TORCH_CHECK(v.dtype() == q_dtype, "query and value must have the same dtype");

    CHECK_DEVICE(q);
    CHECK_DEVICE(k);
    CHECK_DEVICE(v);

    TORCH_CHECK(q.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    TORCH_CHECK(k.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    TORCH_CHECK(v.stride(-1) == 1, "Input tensor must have contiguous last dimension");

    const auto sizes = q.sizes();

    const int batch_size  = sizes[0];
    int seqlen_q          = sizes[1];
    int num_heads         = sizes[2];
    const int head_size_q = sizes[3];
    const int head_size_v = v.sizes()[3];
    const int seqlen_k    = k.size(1);
    const int num_heads_k = k.size(2);
    TORCH_CHECK(batch_size > 0, "batch size must be positive");
    TORCH_CHECK(head_size_q <= 256, "CK only supports head dimension at most 256");
    TORCH_CHECK(head_size_v <= 256, "CK only supports head dimension at most 256");
    TORCH_CHECK(head_size_q % 8 == 0,
                "query, key, value, and out_ must have a head_size_q that is a multiple of 8");
    TORCH_CHECK(head_size_v % 8 == 0,
                "query, key, value, and out_ must have a head_size_q that is a multiple of 8");
    TORCH_CHECK(
        num_heads % num_heads_k == 0,
        "ck_tile::number of heads in key/value must divide ck_tile::number of heads in query");

    CHECK_SHAPE(q, batch_size, seqlen_q, num_heads, head_size_q);
    CHECK_SHAPE(k, batch_size, seqlen_k, num_heads_k, head_size_q);
    CHECK_SHAPE(v, batch_size, seqlen_k, num_heads_k, head_size_v);

    host_args args;

    args.batch    = batch_size;
    args.seqlen_q = seqlen_q;
    args.seqlen_k = seqlen_k;
    args.hdim_q   = head_size_q;
    args.hdim_v   = head_size_v;
    args.nhead_q  = num_heads;
    args.nhead_k  = num_heads_k;

    args.scale_s = softmax_scale;

    args.q_ptr          = q.data_ptr();
    args.batch_stride_q = q.stride(0);
    args.stride_q       = q.stride(1);
    args.nhead_stride_q = q.stride(2);

    args.k_ptr          = k.data_ptr();
    args.batch_stride_k = k.stride(0);
    args.stride_k       = k.stride(1);
    args.nhead_stride_k = k.stride(2);

    args.v_ptr          = v.data_ptr();
    args.batch_stride_v = v.stride(0);
    args.stride_v       = v.stride(1);
    args.nhead_stride_v = v.stride(2);

    auto opts = q.options();
    at::Tensor out =
        torch::empty({batch_size, seqlen_q, num_heads, head_size_v}, opts.dtype(q_dtype));

    args.o_ptr          = out.data_ptr();
    args.batch_stride_o = out.stride(0);
    args.stride_o       = out.stride(1);
    args.nhead_stride_o = out.stride(2);

    if(q_dtype == at::ScalarType::Half)
    {
        launch<get_kernel_t<FmhaFwdFp16>>(args);
    }
    else if(q_dtype == at::ScalarType::BFloat16)
    {
        launch<get_kernel_t<FmhaFwdBf16>>(args);
    }

    return {out};
}

} // namespace torch_itfs
} // namespace aiter
