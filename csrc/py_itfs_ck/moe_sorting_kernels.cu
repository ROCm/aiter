// SPDX-License-Identifier: MIT
// Copyright (c) 2024, Advanced Micro Devices, Inc. All rights reserved.
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include "py_itfs_common.h"

#include "moe_sorting_api.hpp"

void moe_sorting_fwd(torch::Tensor &topk_ids,              // [m, topk]
                     torch::Tensor &topk_weights,          // [m, topk]
                     torch::Tensor &sorted_token_ids,      // [max_num_tokens_padded]
                     torch::Tensor &sorted_weights,        // [max_num_tokens_padded]
                     torch::Tensor &sorted_expert_ids,     // [max_num_m_blocks]
                     torch::Tensor &total_tokens_post_pad, // [1]
                     torch::Tensor &moe_buf,               // [max_num_tokens_padded]
                     int num_experts,
                     int unit_size)
{
    auto dtype = topk_ids.dtype();

    auto dtype_str = torchDTypeToStr(topk_ids.dtype());
    int num_tokens = topk_ids.size(0);
    int topk = topk_ids.size(1);
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    moe_sorting({
                    dtype_str, // index_type
                    "fp32"     // weight_type; // currently always float
                },
                {topk_ids.data_ptr(),              // p_topk_ids
                 topk_weights.data_ptr(),          // p_weights
                 sorted_token_ids.data_ptr(),      // p_sorted_token_ids
                 sorted_weights.data_ptr(),        // p_sorted_weights
                 sorted_expert_ids.data_ptr(),     // p_sorted_expert_ids
                 total_tokens_post_pad.data_ptr(), // p_total_tokens_post_pad
                 moe_buf.data_ptr(),               // p_moe_buf
                 num_tokens, unit_size, num_experts, topk, (int)moe_buf.nbytes()},
                {stream});
}
